#include "hip/hip_runtime.h"
#include "itsolver_gpu.h"

void createSolver(itsolver *bicg)
{
  //Init variables ("public")
  int nrows = bicg->nrows;
  int blocks = bicg->blocks;
  bicg->maxIt=1000;
  bicg->tolmax=1.0e-30; //cv_mem->cv_reltol CAMP selected accuracy (1e-8) //1e-10;//1e-6
#ifndef CSR_SPMV
  bicg->mattype=0;
  printf("BCG Mattype=CSR\n");
#else
  bicg->mattype=1; //CSC
  printf("BCG Mattype=CSC\n");
#endif

  //Auxiliary vectors ("private")
  double ** dr0 = &bicg->dr0;
  double ** dr0h = &bicg->dr0h;
  double ** dn0 = &bicg->dn0;
  double ** dp0 = &bicg->dp0;
  double ** dt = &bicg->dt;
  double ** ds = &bicg->ds;
  double ** dAx2 = &bicg->dAx2;
  double ** dy = &bicg->dy;
  double ** dz = &bicg->dz;
  double ** daux = &bicg->daux;
  double ** ddiag = &bicg->ddiag;

  //Allocate
  hipMalloc(dr0,nrows*sizeof(double));
  hipMalloc(dr0h,nrows*sizeof(double));
  hipMalloc(dn0,nrows*sizeof(double));
  hipMalloc(dp0,nrows*sizeof(double));
  hipMalloc(dt,nrows*sizeof(double));
  hipMalloc(ds,nrows*sizeof(double));
  hipMalloc(dAx2,nrows*sizeof(double));
  hipMalloc(dy,nrows*sizeof(double));
  hipMalloc(dz,nrows*sizeof(double));
  hipMalloc(ddiag,nrows*sizeof(double));
  hipMalloc(daux,nrows*sizeof(double));
  bicg->aux=(double*)malloc(sizeof(double)*blocks);

}

int nextPowerOfTwo(int v){

  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;

  //printf("nextPowerOfTwo %d", v);

  return v;
}


//Based on
// https://github.com/scipy/scipy/blob/3b36a574dc657d1ca116f6e230be694f3de31afc/scipy/sparse/sparsetools/csr.h#L363
void CSRtoCSCandCSCtoCSR(int n_row, int n_col, int* Ap, int* Aj, double* Ax, int* Bp, int* Bi, double* Bx){

  int nnz=Ap[n_row];

  memset(Bp, 0, (n_row+1)*sizeof(int));

  for (int n = 0; n < nnz; n++){
    Bp[Aj[n]]++;
  }

  //cumsum the nnz per column to get Bp[]
  for(int col = 0, cumsum = 0; col < n_col; col++){
    int temp  = Bp[col];
    Bp[col] = cumsum;
    cumsum += temp;
  }
  Bp[n_col] = nnz;

  for(int row = 0; row < n_row; row++){
    for(int jj = Ap[row]; jj < Ap[row+1]; jj++){
      int col  = Aj[jj];
      int dest = Bp[col];

      Bi[dest] = row;
      Bx[dest] = Ax[jj];

      Bp[col]++;
    }
  }

  for(int col = 0, last = 0; col <= n_col; col++){
    int temp  = Bp[col];
    Bp[col] = last;
    last    = temp;
  }

}

void CSRtoCSC(itsolver *bicg){

#ifdef TEST_CSRtoCSC

  //Example configuration taken from KLU Sparse pdf
  int n_row=3;
  int n_col=n_row;
  int nnz=6;
  int Ap[n_row+1]={0,1,3,6};
  int Aj[nnz]={0,0,1,0,1,2};
  double Ax[nnz]={5.,4.,2.,3.,1.,8.};
  int* Bp=(int*)malloc((n_row+1)*sizeof(int));
  int* Bi=(int*)malloc(nnz*sizeof(int));
  double* Bx=(int*)malloc(nnz*sizeof(double));
#else

  //hipMemcpy(bicg->dA,bicg->djA,bicg->nnz*sizeof(int),hipMemcpyDeviceToHost);
  //hipMemcpy(bicg->iA,bicg->diA,(bicg->nrows+1)*sizeof(int),hipMemcpyDeviceToHost);

  int n_row=bicg->nrows;
  int n_col=n_row;
  int nnz=bicg->nnz;
  int* Ap=bicg->iA;
  int* Aj=bicg->jA;
  double* Ax=bicg->A;
  int* Bp=(int*)malloc((bicg->nrows+1)*sizeof(int));
  int* Bi=(int*)malloc(bicg->nnz*sizeof(int));
  double* Bx=(double*)malloc(nnz*sizeof(double));

#endif

  CSRtoCSCandCSCtoCSR(n_row,n_col,Ap,Aj,Ax,Bp,Bi,Bx);

#ifdef TEST_CSRtoCSC

  //Correct result:
  //int Cp[n_row+1]={0,3,5,6};
  //int Ci[nnz]={0,1,2,1,2,2};
  //int Cx[nnz]={5,4,3,2,1,8};

  printf("Bp:\n");
  for(int i=0;i<=n_row;i++)
    printf("%d ",Bp[i]);
  printf("\n");
  printf("Bi:\n");
  for(int i=0;i<nnz;i++)
    printf("%d ",Bi[i]);
  printf("\n");
  printf("Bx:\n");
  for(int i=0;i<nnz;i++)
    printf("%-le ",Bx[i]);
  printf("\n");
  exit(0);

#else

  /*
  for(int i=0;i<bicg->nnz;i++)
    bicg->jA[i]=Bi[i];
  for(int i=0;i<=bicg->nrows;i++)
    bicg->iA[i]=Bp[i];

  hipMemcpy(bicg->djA,bicg->jA,bicg->nnz*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(bicg->diA,bicg->iA,(bicg->nrows+1)*sizeof(int),hipMemcpyHostToDevice);
   */

  hipMemcpy(bicg->diA,Bp,(bicg->nrows+1)*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(bicg->djA,Bi,bicg->nnz*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(bicg->dA,Bx,bicg->nnz*sizeof(double),hipMemcpyHostToDevice);

#endif

  free(Bp);
  free(Bi);
  free(Bx);

}

void CSCtoCSR(itsolver *bicg){

#ifdef TEST_CSCtoCSR

  //Example configuration taken from KLU Sparse pdf
  int n_row=3;
  int n_col=n_row;
  int nnz=6;
  int Ap[n_row+1]={0,3,5,6};
  int Aj[nnz]={0,1,2,1,2,2};
  double Ax[nnz]={5.,4.,3.,2.,1.,8.};
  int* Bp=(int*)malloc((n_row+1)*sizeof(int));
  int* Bi=(int*)malloc(nnz*sizeof(int));
  double* Bx=(double*)malloc(nnz*sizeof(double));

#else

  //hipMemcpy(bicg->iA,bicg->diA,(bicg->nrows+1)*sizeof(int),hipMemcpyDeviceToHost);
  //hipMemcpy(bicg->jA,bicg->djA,bicg->nnz*sizeof(int),hipMemcpyDeviceToHost);
  //hipMemcpy(bicg->A,bicg->dA,bicg->nnz*sizeof(double),hipMemcpyDeviceToHost);

  int n_row=bicg->nrows;
  int n_col=n_row;
  int nnz=bicg->nnz;
  int* Ap=bicg->iA;
  int* Aj=bicg->jA;
  double* Ax=bicg->A;
  int* Bp=(int*)malloc((bicg->nrows+1)*sizeof(int));
  int* Bi=(int*)malloc(bicg->nnz*sizeof(int));
  double* Bx=(double*)malloc(nnz*sizeof(double));

#endif

  CSRtoCSCandCSCtoCSR(n_row,n_col,Ap,Aj,Ax,Bp,Bi,Bx);

#ifdef TEST_CSCtoCSR

  //Correct result:
  //int Cp[n_row+1]={0,1,3,6};
  //int Ci[nnz]={0,0,1,0,1,2};
  //int Cx[nnz]={5,4,2,3,1,8};

  printf("Bp:\n");
  for(int i=0;i<=n_row;i++)
    printf("%d ",Bp[i]);
  printf("\n");
  printf("Bi:\n");
  for(int i=0;i<nnz;i++)
    printf("%d ",Bi[i]);
  printf("\n");
  printf("Bx:\n");
  for(int i=0;i<nnz;i++)
    printf("%-le ",Bx[i]);
  printf("\n");

  exit(0);

#else

  /*
  printf("Bp:\n");
  for(int i=0;i<=n_row;i++)
    printf("%d ",Bp[i]);
  printf("\n");
  printf("Bi:\n");
  for(int i=0;i<nnz;i++)
    printf("%d ",Bi[i]);
  printf("\n");
  printf("Bx:\n");
  for(int i=0;i<nnz;i++)
    printf("%-le ",Bx[i]);
  printf("\n");
   */

  /*

  for(int i=0;i<=bicg->nrows;i++)
    bicg->iA[i]=Bp[i];

  for(int i=0;i<bicg->nnz;i++){
    bicg->jA[i]=Bi[i];
    bicg->A[i]=Bx[i];
  }

  hipMemcpy(bicg->diA,bicg->iA,(bicg->nrows+1)*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(bicg->djA,bicg->jA,bicg->nnz*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(bicg->dA,bicg->A,bicg->nnz*sizeof(double),hipMemcpyHostToDevice);

   */


  hipMemcpy(bicg->diA,Bp,(bicg->nrows+1)*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(bicg->djA,Bi,bicg->nnz*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(bicg->dA,Bx,bicg->nnz*sizeof(double),hipMemcpyHostToDevice);


#endif

  free(Bp);
  free(Bi);
  free(Bx);

}

__device__
void dvcheck_input_gpud(double *x, int len, const char* s)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  //if(i<2)
  if(i<len)
  {
    printf("%s[%d]=%-le\n",s,i,x[i]);
  }
}

//todo instead sending all in one kernel, divide in 2 or 4 kernels with streams and check if
//cuda reassigns better the resources
//todo profiling del dot y ver cuanta occupancy me esta dando de shared memory porque me limita
//el numero de bloques que se ejecutan a la vez(solo se ejecutan a la vez en toda la function
// los bloques que "quepan" con la shared memory available: solution use cudastreams and launch instead
//of only 1 kernel use 2 or 4 to cubrir huecos (de memoria y eso), y tmb reducir la shared
//con una implementacion hibrida del dotxy

//todo add debug variables in some way (maybe pass always it pointer or something like that)
__global__
void solveBcgCuda(
        double *dA, int *djA, int *diA, double *dx, double *dtempv //Input data
        ,int nrows, int blocks, int n_shr_empty, int maxIt, int mattype
        ,int n_cells, double tolmax, double *ddiag //Init variables
        ,double *dr0, double *dr0h, double *dn0, double *dp0
        ,double *dt, double *ds, double *dAx2, double *dy, double *dz
        ,double *daux // Auxiliary vectors
#ifdef PMC_DEBUG_GPU
        ,int *it_pointer //debug
#endif
        //,double *aux_params
        //double *alpha, double *rho0, double* omega0, double *beta,
        //double *rho1, double *temp1, double *temp2 //Auxiliary parameters
)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int active_threads = nrows;

#ifdef BCG_ALL_THREADS
  if(1){
#else
  if(i<active_threads){
#endif

    double alpha,rho0,omega0,beta,rho1,temp1,temp2;
    alpha=rho0=omega0=beta=rho1=temp1=temp2=1.0;

    /*alpha  = 1.0;
    rho0   = 1.0;
    omega0 = 1.0;*/

    //gpu_yequalsconst(dn0,0.0,nrows,blocks,threads);  //n0=0.0 //memset???
    //gpu_yequalsconst(dp0,0.0,nrows,blocks,threads);  //p0=0.0
    /*cudaDevicesetconst(dn0, 0.0, nrows);
    cudaDevicesetconst(dp0, 0.0, nrows);
    cudaDevicesetconst(dt, 0.0, nrows);*/

    cudaDevicesetconst(dr0, 0.0, nrows);
    cudaDevicesetconst(dr0h, 0.0, nrows);
    cudaDevicesetconst(dn0, 0.0, nrows);
    cudaDevicesetconst(dp0, 0.0, nrows);
    cudaDevicesetconst(dt, 0.0, nrows);
    cudaDevicesetconst(ds, 0.0, nrows);
    cudaDevicesetconst(dAx2, 0.0, nrows);
    cudaDevicesetconst(dy, 0.0, nrows);
    cudaDevicesetconst(dz, 0.0, nrows);

#ifdef BASIC_SPMV
    cudaDevicesetconst(dr0, 0.0, nrows);
    __syncthreads();
    cudaDeviceSpmvCSC(dr0,dx,nrows,dA,djA,diA); //y=A*x
#else
    cudaDeviceSpmv(dr0,dx,nrows,dA,djA,diA,n_shr_empty); //y=A*x
#endif

#ifdef DEBUG_SOLVEBCGCUDA_DEEP

    //printf("%d ddiag %-le\n",i,ddiag[i]);
    //printf("%d dr0 %-le\n",i, dr0[i]);

#endif


    //gpu_axpby(dr0,dtempv,1.0,-1.0,nrows,blocks,threads); // r0=1.0*rhs+-1.0r0 //y=ax+by
    cudaDeviceaxpby(dr0,dtempv,1.0,-1.0,nrows);

    __syncthreads();
    //gpu_yequalsx(dr0h,dr0,nrows,blocks,threads);  //r0h=r0
    cudaDeviceyequalsx(dr0h,dr0,nrows);

#ifdef PMC_DEBUG_GPU
    //int it=*it_pointer;
    int it=0;
#else
    int it=0;
#endif

#ifdef DEBUG_SOLVEBCGCUDA_DEEP

    if(i==0){
      //printf("%d dr0[%d] %-le\n",it,i,dr0[i]);
      printf("%d %d rho1 %-le\n",it,i,rho1);
    }

    //dvcheck_input_gpud(dx,nrows,"dx");
    //dvcheck_input_gpud(dr0,nrows,"dr0");

#endif

    do
    {
      //rho1=gpu_dotxy(dr0, dr0h, aux, daux, nrows,(blocks + 1) / 2, threads);
      __syncthreads();

      cudaDevicedotxy(dr0, dr0h, &rho1, nrows, n_shr_empty);

#ifdef DEBUG_SOLVEBCGCUDA_DEEP

    if(i==0){
      //printf("%d dr0[%d] %-le\n",it,i,dr0[i]);
      printf("%d %d rho1 rho0 %-le %-le\n",it,i,rho1,rho0);
    }
    if(isnan(rho1) || rho1==0.0){
      dvcheck_input_gpud(dx,nrows,"dx");
      dvcheck_input_gpud(dr0h,nrows,"dr0h");
      dvcheck_input_gpud(dr0,nrows,"dr0");
    }

#endif

      __syncthreads();
      beta = (rho1 / rho0) * (alpha / omega0);

      __syncthreads();
      //gpu_zaxpbypc(dp0,dr0,dn0,beta,-1.0*omega0*beta,nrows,blocks,threads);   //z = ax + by + c
      cudaDevicezaxpbypc(dp0, dr0, dn0, beta, -1.0 * omega0 * beta, nrows);   //z = ax + by + c

      __syncthreads();
      //gpu_multxy(dy,ddiag,dp0,nrows,blocks,threads);  // precond y= p0*diag
      cudaDevicemultxy(dy, ddiag, dp0, nrows);

      __syncthreads();
      cudaDevicesetconst(dn0, 0.0, nrows);
      //gpu_spmv(dn0,dy,nrows,dA,djA,diA,mattype,blocks,threads);  // n0= A*y
#ifdef BASIC_SPMV
      cudaDevicesetconst(dn0, 0.0, nrows);
      __syncthreads();
      cudaDeviceSpmvCSC(dn0, dy, nrows, dA, djA, diA);
#else
      cudaDeviceSpmv(dn0, dy, nrows, dA, djA, diA,n_shr_empty);
#endif

#ifdef DEBUG_SOLVEBCGCUDA_DEEP

      if(it==0){
        printf("%d %d dy dn0 ddiag %-le %-le %le\n",it,i,dy[i],dn0[i],ddiag[i]);
        //printf("%d %d dn0 %-le\n",it,i,dn0[i]);
        //printf("%d %d &temp1 %p\n",it,i,&temp1);
        //printf("%d %d &test %p\n",it,i,&test);
        //printf("%d %d &i %p\n",it,i,&i);
      }

#endif

      //temp1=gpu_dotxy(dr0h, dn0, aux, daux, nrows,(blocks + 1) / 2, threads);
      cudaDevicedotxy(dr0h, dn0, &temp1, nrows, n_shr_empty);

#ifdef DEBUG_SOLVEBCGCUDA_DEEP

      if(i==0){
        printf("%d %d temp1 %-le\n",it,i,temp1);
        //printf("%d %d &temp1 %p\n",it,i,&temp1);
        //printf("%d %d &test %p\n",it,i,&test);
        //printf("%d %d &i %p\n",it,i,&i);
      }

#endif

      __syncthreads();
      alpha = rho1 / temp1;

      //gpu_zaxpby(1.0,dr0,-1.0*alpha,dn0,ds,nrows,blocks,threads); // a*x + b*y = z
      cudaDevicezaxpby(1.0, dr0, -1.0 * alpha, dn0, ds, nrows);

#ifdef DEBUG_SOLVEBCGCUDA_DEEP

      if(i==0){
        printf("%d ds[%d] %-le\n",it,i,ds[i]);
      }

#endif

      __syncthreads();
      //gpu_multxy(dz,ddiag,ds,nrows,blocks,threads); // precond z=diag*s
      cudaDevicemultxy(dz, ddiag, ds, nrows); // precond z=diag*s

      //gpu_spmv(dt,dz,nrows,dA,djA,diA,mattype,blocks,threads);
#ifdef BASIC_SPMV
      cudaDevicesetconst(dt, 0.0, nrows);
      __syncthreads();
      cudaDeviceSpmvCSC(dt, dz, nrows, dA, djA, diA);
#else
      cudaDeviceSpmv(dt, dz, nrows, dA, djA, diA,n_shr_empty);
#endif

      __syncthreads();///todo find why are needed
      //gpu_multxy(dAx2,ddiag,dt,nrows,blocks,threads);
      cudaDevicemultxy(dAx2, ddiag, dt, nrows);

      __syncthreads();
      //temp1=gpu_dotxy(dz, dAx2, aux, daux, nrows,(blocks + 1) / 2, threads);
      cudaDevicedotxy(dz, dAx2, &temp1, nrows, n_shr_empty);

#ifdef DEBUG_SOLVEBCGCUDA_DEEP

      if(i>=0){
        //printf("%d ddiag[%d] %-le\n",it,i,ddiag[i]);
        //printf("%d dt[%d] %-le\n",it,i,dt[i]);
        //printf("%d dAx2[%d] %-le\n",it,i,dAx2[i]);
        //printf("%d dz[%d] %-le\n",it,i,dz[i]);
      }

      if(i==0){
        printf("%d %d temp1 %-le\n",it,i,temp1);
      }

#endif

      __syncthreads();
      //temp2=gpu_dotxy(dAx2, dAx2, aux, daux, nrows,(blocks + 1) / 2, threads);
      cudaDevicedotxy(dAx2, dAx2, &temp2, nrows, n_shr_empty);

#ifdef DEBUG_SOLVEBCGCUDA_DEEP

      if(i==0){
        printf("%d %d temp2 %-le\n",it,i,temp2);
      }

#endif

      __syncthreads();
      omega0 = temp1 / temp2;
      //gpu_axpy(dx,dy,alpha,nrows,blocks,threads); // x=alpha*y +x
      cudaDeviceaxpy(dx, dy, alpha, nrows); // x=alpha*y +x

      __syncthreads();
      //gpu_axpy(dx,dz,omega0,nrows,blocks,threads);
      cudaDeviceaxpy(dx, dz, omega0, nrows);

      __syncthreads();
      //gpu_zaxpby(1.0,ds,-1.0*omega0,dt,dr0,nrows,blocks,threads);
      cudaDevicezaxpby(1.0, ds, -1.0 * omega0, dt, dr0, nrows);
      cudaDevicesetconst(dt, 0.0, nrows);

      __syncthreads();
      //temp1=gpu_dotxy(dr0, dr0, aux, daux, nrows,(blocks + 1) / 2, threads);
      cudaDevicedotxy(dr0, dr0, &temp1, nrows, n_shr_empty);

      //temp1 = sqrt(temp1);
      temp1 = sqrtf(temp1);

      rho0 = rho1;
  /**/
      __syncthreads();
  /**/

      //if (tid==0) it++;
      it++;
    } while(it<maxIt && temp1>tolmax);//while(it<maxIt && temp1>tolmax);//while(0);

#ifdef DEBUG_SOLVEBCGCUDA_DEEP
    if(tid==0)
      printf("%d %-le %-le\n",it, temp1, tolmax);
#endif

    //if(it>=maxIt-1)
    //  dvcheck_input_gpud(dr0,nrows,999);

    //dvcheck_input_gpud(dr0,nrows,k++);

    //todo itpointer should be an array of n_blocks size, and in cpu reduce to max number
    // (since the max its supposed to be the last to exit)
#ifdef PMC_DEBUG_GPU
   *it_pointer = it;
#endif

  }

/*
if (id == 0) //return aux variables if debugging
{
  aux_params[0]=alpha;
  aux_params[1]=rho0;
  aux_params[2]=omega0;
  aux_params[3]=beta;//0.01;
  aux_params[4]=rho1;//rho1
  aux_params[5]=temp1;
  aux_params[6]=temp2;
}
*/

}

//solveGPU_block: Each block will compute only a cell/group of cells
//Algorithm: Biconjugate gradient
void solveGPU_block(itsolver *bicg, double *dA, int *djA, int *diA, double *dx, double *dtempv)
{
  //Init variables ("public")
  int nrows = bicg->nrows;
  int threads = bicg->threads;
  int maxIt = bicg->maxIt;
  int mattype = bicg->mattype;
  int n_cells = bicg->n_cells;
  double tolmax = bicg->tolmax;
  double *ddiag = bicg->ddiag;

  // Auxiliary vectors ("private")
  double *dr0 = bicg->dr0;
  double *dr0h = bicg->dr0h;
  double *dn0 = bicg->dn0;
  double *dp0 = bicg->dp0;
  double *dt = bicg->dt;
  double *ds = bicg->ds;
  double *dAx2 = bicg->dAx2;
  double *dy = bicg->dy;
  double *dz = bicg->dz;
  double *daux = bicg->daux;

#ifndef DEBUG_SOLVEBCGCUDA
  if(bicg->counterBiConjGrad==0) {
    printf("solveGPUBlock\n");
  }
#endif

//todo eliminate atomicadd in spmv through using CSR or something like that
  //gpu_spmv(dr0,dx,nrows,dA,djA,diA,mattype,bicg->blocks,threads);  // r0= A*x

  /*
  gpu_axpby(dr0,dtempv,1.0,-1.0,nrows,blocks,threads); // r0=1.0*rhs+-1.0r0 //y=ax+by

  gpu_yequalsx(dr0h,dr0,nrows,blocks,threads);  //r0h=r0

  gpu_yequalsconst(dn0,0.0,nrows,blocks,threads);  //n0=0.0 //memset???
  gpu_yequalsconst(dp0,0.0,nrows,blocks,threads);  //p0=0.0

  alpha  = 1.0;
  rho0   = 1.0;
  omega0 = 1.0;
*/
  /*int n_aux_params=7;
  double *aux_params;
  aux_params=(double*)malloc(n_aux_params*sizeof(double));
  double *daux_params;
  hipMalloc(&daux_params,n_aux_params*sizeof(double));*/
  //hipMemcpy(bicg->djA,bicg->jA,7*sizeof(double),hipMemcpyHostToDevice);

  int size_cell = nrows/n_cells;

#ifndef INDEPENDENCY_CELLS

  //todo fix if max_threads_block > bicg->threads then what?
  int max_threads_block = nextPowerOfTwo(size_cell);//bicg->threads;
  //int n_shr_empty = max_threads-size_cell;//nextPowerOfTwo(size_cell)-size_cell;
  //int threads_block = max_threads_block - n_shr_empty; //last multiple of size_cell before max_threads

#else

  int max_threads_block = bicg->threads;//bicg->threads; 128;

  //int n_shr_empty = max_threads%size_cell; //Wrong
  //int n_shr_empty = max_threads-nrows;

#endif

#ifdef BCG_ALL_THREADS

  int threads_block = max_threads_block;
  int n_shr_empty = 0;
  int blocks = (nrows+threads_block-1)/threads_block;

#else
  int n_cells_block =  max_threads_block/size_cell;
  int threads_block = n_cells_block*size_cell;
  int n_shr_empty = max_threads_block-threads_block;
  int blocks = (nrows+threads_block-1)/threads_block;
#endif

#ifndef DEBUG_SOLVEBCGCUDA
  if(bicg->counterBiConjGrad==0) {
    printf("n_cells %d size_cell %d nrows %d nnz %d max_threads_block %d blocks %d threads_block %d n_shr_empty %d\n",
           n_cells,size_cell,nrows,bicg->nnz,max_threads_block,blocks,threads_block,n_shr_empty);
  }
#endif


  /*aux_params[0] = alpha;
  aux_params[1] = rho0;
  aux_params[2] = omega0;
  aux_params[3] = beta;
  aux_params[4] = rho1;
  aux_params[5] = temp1;
  aux_params[6] = temp2;
  hipMemcpy(daux_params, aux_params, n_aux_params * sizeof(double), hipMemcpyHostToDevice);*/

#ifdef PMC_DEBUG_GPU
  int it = 0;
  int *dit_ptr=bicg->counterBiConjGradInternalGPU;
  //int *dit_ptr;
  //hipMalloc((void**)&dit_ptr,sizeof(int));
  //hipMemcpy(dit_ptr, &it, sizeof(int), hipMemcpyHostToDevice);
#endif


  solveBcgCuda << < blocks, threads_block, max_threads_block * sizeof(double) >> >
  //solveBcgCuda << < blocks, threads_block, threads_block * sizeof(double) >> >
          (dA, djA, diA, dx, dtempv, nrows, blocks, n_shr_empty, maxIt, mattype, n_cells
          ,tolmax, ddiag, dr0, dr0h, dn0, dp0, dt, ds, dAx2, dy, dz, daux
#ifdef PMC_DEBUG_GPU
          ,dit_ptr
#endif
          //,daux_params
          );

#ifdef PMC_DEBUG_GPU
  hipMemcpy(&it,dit_ptr,sizeof(int),hipMemcpyDeviceToHost);
  bicg->counterBiConjGradInternal += it;

#ifndef DEBUG_SOLVEBCGCUDA
  if(bicg->counterBiConjGrad==0) {
    printf("counterBiConjGradInternal %d\n",
           bicg->counterBiConjGradInternal);
  }
#endif

#endif

  /*hipDeviceSynchronize();
  hipMemcpy(aux_params, daux_params, n_aux_params * sizeof(double), hipMemcpyDeviceToHost);

  alpha = aux_params[0];
  rho0 = aux_params[1];
  omega0 = aux_params[2];
  beta = aux_params[3];
  rho1 = aux_params[4];
  temp1 = aux_params[5];
  temp2 = aux_params[6];*/
  //printf("temp1 %-le", temp1);
  //printf("rho1 %f", rho1);

  //cudaFreeMem(daux_params);

}

//Algorithm: Biconjugate gradient
void solveGPU(itsolver *bicg, double *dA, int *djA, int *diA, double *dx, double *dtempv)
{
  //Init variables ("public")
  int nrows = bicg->nrows;
  int blocks = bicg->blocks;
  int threads = bicg->threads;
  int maxIt = bicg->maxIt;
  int mattype = bicg->mattype;
  double tolmax = bicg->tolmax;
  double *ddiag = bicg->ddiag;

  // Auxiliary vectors ("private")
  double *dr0 = bicg->dr0;
  double *dr0h = bicg->dr0h;
  double *dn0 = bicg->dn0;
  double *dp0 = bicg->dp0;
  double *dt = bicg->dt;
  double *ds = bicg->ds;
  double *dAx2 = bicg->dAx2;
  double *dy = bicg->dy;
  double *dz = bicg->dz;
  double *aux = bicg->aux;
  double *daux = bicg->daux;

#ifndef DEBUG_SOLVEBCGCUDA
  if(bicg->counterBiConjGrad==0) {
    printf("solveGPU\n");
  }
#endif

  //Function private variables
  double alpha,rho0,omega0,beta,rho1,temp1,temp2;

  gpu_spmv(dr0,dx,nrows,dA,djA,diA,mattype,blocks,threads);  // r0= A*x

  gpu_axpby(dr0,dtempv,1.0,-1.0,nrows,blocks,threads); // r0=1.0*rhs+-1.0r0 //y=ax+by

  gpu_yequalsx(dr0h,dr0,nrows,blocks,threads);  //r0h=r0

  gpu_yequalsconst(dn0,0.0,nrows,blocks,threads);  //n0=0.0 //memset???
  gpu_yequalsconst(dp0,0.0,nrows,blocks,threads);  //p0=0.0

  alpha  = 1.0;
  rho0   = 1.0;
  omega0 = 1.0;

  //printf("temp1 %-le", temp1);
  //printf("rho1 %f", rho1);

#ifdef DEBUG_SOLVEBCGCUDA_DEEP

  double *aux_x1;
  aux_x1=(double*)malloc(bicg->nrows*sizeof(double));

#endif

  //for(int it=0;it<maxIt;it++){
  int it=0;
  do {

    rho1=gpu_dotxy(dr0, dr0h, aux, daux, nrows,(blocks + 1) / 2, threads);//rho1 =<r0,r0h>
    //rho1=gpu_dotxy(dr0, dr0h, aux, daux, nrows,blocks, threads);//rho1 =<r0,r0h>

#ifdef DEBUG_SOLVEBCGCUDA_DEEP
    //good here first iter
    printf("%d rho1 %-le\n",it,rho1);
#endif

    beta=(rho1/rho0)*(alpha/omega0);

    //    cout<<"rho1 "<<rho1<<" beta "<<beta<<endl;

    gpu_zaxpbypc(dp0,dr0,dn0,beta,-1.0*omega0*beta,nrows,blocks,threads);   //z = ax + by + c

    gpu_multxy(dy,ddiag,dp0,nrows,blocks,threads);  // precond y= p0*diag

    gpu_spmv(dn0,dy,nrows,dA,djA,diA,mattype,blocks,threads);  // n0= A*y

    temp1=gpu_dotxy(dr0h, dn0, aux, daux, nrows,(blocks + 1) / 2, threads);
    //temp1=gpu_dotxy(dr0h, dn0, aux, daux, nrows, blocks, threads);

#ifdef DEBUG_SOLVEBCGCUDA_DEEP
    printf("%d temp1 %-le\n",it,temp1);
#endif

    alpha=rho1/temp1;

    //       cout<<"temp1 "<<temp1<<" alpha "<<alpha<<endl;

    gpu_zaxpby(1.0,dr0,-1.0*alpha,dn0,ds,nrows,blocks,threads);

#ifdef DEBUG_SOLVEBCGCUDA_DEEP
    hipMemcpy(aux_x1,ds,bicg->nrows*sizeof(double),hipMemcpyDeviceToHost);

    printf("%d ds[0] %-le\n",it,aux_x1[0]);

#endif

    gpu_multxy(dz,ddiag,ds,nrows,blocks,threads); // precond z=diag*s

    gpu_spmv(dt,dz,nrows,dA,djA,diA,mattype,blocks,threads);

    gpu_multxy(dAx2,ddiag,dt,nrows,blocks,threads);

    temp1=gpu_dotxy(dz, dAx2, aux, daux, nrows,(blocks + 1) / 2, threads);
    //temp1=gpu_dotxy(dz, dAx2, aux, daux, nrows,blocks, threads);

#ifdef DEBUG_SOLVEBCGCUDA_DEEP
    hipMemcpy(aux_x1,dAx2,bicg->nrows*sizeof(double),hipMemcpyDeviceToHost);

    for(int i=0; i<bicg->nrows; i++){
      //printf("%d ddiag[%i] %-le\n",it,i,aux_x1[i]);
      //printf("%d dt[%i] %-le\n",it,i,aux_x1[i]);
      //printf("%d dAx2[%i] %-le\n",it,i,aux_x1[i]);
      //printf("%d dz[%i] %-le\n",it,i,aux_x1[i]);
    }

    printf("%d temp1 %-le\n",it,temp1);
#endif

    temp2=gpu_dotxy(dAx2, dAx2, aux, daux, nrows,(blocks + 1) / 2, threads);
    //temp2=gpu_dotxy(dAx2, dAx2, aux, daux, nrows,blocks, threads);

#ifdef DEBUG_SOLVEBCGCUDA_DEEP
    printf("%d temp2 %-le\n",it,temp2);
#endif

    omega0= temp1/temp2;

    gpu_axpy(dx,dy,alpha,nrows,blocks,threads); // x=alpha*y +x

    gpu_axpy(dx,dz,omega0,nrows,blocks,threads);

    gpu_zaxpby(1.0,ds,-1.0*omega0,dt,dr0,nrows,blocks,threads);

    temp1=gpu_dotxy(dr0, dr0, aux, daux, nrows,(blocks + 1) / 2, threads);
    //temp1=gpu_dotxy(dr0, dr0, aux, daux, nrows,blocks, threads);
    temp1=sqrt(temp1);

  //cout<<it<<": "<<temp1<<endl;

    rho0=rho1;

    //if(temp1<tolmax){
    //  break;}}

    it++;
  }while(it<maxIt && temp1>tolmax);

#ifdef PMC_DEBUG_GPU
  bicg->counterBiConjGradInternal += it;
#endif

#ifdef DEBUG_SOLVEBCGCUDA_DEEP
  free(aux_x1);
#endif

}

void free_itsolver(itsolver *bicg)
{
  //Auxiliary vectors ("private")
  double ** dr0 = &bicg->dr0;
  double ** dr0h = &bicg->dr0h;
  double ** dn0 = &bicg->dn0;
  double ** dp0 = &bicg->dp0;
  double ** dt = &bicg->dt;
  double ** ds = &bicg->ds;
  double ** dAx2 = &bicg->dAx2;
  double ** dy = &bicg->dy;
  double ** dz = &bicg->dz;
  double ** daux = &bicg->daux;
  double ** ddiag = &bicg->ddiag;

  hipFree(dr0);
  hipFree(dr0h);
  hipFree(dn0);
  hipFree(dp0);
  hipFree(dt);
  hipFree(ds);
  hipFree(dAx2);
  hipFree(dy);
  hipFree(dz);
  hipFree(ddiag);
  hipFree(daux);
  free(bicg->aux);

}

 /*
void setUpSolver(itsolver *bicg, double reltol, double *ewt, int tnrows,int tnnz,double *tA, int *tjA, int *tiA, int tmattype, int qmax, double *dACamp, double *dftempCamp);
{

  bicg.tolmax=reltol;

}
*/
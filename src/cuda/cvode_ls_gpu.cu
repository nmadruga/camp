#include "hip/hip_runtime.h"
/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
 * Illinois at Urbana-Champaign
 * SPDX-License-Identifier: MIT
 */

#include "itsolver_gpu.h"

extern "C" {
#include "cvode_ls_gpu.h"
}

#ifdef CAMP_USE_MPI
#include <mpi.h>
#endif

#define CV_SUCCESS               0

#define DO_ERROR_TEST    +2
#define PREDICT_AGAIN    +3
#define CONV_FAIL        +4
#define TRY_AGAIN        +5
#define FIRST_CALL       +6
#define PREV_CONV_FAIL   +7
#define PREV_ERR_FAIL    +8
#define RHSFUNC_RECVR    +9

#define NUM_TESTS    5     /* number of error test quantities     */

/*=================================================================*/
/*             CVODE Private Constants                             */
/*=================================================================*/

#define PT1     RCONST(0.1)     /* real 0.1     */
#define POINT2  RCONST(0.2)     /* real 0.2     */
#define FOURTH  RCONST(0.25)    /* real 0.25    */
#define TWO     RCONST(2.0)     /* real 2.0     */
#define THREE   RCONST(3.0)     /* real 3.0     */
#define FOUR    RCONST(4.0)     /* real 4.0     */
#define FIVE    RCONST(5.0)     /* real 5.0     */
#define TWELVE  RCONST(12.0)    /* real 12.0    */
#define HUNDRED RCONST(100.0)   /* real 100.0   */
#define CAMP_TINY RCONST(1.0e-30) /* small number for CAMP */

#define DO_ERROR_TEST    +2
#define PREDICT_AGAIN    +3

#define CONV_FAIL        +4
#define TRY_AGAIN        +5

#define FIRST_CALL       +6
#define PREV_CONV_FAIL   +7
#define PREV_ERR_FAIL    +8

#define RHSFUNC_RECVR    +9

#define RTFOUND          +1
#define CLOSERT          +3

#define CV_NN  0
#define CV_SS  1
#define CV_SV  2
#define CV_WF  3

#define FUZZ_FACTOR RCONST(100.0)

#define HLB_FACTOR RCONST(100.0)
#define HUB_FACTOR RCONST(0.1)
#define H_BIAS     HALF
#define MAX_ITERS  4000

#define CORTES RCONST(0.1)

#define THRESH RCONST(1.5)
#define ETAMX1 RCONST(10000.0)
#define ETAMX2 RCONST(10.0)
#define ETAMX3 RCONST(10.0)
#define ETAMXF RCONST(0.2)
#define ETAMIN RCONST(0.1)
#define ETACF  RCONST(0.25)
#define ADDON  RCONST(0.000001)
#define BIAS1  RCONST(6.0)
#define BIAS2  RCONST(6.0)
#define BIAS3  RCONST(10.0)
#define ONEPSM RCONST(1.000001)

#define SMALL_NST    10
#define MXNCF        10
#define MXNEF         7
#define MXNEF1        3
#define SMALL_NEF     2
#define LONG_WAIT    10

#define NLS_MAXCOR 3
#define CRDOWN RCONST(0.3)
#define DGMAX  RCONST(0.3)

#define RDIV      TWO
#define MSBP       20

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

static void HandleError(hipError_t err,
                        const char *file,
                        int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err),
           file, line);
    exit(EXIT_FAILURE);
  }
}

int cvHandleFailure_gpu(CVodeMem cv_mem, int flag)
{

  /* Set vector of  absolute weighted local errors */
  /*
  N_VProd(acor, ewt, tempv);
  N_VAbs(tempv, tempv);
  */

  //printf("cvHandleFailure_gpu start\n");

  /* Depending on flag, print error message and return error flag */
  switch (flag) {
    case CV_ERR_FAILURE:
      cvProcessError(cv_mem, CV_ERR_FAILURE, "CVODE", "CVode", MSGCV_ERR_FAILS,
                     cv_mem->cv_tn, cv_mem->cv_h);
      break;
    case CV_CONV_FAILURE:
      cvProcessError(cv_mem, CV_CONV_FAILURE, "CVODE", "CVode", MSGCV_CONV_FAILS,
                     cv_mem->cv_tn, cv_mem->cv_h);
      break;
    case CV_LSETUP_FAIL:
      cvProcessError(cv_mem, CV_LSETUP_FAIL, "CVODE", "CVode", MSGCV_SETUP_FAILED,
                     cv_mem->cv_tn);
      break;
    case CV_LSOLVE_FAIL:
      cvProcessError(cv_mem, CV_LSOLVE_FAIL, "CVODE", "CVode", MSGCV_SOLVE_FAILED,
                     cv_mem->cv_tn);
      break;
    case CV_RHSFUNC_FAIL:
      cvProcessError(cv_mem, CV_RHSFUNC_FAIL, "CVODE", "CVode", MSGCV_RHSFUNC_FAILED,
                     cv_mem->cv_tn);
      break;
    case CV_UNREC_RHSFUNC_ERR:
      cvProcessError(cv_mem, CV_UNREC_RHSFUNC_ERR, "CVODE", "CVode", MSGCV_RHSFUNC_UNREC,
                     cv_mem->cv_tn);
      break;
    case CV_REPTD_RHSFUNC_ERR:
      cvProcessError(cv_mem, CV_REPTD_RHSFUNC_ERR, "CVODE", "CVode", MSGCV_RHSFUNC_REPTD,
                     cv_mem->cv_tn);
      break;
    case CV_RTFUNC_FAIL:
      cvProcessError(cv_mem, CV_RTFUNC_FAIL, "CVODE", "CVode", MSGCV_RTFUNC_FAILED,
                     cv_mem->cv_tn);
      break;
    case CV_TOO_CLOSE:
      cvProcessError(cv_mem, CV_TOO_CLOSE, "CVODE", "CVode", MSGCV_TOO_CLOSE);
      break;
    default:
      return(CV_SUCCESS);
  }

  return(flag);
}


int cvInitialSetup_gpu(CVodeMem cv_mem){
  int ier;
  if (cv_mem->cv_itol == CV_NN) {
    cvProcessError(cv_mem, CV_ILL_INPUT, "CVODE", "cvInitialSetup", MSGCV_NO_TOLS);
    return(CV_ILL_INPUT);
  }
  if (cv_mem->cv_user_efun) cv_mem->cv_e_data = cv_mem->cv_user_data;
  else                      cv_mem->cv_e_data = cv_mem;
  ier = cv_mem->cv_efun(cv_mem->cv_zn[0], cv_mem->cv_ewt, cv_mem->cv_e_data);
  if (ier != 0) {
    if (cv_mem->cv_itol == CV_WF)
      cvProcessError(cv_mem, CV_ILL_INPUT, "CVODE", "cvInitialSetup", MSGCV_EWT_FAIL);
    else
      cvProcessError(cv_mem, CV_ILL_INPUT, "CVODE", "cvInitialSetup", MSGCV_BAD_EWT);
    return(CV_ILL_INPUT);
  }
  if (cv_mem->cv_iter == CV_NEWTON) {
    if (cv_mem->cv_lsolve == NULL) {
      cvProcessError(cv_mem, CV_ILL_INPUT, "CVODE", "cvInitialSetup", MSGCV_LSOLVE_NULL);
      return(CV_ILL_INPUT);
    }
    if (cv_mem->cv_linit != NULL) {
      ier = cv_mem->cv_linit(cv_mem);
      if (ier != 0) {
        cvProcessError(cv_mem, CV_LINIT_FAIL, "CVODE", "cvInitialSetup", MSGCV_LINIT_FAIL);
        return(CV_LINIT_FAIL);
      }
    }
  }
  return(CV_SUCCESS);
}

int cvHin_gpu(CVodeMem cv_mem, realtype tout)
{
  int retval, sign, count1, count2;
  realtype tdiff, tdist, tround, hlb, hub;
  realtype hg, hgs, hs, hnew, hrat, h0, yddnrm;
  booleantype hgOK, hnewOK;

  /* If tout is too close to tn, give up */

  if ((tdiff = tout-cv_mem->cv_tn) == ZERO) return(CV_TOO_CLOSE);

  sign = (tdiff > ZERO) ? 1 : -1;
  tdist = SUNRabs(tdiff);
  tround = cv_mem->cv_uround * SUNMAX(SUNRabs(cv_mem->cv_tn), SUNRabs(tout));

  if (tdist < TWO*tround) return(CV_TOO_CLOSE);

  /*
     Set lower and upper bounds on h0, and take geometric mean
     as first trial value.
     Exit with this value if the bounds cross each other.
  */

  hlb = HLB_FACTOR * tround;
  hub = cvUpperBoundH0_gpu(cv_mem, tdist);

  hg  = SUNRsqrt(hlb*hub);

  if (hub < hlb) {
    if (sign == -1) cv_mem->cv_h = -hg;
    else            cv_mem->cv_h =  hg;
    return(CV_SUCCESS);
  }

  /* Outer loop */

  hnewOK = SUNFALSE;
  hs = hg;         /* safeguard against 'uninitialized variable' warning */

  for(count1 = 1; count1 <= MAX_ITERS; count1++) {

    /* Attempts to estimate ydd */

    hgOK = SUNFALSE;

    for (count2 = 1; count2 <= MAX_ITERS; count2++) {
      hgs = hg*sign;
      retval = cvYddNorm_gpu(cv_mem, hgs, &yddnrm);
      /* If f() failed unrecoverably, give up */
      if (retval < 0) return(CV_RHSFUNC_FAIL);
      /* If successful, we can use ydd */
      if (retval == CV_SUCCESS) {hgOK = SUNTRUE; break;}
      /* f() failed recoverably; cut step size and test it again */
      hg *= POINT2;
    }

    /* If f() failed recoverably MAX_ITERS times */

    if (!hgOK) {
      /* Exit if this is the first or second pass. No recovery possible */
      if (count1 <= 2) return(CV_REPTD_RHSFUNC_ERR);
      /* We have a fall-back option. The value hs is a previous hnew which
         passed through f(). Use it and break */
      hnew = hs;
      break;
    }

    /* The proposed step size is feasible. Save it. */
    hs = hg;

    /* If the stopping criteria was met, or if this is the last pass, stop */
    if ( (hnewOK) || (count1 == MAX_ITERS))  {hnew = hg; break;}

    /* Propose new step size */
    hnew = (yddnrm*hub*hub > TWO) ? SUNRsqrt(TWO/yddnrm) : SUNRsqrt(hg*hub);
    hrat = hnew/hg;

    /* Accept hnew if it does not differ from hg by more than a factor of 2 */
    if ((hrat > HALF) && (hrat < TWO)) {
      hnewOK = SUNTRUE;
    }

    /* After one pass, if ydd seems to be bad, use fall-back value. */
    if ((count1 > 1) && (hrat > TWO)) {
      hnew = hg;
      hnewOK = SUNTRUE;
    }

    /* Send this value back through f() */
    hg = hnew;

  }

  /* Apply bounds, bias factor, and attach sign */

  h0 = H_BIAS*hnew;
  if (h0 < hlb) h0 = hlb;
  if (h0 > hub) h0 = hub;
  if (sign == -1) h0 = -h0;
  cv_mem->cv_h = h0;

  return(CV_SUCCESS);
}

realtype cvUpperBoundH0_gpu(CVodeMem cv_mem, realtype tdist)
{
  realtype hub_inv, hub;
  N_Vector temp1, temp2;

  /*
   * Bound based on |y0|/|y0'| -- allow at most an increase of
   * HUB_FACTOR in y0 (based on a forward Euler step). The weight
   * factor is used as a safeguard against zero components in y0.
   */

  temp1 = cv_mem->cv_tempv;
  temp2 = cv_mem->cv_acor;

  N_VAbs(cv_mem->cv_zn[0], temp2);
  cv_mem->cv_efun(cv_mem->cv_zn[0], temp1, cv_mem->cv_e_data);
  N_VInv(temp1, temp1);
  N_VLinearSum(HUB_FACTOR, temp2, ONE, temp1, temp1);

  N_VAbs(cv_mem->cv_zn[1], temp2);

  N_VDiv(temp2, temp1, temp1);
  hub_inv = N_VMaxNorm(temp1);

  /*
   * bound based on tdist -- allow at most a step of magnitude
   * HUB_FACTOR * tdist
   */

  hub = HUB_FACTOR*tdist;

  /* Use the smaller of the two */

  if (hub*hub_inv > ONE) hub = ONE/hub_inv;

  return(hub);
}

int cvYddNorm_gpu(CVodeMem cv_mem, realtype hg, realtype *yddnrm)
{
  int retval;

  N_VLinearSum(hg, cv_mem->cv_zn[1], ONE, cv_mem->cv_zn[0], cv_mem->cv_y);
  //retval = cv_mem->cv_f(cv_mem->cv_tn+hg, cv_mem->cv_y,
  //                      cv_mem->cv_tempv, cv_mem->cv_user_data);
  retval = f(cv_mem->cv_tn+hg, cv_mem->cv_y, cv_mem->cv_tempv, cv_mem->cv_user_data);
  cv_mem->cv_nfe++;
  if (retval < 0) return(CV_RHSFUNC_FAIL);
  if (retval > 0) return(RHSFUNC_RECVR);

  N_VLinearSum(ONE, cv_mem->cv_tempv, -ONE, cv_mem->cv_zn[1], cv_mem->cv_tempv);
  N_VScale(ONE/hg, cv_mem->cv_tempv, cv_mem->cv_tempv);

  *yddnrm = N_VWrmsNorm(cv_mem->cv_tempv, cv_mem->cv_ewt);

  return(CV_SUCCESS);
}

int cvRcheck1_gpu(CVodeMem cv_mem)
{
  int i, retval;
  realtype smallh, hratio, tplus;
  booleantype zroot;

  for (i = 0; i < cv_mem->cv_nrtfn; i++) cv_mem->cv_iroots[i] = 0;
  cv_mem->cv_tlo = cv_mem->cv_tn;
  cv_mem->cv_ttol = (SUNRabs(cv_mem->cv_tn) + SUNRabs(cv_mem->cv_h)) *
                    cv_mem->cv_uround*HUNDRED;

  /* Evaluate g at initial t and check for zero values. */
  retval = cv_mem->cv_gfun(cv_mem->cv_tlo, cv_mem->cv_zn[0],
                           cv_mem->cv_glo, cv_mem->cv_user_data);
  cv_mem->cv_nge = 1;
  if (retval != 0) return(CV_RTFUNC_FAIL);

  zroot = SUNFALSE;
  for (i = 0; i < cv_mem->cv_nrtfn; i++) {
    if (SUNRabs(cv_mem->cv_glo[i]) == ZERO) {
      zroot = SUNTRUE;
      cv_mem->cv_gactive[i] = SUNFALSE;
    }
  }
  if (!zroot) return(CV_SUCCESS);

  /* Some g_i is zero at t0; look at g at t0+(small increment). */
  hratio = SUNMAX(cv_mem->cv_ttol/SUNRabs(cv_mem->cv_h), PT1);
  smallh = hratio*cv_mem->cv_h;
  tplus = cv_mem->cv_tlo + smallh;
  N_VLinearSum(ONE, cv_mem->cv_zn[0], hratio,
               cv_mem->cv_zn[1], cv_mem->cv_y);
  retval = cv_mem->cv_gfun(tplus, cv_mem->cv_y,
                           cv_mem->cv_ghi, cv_mem->cv_user_data);
  cv_mem->cv_nge++;
  if (retval != 0) return(CV_RTFUNC_FAIL);

  /* We check now only the components of g which were exactly 0.0 at t0
   * to see if we can 'activate' them. */
  for (i = 0; i < cv_mem->cv_nrtfn; i++) {
    if (!cv_mem->cv_gactive[i] && SUNRabs(cv_mem->cv_ghi[i]) != ZERO) {
      cv_mem->cv_gactive[i] = SUNTRUE;
      cv_mem->cv_glo[i] = cv_mem->cv_ghi[i];
    }
  }
  return(CV_SUCCESS);
}

int cvRcheck2_gpu(CVodeMem cv_mem)
{
  int i, retval;
  realtype smallh, hratio, tplus;
  booleantype zroot;

  if (cv_mem->cv_irfnd == 0) return(CV_SUCCESS);

  (void) CVodeGetDky(cv_mem, cv_mem->cv_tlo, 0, cv_mem->cv_y);
  retval = cv_mem->cv_gfun(cv_mem->cv_tlo, cv_mem->cv_y,
                           cv_mem->cv_glo, cv_mem->cv_user_data);
  cv_mem->cv_nge++;
  if (retval != 0) return(CV_RTFUNC_FAIL);

  zroot = SUNFALSE;
  for (i = 0; i < cv_mem->cv_nrtfn; i++) cv_mem->cv_iroots[i] = 0;
  for (i = 0; i < cv_mem->cv_nrtfn; i++) {
    if (!cv_mem->cv_gactive[i]) continue;
    if (SUNRabs(cv_mem->cv_glo[i]) == ZERO) {
      zroot = SUNTRUE;
      cv_mem->cv_iroots[i] = 1;
    }
  }
  if (!zroot) return(CV_SUCCESS);

  /* One or more g_i has a zero at tlo.  Check g at tlo+smallh. */
  cv_mem->cv_ttol = (SUNRabs(cv_mem->cv_tn) + SUNRabs(cv_mem->cv_h)) *
                    cv_mem->cv_uround * HUNDRED;
  smallh = (cv_mem->cv_h > ZERO) ? cv_mem->cv_ttol : -cv_mem->cv_ttol;
  tplus = cv_mem->cv_tlo + smallh;
  if ( (tplus - cv_mem->cv_tn)*cv_mem->cv_h >= ZERO) {
    hratio = smallh/cv_mem->cv_h;
    N_VLinearSum(ONE, cv_mem->cv_y, hratio, cv_mem->cv_zn[1], cv_mem->cv_y);
  } else {
    (void) CVodeGetDky(cv_mem, tplus, 0, cv_mem->cv_y);
  }
  retval = cv_mem->cv_gfun(tplus, cv_mem->cv_y,
                           cv_mem->cv_ghi, cv_mem->cv_user_data);
  cv_mem->cv_nge++;
  if (retval != 0) return(CV_RTFUNC_FAIL);

  /* Check for close roots (error return), for a new zero at tlo+smallh,
  and for a g_i that changed from zero to nonzero. */
  zroot = SUNFALSE;
  for (i = 0; i < cv_mem->cv_nrtfn; i++) {
    if (!cv_mem->cv_gactive[i]) continue;
    if (SUNRabs(cv_mem->cv_ghi[i]) == ZERO) {
      if (cv_mem->cv_iroots[i] == 1) return(CLOSERT);
      zroot = SUNTRUE;
      cv_mem->cv_iroots[i] = 1;
    } else {
      if (cv_mem->cv_iroots[i] == 1)
        cv_mem->cv_glo[i] = cv_mem->cv_ghi[i];
    }
  }
  if (zroot) return(RTFOUND);
  return(CV_SUCCESS);
}

int cvRcheck3_gpu(CVodeMem cv_mem)
{
  int i, ier, retval;

  /* Set thi = tn or tout, whichever comes first; set y = y(thi). */
  if (cv_mem->cv_taskc == CV_ONE_STEP) {
    cv_mem->cv_thi = cv_mem->cv_tn;
    N_VScale(ONE, cv_mem->cv_zn[0], cv_mem->cv_y);
  }
  if (cv_mem->cv_taskc == CV_NORMAL) {
    if ( (cv_mem->cv_toutc - cv_mem->cv_tn)*cv_mem->cv_h >= ZERO) {
      cv_mem->cv_thi = cv_mem->cv_tn;
      N_VScale(ONE, cv_mem->cv_zn[0], cv_mem->cv_y);
    } else {
      cv_mem->cv_thi = cv_mem->cv_toutc;
      (void) CVodeGetDky(cv_mem, cv_mem->cv_thi, 0, cv_mem->cv_y);
    }
  }

  /* Set ghi = g(thi) and call cvRootfind to search (tlo,thi) for roots. */
  retval = cv_mem->cv_gfun(cv_mem->cv_thi, cv_mem->cv_y,
                           cv_mem->cv_ghi, cv_mem->cv_user_data);
  cv_mem->cv_nge++;
  if (retval != 0) return(CV_RTFUNC_FAIL);

  cv_mem->cv_ttol = (SUNRabs(cv_mem->cv_tn) + SUNRabs(cv_mem->cv_h)) *
                    cv_mem->cv_uround * HUNDRED;
  ier = cvRootfind_gpu(cv_mem);
  if (ier == CV_RTFUNC_FAIL) return(CV_RTFUNC_FAIL);
  for(i=0; i<cv_mem->cv_nrtfn; i++) {
    if(!cv_mem->cv_gactive[i] && cv_mem->cv_grout[i] != ZERO)
      cv_mem->cv_gactive[i] = SUNTRUE;
  }
  cv_mem->cv_tlo = cv_mem->cv_trout;
  for (i = 0; i < cv_mem->cv_nrtfn; i++)
    cv_mem->cv_glo[i] = cv_mem->cv_grout[i];

  /* If no root found, return CV_SUCCESS. */
  if (ier == CV_SUCCESS) return(CV_SUCCESS);

  /* If a root was found, interpolate to get y(trout) and return.  */
  (void) CVodeGetDky(cv_mem, cv_mem->cv_trout, 0, cv_mem->cv_y);
  return(RTFOUND);
}

int cvRootfind_gpu(CVodeMem cv_mem)
{
  realtype alph, tmid, gfrac, maxfrac, fracint, fracsub;
  int i, retval, imax, side, sideprev;
  booleantype zroot, sgnchg;

  imax = 0;

  /* First check for change in sign in ghi or for a zero in ghi. */
  maxfrac = ZERO;
  zroot = SUNFALSE;
  sgnchg = SUNFALSE;
  for (i = 0;  i < cv_mem->cv_nrtfn; i++) {
    if(!cv_mem->cv_gactive[i]) continue;
    if (SUNRabs(cv_mem->cv_ghi[i]) == ZERO) {
      if(cv_mem->cv_rootdir[i]*cv_mem->cv_glo[i] <= ZERO) {
        zroot = SUNTRUE;
      }
    } else {
      if ( (cv_mem->cv_glo[i]*cv_mem->cv_ghi[i] < ZERO) &&
           (cv_mem->cv_rootdir[i]*cv_mem->cv_glo[i] <= ZERO) ) {
        gfrac = SUNRabs(cv_mem->cv_ghi[i]/(cv_mem->cv_ghi[i] - cv_mem->cv_glo[i]));
        if (gfrac > maxfrac) {
          sgnchg = SUNTRUE;
          maxfrac = gfrac;
          imax = i;
        }
      }
    }
  }

  /* If no sign change was found, reset trout and grout.  Then return
     CV_SUCCESS if no zero was found, or set iroots and return RTFOUND.  */
  if (!sgnchg) {
    cv_mem->cv_trout = cv_mem->cv_thi;
    for (i = 0; i < cv_mem->cv_nrtfn; i++) cv_mem->cv_grout[i] = cv_mem->cv_ghi[i];
    if (!zroot) return(CV_SUCCESS);
    for (i = 0; i < cv_mem->cv_nrtfn; i++) {
      cv_mem->cv_iroots[i] = 0;
      if(!cv_mem->cv_gactive[i]) continue;
      if ( (SUNRabs(cv_mem->cv_ghi[i]) == ZERO) &&
           (cv_mem->cv_rootdir[i]*cv_mem->cv_glo[i] <= ZERO) )
        cv_mem->cv_iroots[i] = cv_mem->cv_glo[i] > 0 ? -1 : 1;
    }
    return(RTFOUND);
  }

  /* Initialize alph to avoid compiler warning */
  alph = ONE;

  /* A sign change was found.  Loop to locate nearest root. */

  side = 0;  sideprev = -1;
  for(;;) {                                    /* Looping point */

    /* If interval size is already less than tolerance ttol, break. */
    if (SUNRabs(cv_mem->cv_thi - cv_mem->cv_tlo) <= cv_mem->cv_ttol) break;

    /* Set weight alph.
       On the first two passes, set alph = 1.  Thereafter, reset alph
       according to the side (low vs high) of the subinterval in which
       the sign change was found in the previous two passes.
       If the sides were opposite, set alph = 1.
       If the sides were the same, then double alph (if high side),
       or halve alph (if low side).
       The next guess tmid is the secant method value if alph = 1, but
       is closer to tlo if alph < 1, and closer to thi if alph > 1.    */

    if (sideprev == side) {
      alph = (side == 2) ? alph*TWO : alph*HALF;
    } else {
      alph = ONE;
    }

    /* Set next root approximation tmid and get g(tmid).
       If tmid is too close to tlo or thi, adjust it inward,
       by a fractional distance that is between 0.1 and 0.5.  */
    tmid = cv_mem->cv_thi - (cv_mem->cv_thi - cv_mem->cv_tlo) *
                            cv_mem->cv_ghi[imax] / (cv_mem->cv_ghi[imax] - alph*cv_mem->cv_glo[imax]);
    if (SUNRabs(tmid - cv_mem->cv_tlo) < HALF*cv_mem->cv_ttol) {
      fracint = SUNRabs(cv_mem->cv_thi - cv_mem->cv_tlo)/cv_mem->cv_ttol;
      fracsub = (fracint > FIVE) ? PT1 : HALF/fracint;
      tmid = cv_mem->cv_tlo + fracsub*(cv_mem->cv_thi - cv_mem->cv_tlo);
    }
    if (SUNRabs(cv_mem->cv_thi - tmid) < HALF*cv_mem->cv_ttol) {
      fracint = SUNRabs(cv_mem->cv_thi - cv_mem->cv_tlo)/cv_mem->cv_ttol;
      fracsub = (fracint > FIVE) ? PT1 : HALF/fracint;
      tmid = cv_mem->cv_thi - fracsub*(cv_mem->cv_thi - cv_mem->cv_tlo);
    }

    (void) CVodeGetDky(cv_mem, tmid, 0, cv_mem->cv_y);
    retval = cv_mem->cv_gfun(tmid, cv_mem->cv_y, cv_mem->cv_grout,
                             cv_mem->cv_user_data);
    cv_mem->cv_nge++;
    if (retval != 0) return(CV_RTFUNC_FAIL);

    /* Check to see in which subinterval g changes sign, and reset imax.
       Set side = 1 if sign change is on low side, or 2 if on high side.  */
    maxfrac = ZERO;
    zroot = SUNFALSE;
    sgnchg = SUNFALSE;
    sideprev = side;
    for (i = 0;  i < cv_mem->cv_nrtfn; i++) {
      if(!cv_mem->cv_gactive[i]) continue;
      if (SUNRabs(cv_mem->cv_grout[i]) == ZERO) {
        if(cv_mem->cv_rootdir[i]*cv_mem->cv_glo[i] <= ZERO) zroot = SUNTRUE;
      } else {
        if ( (cv_mem->cv_glo[i]*cv_mem->cv_grout[i] < ZERO) &&
             (cv_mem->cv_rootdir[i]*cv_mem->cv_glo[i] <= ZERO) ) {
          gfrac = SUNRabs(cv_mem->cv_grout[i]/(cv_mem->cv_grout[i] - cv_mem->cv_glo[i]));
          if (gfrac > maxfrac) {
            sgnchg = SUNTRUE;
            maxfrac = gfrac;
            imax = i;
          }
        }
      }
    }
    if (sgnchg) {
      /* Sign change found in (tlo,tmid); replace thi with tmid. */
      cv_mem->cv_thi = tmid;
      for (i = 0; i < cv_mem->cv_nrtfn; i++)
        cv_mem->cv_ghi[i] = cv_mem->cv_grout[i];
      side = 1;
      /* Stop at root thi if converged; otherwise loop. */
      if (SUNRabs(cv_mem->cv_thi - cv_mem->cv_tlo) <= cv_mem->cv_ttol) break;
      continue;  /* Return to looping point. */
    }

    if (zroot) {
      /* No sign change in (tlo,tmid), but g = 0 at tmid; return root tmid. */
      cv_mem->cv_thi = tmid;
      for (i = 0; i < cv_mem->cv_nrtfn; i++)
        cv_mem->cv_ghi[i] = cv_mem->cv_grout[i];
      break;
    }

    /* No sign change in (tlo,tmid), and no zero at tmid.
       Sign change must be in (tmid,thi).  Replace tlo with tmid. */
    cv_mem->cv_tlo = tmid;
    for (i = 0; i < cv_mem->cv_nrtfn; i++)
      cv_mem->cv_glo[i] = cv_mem->cv_grout[i];
    side = 2;
    /* Stop at root thi if converged; otherwise loop back. */
    if (SUNRabs(cv_mem->cv_thi - cv_mem->cv_tlo) <= cv_mem->cv_ttol) break;

  } /* End of root-search loop */

  /* Reset trout and grout, set iroots, and return RTFOUND. */
  cv_mem->cv_trout = cv_mem->cv_thi;
  for (i = 0; i < cv_mem->cv_nrtfn; i++) {
    cv_mem->cv_grout[i] = cv_mem->cv_ghi[i];
    cv_mem->cv_iroots[i] = 0;
    if(!cv_mem->cv_gactive[i]) continue;
    if ( (SUNRabs(cv_mem->cv_ghi[i]) == ZERO) &&
         (cv_mem->cv_rootdir[i]*cv_mem->cv_glo[i] <= ZERO) )
      cv_mem->cv_iroots[i] = cv_mem->cv_glo[i] > 0 ? -1 : 1;
    if ( (cv_mem->cv_glo[i]*cv_mem->cv_ghi[i] < ZERO) &&
         (cv_mem->cv_rootdir[i]*cv_mem->cv_glo[i] <= ZERO) )
      cv_mem->cv_iroots[i] = cv_mem->cv_glo[i] > 0 ? -1 : 1;
  }
  return(RTFOUND);
}
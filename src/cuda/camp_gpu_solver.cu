#include "hip/hip_runtime.h"
/* Copyright (C) 2019 Christian Guzman
 * Licensed under the GNU General Public License version 1 or (at your
 * option) any later version. See the file COPYING for details.
 *
 * Interface Host-Device (CPU-GPU) to compute reaction-specific functions on GPU
 *
 */

#include "itsolver_gpu.h"

extern "C" {
#include "camp_gpu_solver.h"
#include "rxns_gpu.h"
#include "aeros/aero_rep_gpu_solver.h"
#include "time_derivative_gpu.h"

// Reaction types (Must match parameters defined in pmc_rxn_factory)
#define RXN_ARRHENIUS 1
#define RXN_TROE 2
#define RXN_CMAQ_H2O2 3
#define RXN_CMAQ_OH_HNO3 4
#define RXN_PHOTOLYSIS 5
#define RXN_HL_PHASE_TRANSFER 6
#define RXN_AQUEOUS_EQUILIBRIUM 7
#define RXN_SIMPOL_PHASE_TRANSFER 10
#define RXN_CONDENSED_PHASE_ARRHENIUS 11
#define RXN_FIRST_ORDER_LOSS 12
#define RXN_EMISSION 13
#define RXN_WET_DEPOSITION 14

#define STREAM_RXN_ENV_GPU 0
#define STREAM_ENV_GPU 1
#define STREAM_DERIV_GPU 2

// Status codes for calls to camp_solver functions
#define CAMP_SOLVER_SUCCESS 0
#define CAMP_SOLVER_FAIL 1

//GPU async stream related variables to ensure robustness
//int n_solver_objects=0; //Number of solver_new_gpu calls
//hipStream_t *stream_gpu; //GPU streams to async computation/data movement
//int n_streams = 16;

//Gpu hardware info
//int md->max_n_gpu_thread;
//int md->max_n_gpu_blocks;

static void HandleError(hipError_t err,
                        const char *file,
                        int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err),
           file, line);
    exit(EXIT_FAILURE);
  }
}

/** \brief Allocate GPU solver variables
 *
 * \param n_dep_var number of solver variables per grid cell
 * \param n_state_var Number of variables on the state array per grid cell
 * \param n_rxn Number of reactions to include
 * \param n_rxn_int_param Total number of integer reaction parameters
 * \param n_rxn_float_param Total number of floating-point reaction parameters
 * \param n_cells Number of grid cells to solve simultaneously
 */
void solver_new_gpu_cu(SolverData *sd, int n_dep_var,
                       int n_state_var, int n_rxn,
                       int n_rxn_int_param, int n_rxn_float_param, int n_rxn_env_param,
                       int n_cells) {
  //TODO: Select what % of data we want to compute on GPU simultaneously with CPU remaining %
  //Lengths
  ModelData *md = &(sd->model_data);
  md->state_size = n_state_var * n_cells * sizeof(double);
  md->deriv_size = n_dep_var * n_cells * sizeof(double);
  md->env_size = PMC_NUM_ENV_PARAM_ * n_cells * sizeof(double); //Temp and pressure
  md->rxn_env_data_size = n_rxn_env_param * n_cells * sizeof(double);
  md->rxn_env_data_idx_size = (n_rxn+1) * sizeof(int);
  md->map_state_deriv_size = n_dep_var * n_cells * sizeof(int);
  md->small_data = 0;

  //Allocate streams array and update variables related to streams
  //md->md_id = n_solver_objects;
  //if(n_solver_objects==0){
    //stream_gpu = (hipStream_t *)malloc(n_streams_limit * sizeof(hipStream_t));
      //md->stream_gpu = (hipStream_t *)malloc(n_streams * sizeof(hipStream_t));
  //}
  //n_solver_objects++;

  //Detect if we are working with few data values
  //todo check if it's worth to maintain this case (we will use small_data?)
  if (n_dep_var*n_cells < DATA_SIZE_LIMIT_OPT){
    md->small_data = 0;//1;
  }

  //Set working GPU: we have 4 gpu available on power9. as default, it should be assign to gpu 0
  int device=0;
  hipSetDevice(device);

  //Set GPU properties
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  //Set max threads without triggering too many resources error
  md->max_n_gpu_thread = prop.maxThreadsPerBlock/2;
  md->max_n_gpu_blocks = prop.maxGridSize[1];
  int n_blocks = (n_rxn + md->max_n_gpu_thread - 1) / md->max_n_gpu_thread;

  //GPU allocation
  ModelDataGPU *mGPU = &sd->mGPU;
  HANDLE_ERROR(hipMalloc((void **) &mGPU->deriv_data, md->deriv_size));
  mGPU->n_rxn=md->n_rxn;
  mGPU->n_rxn_env_data=md->n_rxn_env_data;
  mGPU->n_aero_phase=md->n_aero_phase;
  mGPU->n_added_aero_phases=md->n_added_aero_phases;
  mGPU->n_aero_rep=md->n_added_aero_reps;
  mGPU->n_aero_rep_env_data=md->n_aero_rep_env_data;

  hipMalloc((void **) &mGPU->state, md->state_size);
  hipMalloc((void **) &mGPU->env, md->env_size);
  hipMalloc((void **) &mGPU->rxn_env_data, md->rxn_env_data_size);
  hipMalloc((void **) &mGPU->rxn_env_data_idx, md->rxn_env_data_idx_size);
  HANDLE_ERROR(hipMalloc((void **) &mGPU->map_state_deriv, md->map_state_deriv_size));


  time_derivative_initialize_gpu(sd);

  //Mapping state-deriv
  md->map_state_deriv = (int *)malloc(md->map_state_deriv_size);
  int i_dep_var = 0;
  for (int i_cell = 0; i_cell < n_cells; i_cell++) {
    for (int i_spec = 0; i_spec < n_state_var; i_spec++) {
      if (md->var_type[i_spec] == CHEM_SPEC_VARIABLE) {
        md->map_state_deriv[i_dep_var] = i_spec + i_cell * n_state_var;
        //printf("%d %d, %d %d %d\n", md->map_state_deriv_size/sizeof(int),
        //       md->map_state_deriv[i_dep_var],n_state_var, i_spec, i_cell, i_dep_var);
        i_dep_var++;
      }
    }
  }

  HANDLE_ERROR(hipMemcpy(mGPU->map_state_deriv, md->map_state_deriv,
                          md->map_state_deriv_size, hipMemcpyHostToDevice));

  //HANDLE_ERROR(hipMemcpy(md->int_pointer_gpu, int_pointer, rxn_int_length*sizeof(int), hipMemcpyHostToDevice));

  //GPU allocation few data on pinned memory
  if(md->small_data){
    //Notice auxiliar variables are created because we
    // can't pin directly variables initialized before
    hipHostMalloc((void**)&md->deriv_aux, md->deriv_size);
  }
  else{
    md->deriv_aux = (realtype *)malloc(md->deriv_size);
  }

  printf("small_data:%d\n", md->small_data);
  //printf("threads_per_block :%d\n", md->max_n_gpu_thread);

  //GPU create streams
  //for (int i = 0; i < n_streams; ++i)
  //  HANDLE_ERROR( hipStreamCreate(&md->stream_gpu[i]) );

  // Warning if exceeding GPU limits
  if( n_blocks > md->max_n_gpu_blocks){
    printf("\nWarning: More blocks assigned: %d than maximum block numbers: %d",
           n_blocks, md->max_n_gpu_blocks);
  }

#ifdef PMC_DEBUG_PRINT_GPU_SPECS
  print_gpu_specs();
#endif

#ifdef PMC_DEBUG_GPU

  md->timeDerivKernel=0.0;
  hipEventCreate(&md->startDerivKernel);
  hipEventCreate(&md->stopDerivKernel);

#endif

}

/** \brief Set reaction data on GPU prepared structure. RXN data is divided
 * into two different matrix, per double and int data respectively. Matrix are
 * reversed to improve memory access on GPU.
 *
 * \param md Pointer to the model data
 */

void set_reverse_int_double_rxn(
  int n_rxn, int rxn_env_data_idx_size,
  int *rxn_int_data, double *rxn_float_data,
  int *rxn_int_indices, int *rxn_float_indices,
  int *rxn_env_idx,
  SolverData *sd
) {

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

  unsigned int int_max_length = 0;
  unsigned int double_max_length = 0;

  //RXN lengths
  unsigned int int_lengths[n_rxn];
  unsigned int double_lengths[n_rxn];

  //Position on the matrix for each row
  unsigned int rxn_position[n_rxn];

  //Get lengths for int and double arrays
  for (int i_rxn = 0; i_rxn < n_rxn; i_rxn++) {

    //Get RXN lengths
    int_lengths[i_rxn] = rxn_int_indices[i_rxn+1] - rxn_int_indices[i_rxn];
    double_lengths[i_rxn] = rxn_float_indices[i_rxn+1] - rxn_float_indices[i_rxn];

    //Update max size
    if(int_lengths[i_rxn]>int_max_length) int_max_length=int_lengths[i_rxn];
    if(double_lengths[i_rxn]>double_max_length) double_max_length=double_lengths[i_rxn];

    //Set initial position
    rxn_position[i_rxn] = i_rxn;

  }

  //Total lengths of rxn structure
  unsigned int rxn_int_length=n_rxn*int_max_length;
  unsigned int rxn_double_length=n_rxn*double_max_length;

  //Allocate int and double rxn data separately
  //Add -1 to avoid access and have a square matrix
  int *rxn_int = (int *) malloc(rxn_int_length * sizeof(int));
  memset(rxn_int, -1, rxn_int_length * sizeof(int));

  //Add 0 to avoid access and have a square matrix
  double *rxn_double = (double*)calloc(rxn_double_length, sizeof(double));

  int rxn_env_data_idx_aux[n_rxn];

  for (int i_rxn = 0; i_rxn < n_rxn; i_rxn++) {
    int i_pos=rxn_position[i_rxn];//i_rxn;//rxn_position[i_rxn];//for bubblesort
    for (int j = 0; j < int_lengths[i_pos]; j++){
      int *rxn_int_data_aux = &(rxn_int_data[rxn_int_indices[i_pos]]);
      rxn_int[n_rxn*j + i_rxn] = rxn_int_data_aux[j];
    }
    for (int j = 0; j < double_lengths[i_pos]; j++) {
      double *rxn_float_data_aux = &(rxn_float_data[rxn_float_indices[i_pos]]);
      rxn_double[n_rxn*j + i_rxn] = rxn_float_data_aux[j];
    }
    //Reorder the rate indices
    //Todo update on main code the rxn_env_data to read consecutively in cpu
    rxn_env_data_idx_aux[i_rxn] = rxn_env_idx[i_pos];
  }

  //GPU allocation
  hipMalloc((void **) &mGPU->rxn_int, rxn_int_length * sizeof(int));
  hipMalloc((void **) &mGPU->rxn_double, rxn_double_length * sizeof(double));

  //Save data to GPU
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_int, rxn_int, rxn_int_length*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_double, rxn_double, rxn_double_length*sizeof(double), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_env_data_idx, rxn_env_data_idx_aux, rxn_env_data_idx_size, hipMemcpyHostToDevice));

  free(rxn_int);
  free(rxn_double);

}

void set_int_double_rxn(
        int n_rxn, int rxn_env_data_idx_size,
        int *rxn_int_data, double *rxn_float_data,
        int *rxn_int_indices, int *rxn_float_indices,
        int *rxn_env_idx,
        SolverData *sd
) {

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

  //GPU allocation
  hipMalloc((void **) &mGPU->rxn_int, (md->n_rxn_int_param + md->n_rxn)*sizeof(int));
  hipMalloc((void **) &mGPU->rxn_double, md->n_rxn_float_param*sizeof(double));
#ifdef REVERSE_INT_FLOAT_MATRIX
#else
  hipMalloc((void **) &mGPU->rxn_int_indices, (md->n_rxn+1)*sizeof(int));
  hipMalloc((void **) &mGPU->rxn_float_indices, (md->n_rxn+1)*sizeof(int));
#endif

  //Save data to GPU
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_int, rxn_int_data,(md->n_rxn_int_param + md->n_rxn)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_double, rxn_float_data, md->n_rxn_float_param*sizeof(double), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_env_data_idx, rxn_env_idx, rxn_env_data_idx_size, hipMemcpyHostToDevice));
#ifdef REVERSE_INT_FLOAT_MATRIX
#else
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_int_indices, md->rxn_int_indices,(md->n_rxn+1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_float_indices, md->rxn_float_indices,(md->n_rxn+1)*sizeof(int), hipMemcpyHostToDevice));
#endif

}

void set_int_double_aero(
        SolverData *sd
) {

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

  //GPU allocation
  hipMalloc((void **) &mGPU->aero_phase_int_indices, (md->n_aero_phase + 1) * sizeof(int));
  hipMalloc((void **) &mGPU->aero_phase_float_indices, (md->n_aero_phase + 1) * sizeof(int));
  hipMalloc((void **) &mGPU->aero_phase_int_data, md->n_aero_phase_int_param * sizeof(int));
  hipMalloc((void **) &mGPU->aero_phase_float_data, md->n_aero_phase_float_param * sizeof(double));

  hipMalloc((void **) &mGPU->aero_rep_int_indices, (md->n_aero_rep + 1) * sizeof(int));
  hipMalloc((void **) &mGPU->aero_rep_float_indices, (md->n_aero_rep + 1) * sizeof(int));
  hipMalloc((void **) &mGPU->aero_rep_env_idx, (md->n_aero_rep + 1) * sizeof(int));
  hipMalloc((void **) &mGPU->aero_rep_int_data, (md->n_aero_rep_int_param + md->n_aero_rep) * sizeof(int));
  hipMalloc((void **) &mGPU->aero_rep_float_data, md->n_aero_rep_float_param * sizeof(double));
  //hipMalloc((void **) &mGPU->grid_cell_aero_rep_env_data, (md->n_aero_rep_env_data*md->n_cells) * sizeof(double));
  hipMalloc((void **) &mGPU->aero_rep_env_data, (md->n_aero_rep_env_data*md->n_cells) * sizeof(double));

  //Save data to GPU
  HANDLE_ERROR(hipMemcpy(mGPU->aero_phase_int_indices, md->aero_phase_int_indices, (md->n_aero_phase + 1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_phase_float_indices, md->aero_phase_float_indices, (md->n_aero_phase + 1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_phase_int_data, md->aero_phase_int_data, md->n_aero_phase_int_param*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_phase_float_data, md->aero_phase_float_data, md->n_aero_phase_float_param*sizeof(double), hipMemcpyHostToDevice));

  HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_int_indices, md->aero_rep_int_indices, (md->n_aero_rep + 1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_float_indices, md->aero_rep_float_indices, (md->n_aero_rep + 1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_env_idx, md->aero_rep_env_idx, (md->n_aero_rep + 1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_int_data, md->aero_rep_int_data, (md->n_aero_rep_int_param + md->n_aero_rep)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_float_data, md->aero_rep_float_data, md->n_aero_rep_float_param*sizeof(double), hipMemcpyHostToDevice));
  //HANDLE_ERROR(hipMemcpy(mGPU->grid_cell_aero_rep_env_data, md->grid_cell_aero_rep_env_data, (md->n_aero_rep_env_data*md->n_cells)*sizeof(double), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_env_data, md->aero_rep_env_data, (md->n_aero_rep_env_data*md->n_cells)*sizeof(double), hipMemcpyHostToDevice));

}

void solver_init_int_double_gpu(SolverData *sd) {

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

#ifdef REVERSE_INT_FLOAT_MATRIX

  set_reverse_int_double_rxn(
          md->n_rxn, md->rxn_env_data_idx_size,
          md->rxn_int_data, md->rxn_float_data,
          md->rxn_int_indices, md->rxn_float_indices, md->rxn_env_idx,
          sd
  );

#else

  set_int_double_rxn(
          md->n_rxn, md->rxn_env_data_idx_size,
          md->rxn_int_data, md->rxn_float_data,
          md->rxn_int_indices, md->rxn_float_indices, md->rxn_env_idx,
          sd
  );

#endif


/*
  set_int_double_aero(
          sd
  );

 */

}

void init_j_state_deriv_solver_gpu(SolverData *sd, double *J){

  ModelData *md = &(sd->model_data);

  //todo reduce allocations (use tmp pointers from cvode for j_tmp)
  md->jac_size = md->n_per_cell_solver_jac_elem * md->n_cells * sizeof(double);
  md->nnz_J_solver = SM_NNZ_S(md->J_solver);
  md->nrows_J_solver = SM_NP_S(md->J_solver);

  ModelDataGPU *mGPU = &sd->mGPU;
  hipMalloc((void **) &mGPU->J, md->jac_size);
  hipMalloc((void **) &mGPU->J_solver, md->jac_size);
  hipMalloc((void **) &mGPU->jJ_solver, md->nnz_J_solver*sizeof(int));
  hipMalloc((void **) &mGPU->iJ_solver, (md->nrows_J_solver+1)*sizeof(int));
  hipMalloc((void **) &mGPU->J_state, md->deriv_size);
  hipMalloc((void **) &mGPU->J_deriv, md->deriv_size);
  hipMalloc((void **) &mGPU->J_tmp, md->deriv_size);
  hipMalloc((void **) &mGPU->J_tmp2, md->deriv_size);


  double *J_solver = SM_DATA_S(md->J_solver);
  //Transfer sunindextype to int
  int *jJ_solver=(int*)malloc(sizeof(int)*md->nnz_J_solver);
  int *iJ_solver=(int*)malloc(sizeof(int)*md->nrows_J_solver+1);
  for(int i=0;i<md->nnz_J_solver;i++)
    jJ_solver[i]=SM_INDEXVALS_S(md->J_solver)[i];
  for(int i=0;i<=md->nrows_J_solver;i++)
    iJ_solver[i]=SM_INDEXPTRS_S(md->J_solver)[i];
  double *J_state = N_VGetArrayPointer(md->J_state);
  double *J_deriv = N_VGetArrayPointer(md->J_deriv);
  double *J_tmp = N_VGetArrayPointer(md->J_tmp);
  double *J_tmp2 = N_VGetArrayPointer(md->J_tmp2);

  HANDLE_ERROR(hipMemcpy(mGPU->J, J, md->jac_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_solver, J_solver, md->jac_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->jJ_solver, jJ_solver, md->nnz_J_solver*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->iJ_solver, iJ_solver, (md->nrows_J_solver+1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_state, J_state, md->deriv_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_deriv, J_deriv, md->deriv_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_tmp, J_tmp, md->deriv_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemset(mGPU->J_tmp2, 0.0, md->deriv_size));


  if(md->small_data){
    hipHostMalloc((void**)&md->jac_aux, md->jac_size);
  }

}

void update_jac_data_gpu(SolverData *sd, double *J){

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

  double *J_solver = SM_DATA_S(md->J_solver);
  double *J_state = N_VGetArrayPointer(md->J_state);
  double *J_deriv = N_VGetArrayPointer(md->J_deriv);
  HANDLE_ERROR(hipMemcpy(mGPU->J, J, md->jac_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_solver, J_solver, md->jac_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_state, J_state, md->deriv_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_deriv, J_deriv, md->deriv_size, hipMemcpyHostToDevice));

  //HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_float_data, md->aero_rep_float_data, md->n_aero_rep_float_param*sizeof(double), hipMemcpyHostToDevice));

}

void update_aero_contrib_gpu(SolverData *sd){

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

  HANDLE_ERROR(hipMemcpy(mGPU->state, md->total_state, md->state_size, hipMemcpyHostToDevice));
  //HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_float_data, md->aero_rep_float_data, md->n_aero_rep_float_param*sizeof(double), hipMemcpyHostToDevice));

  int num_spec = md->n_per_cell_dep_var*md->n_cells;
  HANDLE_ERROR(hipMemcpy(mGPU->production_rates, sd->time_deriv.production_rates, num_spec*sizeof(mGPU->production_rates), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->loss_rates, sd->time_deriv.loss_rates, num_spec*sizeof(mGPU->loss_rates), hipMemcpyHostToDevice));

}


void rxn_update_env_state_gpu(SolverData *sd){

  ModelData *md = &(sd->model_data);
  int n_cells = md->n_cells;
  int n_rxn = md->n_rxn;
  int n_threads = n_rxn*n_cells; //Reaction group per number of repetitions/cells
  double *rxn_env_data = md->rxn_env_data;
  double *env = md->total_env;
  int n_blocks = ((n_threads + md->max_n_gpu_thread - 1) / md->max_n_gpu_thread);
  ModelDataGPU *mGPU = &sd->mGPU;

  //Faster, use for few values
  if (md->small_data){
    //This method of passing them as a function parameter has a theoric maximum of 4kb of data
    mGPU->rxn_env_data = rxn_env_data;
    mGPU->env = env;
  }
  //Slower, use for large values
  else{
    //Async memcpy
    //HANDLE_ERROR(hipMemcpyAsync(md->rxn_env_data_gpu, rxn_env_data,
    //        md->rxn_env_data_size, hipMemcpyHostToDevice, md->stream_gpu[STREAM_RXN_ENV_GPU]));
    //HANDLE_ERROR(hipMemcpyAsync(md->env_gpu, env, md->env_size,
    //        hipMemcpyHostToDevice, md->stream_gpu[STREAM_ENV_GPU]));


    HANDLE_ERROR(hipMemcpy(mGPU->rxn_env_data, rxn_env_data, md->rxn_env_data_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(mGPU->env, env, md->env_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(mGPU->state, md->total_state, md->state_size, hipMemcpyHostToDevice));

  }
}


__global__
void camp_solver_check_model_state_cuda(double *state_init, double *y,
        int *map_state_deriv, double threshhold, double replacement_value, int *status,
        int deriv_length_cell, int n_cells)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int active_threads = n_cells*deriv_length_cell;

  if(tid<active_threads) {

    if (y[tid] > -SMALL) {
      state_init[map_state_deriv[tid]] =
      y[tid] > threshhold ?
      y[tid] : replacement_value;

      //state_init[map_state_deriv[tid]] = 0.1;
      //printf("tid %d map_state_deriv %d\n", tid, map_state_deriv[tid]);
    } else {
      *status = CAMP_SOLVER_FAIL;
#ifdef FAILURE_DETAIL
      printf("\nFailed model state update gpu (Negative value on 'y'):[spec %d] = %le",tid,y[tid]);
#endif
    }
  }

}

int camp_solver_check_model_state_gpu(N_Vector solver_state, SolverData *sd,
                                      double threshhold, double replacement_value)
{
  ModelData *md = &(sd->model_data);
  itsolver *bicg = &(sd->bicg);
  int status = CAMP_SOLVER_SUCCESS; //0
  int n_cells = md->n_cells;
  int n_state_var = md->n_per_cell_state_var;
  int n_dep_var = md->n_per_cell_dep_var;
  int n_threads = n_dep_var*n_cells;
  int n_blocks = ((n_threads + md->max_n_gpu_thread - 1) / md->max_n_gpu_thread);
  int *var_type = md->var_type;
  double *state = md->total_state;
  double *y = NV_DATA_S(solver_state);
  ModelDataGPU *mGPU = &sd->mGPU;

/*
  //HANDLE_ERROR(hipMemcpy(md->deriv_aux, bicg->dcv_y, md->deriv_size, hipMemcpyDeviceToHost));
  if(sd->counterDerivCPU<=5){
    printf("counterDeriv2 %d \n", sd->counterDerivCPU);
    for (int i = 0; i < NV_LENGTH_S(solver_state); i++) {
        //printf("(%d) %-le ", i + 1, NV_DATA_S(deriv)[i]);
      if(y[i]!=md->deriv_aux[i]) {
        printf("(%d) dy %-le y %-le\n", i + 1, md->deriv_aux[i], y[i]);
      }
    }
  }
*/



  camp_solver_check_model_state_cuda << < n_blocks, md->max_n_gpu_thread >> >
   (mGPU->state, bicg->dcv_y, mGPU->map_state_deriv,
   threshhold, replacement_value, &status, n_dep_var, n_cells);

  HANDLE_ERROR(hipMemcpy(md->total_state, mGPU->state, md->state_size, hipMemcpyDeviceToHost));




#ifdef DEBUG_CHECK_MODEL_STATE_CUDA
  for (int i_cell = 0; i_cell < n_cells; i_cell++) {
   for (int i_dep_var = 0; i_dep_var < n_dep_var; i_dep_var++) {

     printf("(%d) %-le \n", i_dep_var+1,
            md->total_state[md->map_state_deriv[i_dep_var]]);
   }
}
#endif

  return status;
}


void camp_solver_update_model_state_gpu(N_Vector solver_state, SolverData *sd,
                                       double threshhold, double replacement_value)
{
  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;
  HANDLE_ERROR(hipMemcpy(mGPU->state, md->total_state, md->state_size, hipMemcpyHostToDevice));

}

__device__ void solveRXN(
#ifdef BASIC_CALC_DERIV
        double *deriv_data,
#else
        TimeDerivativeGPU deriv_data,
#endif
       double time_step,
       ModelDataGPU *md
)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

#ifdef REVERSE_INT_FLOAT_MATRIX

  double *rxn_float_data = &( md->rxn_double[md->i_rxn]);
  int *int_data = &(md->rxn_int[md->i_rxn]);
  int rxn_type = int_data[0];
  int *rxn_int_data = (int *) &(int_data[1*md->n_rxn]);

#else

  double *rxn_float_data = &( md->rxn_double[md->rxn_float_indices[md->i_rxn]]);
  int *int_data = &(md->rxn_int[md->rxn_int_indices[md->i_rxn]]);

  //double *rxn_float_data = &( md->rxn_double[md->i_rxn]);
  //int *int_data = &(md->rxn_int[md->i_rxn]);


  int rxn_type = int_data[0];
  int *rxn_int_data = (int *) &(int_data[1]);

#endif

  //Get indices for rates
  double *rxn_env_data = &(md->rxn_env_data
  [md->n_rxn_env_data*md->i_cell+md->rxn_env_data_idx[md->i_rxn]]);

#ifdef DEBUG_DERIV_GPU
  if(tid==0){
    printf("[DEBUG] GPU solveRXN tid %d, \n", tid);
  }
#endif

  switch (rxn_type) {
    //case RXN_AQUEOUS_EQUILIBRIUM :
    //fix run-time error
    //rxn_gpu_aqueous_equilibrium_calc_deriv_contrib(md, deriv_data, rxn_int_data,
    //                                               rxn_float_data, rxn_env_data,time_step);
    //break;
    case RXN_ARRHENIUS :
      rxn_gpu_arrhenius_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                           rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CMAQ_H2O2 :
      rxn_gpu_CMAQ_H2O2_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                          rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CMAQ_OH_HNO3 :
      rxn_gpu_CMAQ_OH_HNO3_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                           rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CONDENSED_PHASE_ARRHENIUS :
      //rxn_gpu_condensed_phase_arrhenius_calc_deriv_contrib(md, deriv_data, rxn_int_data,
      //                                     rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_EMISSION :
      //rxn_gpu_emission_calc_deriv_contrib(md, deriv_data, rxn_int_data,
      //                                     rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_FIRST_ORDER_LOSS :
      //rxn_gpu_first_order_loss_calc_deriv_contrib(md, deriv_data, rxn_int_data,
      //                                     rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_HL_PHASE_TRANSFER :
      //rxn_gpu_HL_phase_transfer_calc_deriv_contrib(md, deriv_data, rxn_int_data,
      //                                             rxn_float_data, rxn_env_data,time_stepn);
      break;
    case RXN_PHOTOLYSIS :
      rxn_gpu_photolysis_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                           rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_SIMPOL_PHASE_TRANSFER :
      //rxn_gpu_SIMPOL_phase_transfer_calc_deriv_contrib(md, deriv_data,
      //        rxn_int_data, rxn_float_data, rxn_env_data, time_step);
      break;
    case RXN_TROE :
#ifdef BASIC_CALC_DERIV
#else
      rxn_gpu_troe_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                      rxn_float_data, rxn_env_data,time_step);
#endif
      break;
    case RXN_WET_DEPOSITION :
      //rxn_gpu_wet_deposition_calc_deriv_contrib(md, deriv_data, rxn_int_data,
      //                                     rxn_float_data, rxn_env_data,time_step);
      break;
  }
/*
*/

}

/** \brief GPU function: Solve derivative
 */
__global__ void solveDerivative(
#ifdef PMC_DEBUG_GPU
                          int counterDeriv2,
#endif
  double time_step, int deriv_length_cell, int state_size_cell,
  int n_cells,
  int i_kernel, int threads_block, double *y,
  double threshhold, double replacement_value, ModelDataGPU md_object
  ) //Interface CPU/GPU
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int tid_cell=tid%deriv_length_cell;
  int active_threads = n_cells*deriv_length_cell;
  ModelDataGPU *md = &md_object;

#ifdef DEBUG_DERIV_GPU
  if(tid==0){
    printf("[DEBUG] GPU solveDerivative tid %d, \n", tid);
  }__syncthreads();
#endif

  if(tid<active_threads){

  /* Use when all parts that need state are on the GPU (e.g. Jacobian)
    state_init[map_state_deriv[tid]] =
          y[tid] > threshhold ?
          y[tid] : replacement_value;
  */

  //N_VLinearSum(1.0, y, -1.0, md->J_state, md->J_tmp);
  cudaDevicezaxpby(1.0, y, -1.0, md->J_state, md->J_tmp, active_threads);
  //SUNMatMatvec(md->J_solver, md->J_tmp, md->J_tmp2);
  cudaDeviceSpmvCSC_block(md->J_tmp2, md->J_tmp, active_threads, md->J_solver, md->jJ_solver, md->iJ_solver);
  //N_VLinearSum(1.0, md->J_deriv, 1.0, md->J_tmp2, md->J_tmp);
  cudaDevicezaxpby(1.0, md->J_deriv, 1.0, md->J_tmp2, md->J_tmp, active_threads);
  cudaDevicesetconst(md->J_tmp2, 0.0, active_threads); //Reset for next iter

  //Debug
/*
  if(counterDeriv2<=10){
       printf("(%d) y %-le J_state %-le J_solver %-le J_tmp %-le J_tmp2 %-le J_deriv %-le\n",tid+1,
              y[tid], J_state[tid], J_solver[tid], J_tmp[tid], J_tmp2[tid], J_deriv[tid]);
       printf("gpu threads %d\n", active_threads);
    }
*/

#ifdef BASIC_CALC_DERIV
    md->i_rxn=tid%n_rxn;
    double *deriv_init = md->deriv_data;
    md->deriv_data = &( md->deriv_init[deriv_length_cell*md->i_cell]);
    if(tid < n_rxn*n_cells){
        solveRXN(deriv_data, time_step, md);
    }
#else
    TimeDerivativeGPU deriv_data;
    deriv_data.num_spec = deriv_length_cell*n_cells;

#ifndef AEROS_CPU
#else
    deriv_data.production_rates = md->production_rates;
    deriv_data.loss_rates = md->loss_rates;
    time_derivative_reset_gpu(deriv_data);
    __syncthreads();
#endif

    int i_cell = tid/deriv_length_cell;
    md->i_cell = i_cell;
    deriv_data.production_rates = &( md->production_rates[deriv_length_cell*i_cell]);
    deriv_data.loss_rates = &( md->loss_rates[deriv_length_cell*i_cell]);

    md->grid_cell_state = &( md->state[state_size_cell*i_cell]);
    md->grid_cell_env = &( md->env[PMC_NUM_ENV_PARAM_*i_cell]);

    /*
    md->grid_cell_aero_rep_env_data =
    &(md->aero_rep_env_data[md->i_cell*md->n_aero_rep_env_data]);

    //Filter threads for n_aero_rep
    int n_aero_rep = md->n_aero_rep;
    if( tid_cell < n_aero_rep) {
      int n_iters = n_aero_rep / deriv_length_cell;
      //Repeat if there are more reactions than species
      for (int i = 0; i < n_iters; i++) {
        md->i_aero_rep = tid_cell + i*deriv_length_cell;

        aero_rep_gpu_update_state(md);
      }

      //Limit tid to pending rxns to compute
      int residual=n_aero_rep-(deriv_length_cell*n_iters);
      if(tid_cell < residual){
        md->i_aero_rep = tid_cell + deriv_length_cell*n_iters;

        aero_rep_gpu_update_state(md);
      }
    }
     */

    //Filter threads for n_rxn
    int n_rxn = md->n_rxn;
    if( tid_cell < n_rxn) {
      int n_iters = n_rxn / deriv_length_cell;
      //Repeat if there are more reactions than species
      for (int i = 0; i < n_iters; i++) {
        md->i_rxn = tid_cell + i*deriv_length_cell;

        solveRXN(deriv_data, time_step, md);
      }

      //Limit tid to pending rxns to compute
      int residual=n_rxn-(deriv_length_cell*n_iters);
      if(tid_cell < residual){
        md->i_rxn = tid_cell + deriv_length_cell*n_iters;

        solveRXN(deriv_data, time_step, md);
      }
    }
    __syncthreads();

    /*if(tid==0){
      printf("tid %d time_deriv.production_rates %-le time_deriv.loss_rates %-le\n",
              tid, deriv_data.production_rates[tid],
             deriv_data.loss_rates[tid]);
    }*/

    deriv_data.production_rates = md->production_rates;
    deriv_data.loss_rates = md->loss_rates;
    __syncthreads();
    time_derivative_output_gpu(deriv_data, md->deriv_data, md->J_tmp,0);
#endif

    /*
    if(tid<deriv_data.num_spec && tid>1022){
      //if(tid<1){
      //deriv_init[tid] = deriv_data.production_rates[tid];
      //deriv_init[tid] = deriv_data.loss_rates[tid];
      printf("tid %d time_deriv.production_rates %-le time_deriv.loss_rates %-le"
             "deriv_init %-le\n",
             tid, deriv_data.production_rates[tid],
             deriv_data.loss_rates[tid],
             //deriv_data.loss_rates[tid]);
             deriv_init[tid]);
    }*/

  }

}

/** \brief Calculate the time derivative \f$f(t,y)\f$ on GPU
 *
 * \param md Pointer to the model data
 * \param deriv NVector to hold the calculated vector
 * \param time_step Current model time step (s)
 */
void rxn_calc_deriv_gpu(SolverData *sd, N_Vector deriv, double time_step,
        double threshhold, double replacement_value) {

  ModelData *md = &(sd->model_data);
  itsolver *bicg = &(sd->bicg);
  double *deriv_data = N_VGetArrayPointer(deriv);
  int n_cells = md->n_cells;
  int n_kernels = 1; // Divide load into multiple kernel calls
  //todo n_kernels case division left residual, an extra kernel computes remain residual
#ifdef BASIC_CALC_DERIV
  int total_threads = md->n_rxn*n_cells/n_kernels; //Reaction group per number of repetitions/cells
  int threads_block = md->max_n_gpu_thread;
#else
  int n_per_cell_dep_var = md->n_per_cell_dep_var;
  int total_threads = n_per_cell_dep_var * n_cells/n_kernels;
  int n_shr_empty = md->max_n_gpu_thread%n_per_cell_dep_var;
  int threads_block = md->max_n_gpu_thread - n_shr_empty; //last multiple of size_cell before max_threads
#endif
  int n_blocks = ((total_threads + threads_block - 1) / threads_block);
  double *J_tmp = N_VGetArrayPointer(md->J_tmp);
  ModelDataGPU *mGPU = &sd->mGPU;

#ifndef DERIV_CPU_ON_GPU

  //Transfer cv_ftemp() not needed because bicg->dftemp=md->deriv_data_gpu;
  //hipMemcpy(cv_ftemp_data,bicg->dftemp,bicg->nrows*sizeof(double),hipMemcpyDeviceToHost);

  HANDLE_ERROR(hipMemcpy(mGPU->deriv_data, deriv_data, md->deriv_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->state, md->total_state, md->state_size, hipMemcpyHostToDevice));


#else

 //debug
  /*if(sd->counterDerivCPU>=0){
    printf("camp solver_run start [(id),conc], n_state_var %d, n_cells %d\n", md->n_per_cell_state_var, n_cells);
    printf("n_deriv %d\n", md->n_per_cell_dep_var);
    for (int i = 0; i < md->n_per_cell_state_var*n_cells; i++) {
      //printf("(%d) %-le \n",i+1, md->total_state[i]);
    }
  }*/

#ifdef BASIC_CALC_DERIV
  //Reset deriv gpu
  //check if cudamemset work fine with doubles
  HANDLE_ERROR(hipMemset(md->deriv_data_gpu, 0.0, md->deriv_size));
#endif

#ifdef PMC_DEBUG_GPU
  //timeDerivSend += (clock() - t1);
  //clock_t t2 = clock();

  hipEventRecord(md->startDerivKernel);

#endif

#ifndef AEROS_CPU

  update_aero_contrib_gpu(sd);

#endif

  //Loop to test multiple kernel executions
  for (int i_kernel=0; i_kernel<n_kernels; i_kernel++){
    //hipDeviceSynchronize();
    solveDerivative << < (n_blocks), threads_block >> >(
#ifdef PMC_DEBUG_GPU
    sd->counterDerivCPU,
#endif
     time_step, md->n_per_cell_dep_var,
     md->n_per_cell_state_var,n_cells,
     i_kernel, threads_block, bicg->dcv_y,
     threshhold, replacement_value, sd->mGPU
     );
  }

#ifdef PMC_DEBUG_GPU
  /*hipDeviceSynchronize();
  timeDerivKernel += (clock() - t2);
  t3 = clock();*/


  hipEventRecord(md->stopDerivKernel);
  hipEventSynchronize(md->stopDerivKernel);
  float msDerivKernel = 0.0;
  hipEventElapsedTime(&msDerivKernel, md->startDerivKernel, md->stopDerivKernel);
  md->timeDerivKernel+= msDerivKernel;


#endif

  //Use pinned memory for few values
  if (md->small_data){

    HANDLE_ERROR(hipMemcpy(md->deriv_aux, mGPU->deriv_data, md->deriv_size, hipMemcpyDeviceToHost));

    memcpy(deriv_data, md->deriv_aux, md->deriv_size);
  }
  else {
    //Async
    //HANDLE_ERROR(hipMemcpyAsync(md->deriv_aux, md->deriv_data_gpu,
    //md->deriv_size, hipMemcpyDeviceToHost, md->stream_gpu[STREAM_DERIV_GPU]));

    //Sync
    //HANDLE_ERROR(hipMemcpy(md->deriv_aux, md->deriv_data_gpu, md->deriv_size, hipMemcpyDeviceToHost));

    //todo i think not necessary
    HANDLE_ERROR(hipMemcpy(deriv_data, mGPU->deriv_data, md->deriv_size, hipMemcpyDeviceToHost));

  }

  //hipDeviceSynchronize();

 //debug
/*
  if(sd->counterDerivGPU<=0 ){
    int size_j = NV_LENGTH_S(deriv);
    printf("length_deriv %d \n", size_j);
    for (int i = 0; i < 1; i++) {//n_cells
      printf("cell %d \n", i);
      for (int j = 0; j < size_j; j++) {  // NV_LENGTH_S(deriv)
        printf("(%d) %-le ", j + 1, NV_DATA_S(deriv)[j+i*size_j]);
      }
      printf("\n");
    }
  }
*/

#ifdef PMC_DEBUG_GPU
  /*timeDerivReceive += (clock() - t3);
  timeDeriv += (clock() - t1);
  t3 = clock();*/
#endif

#endif

}

void get_f_from_gpu(SolverData *sd){

  //HANDLE_ERROR(hipMemcpy(mGPU->state, J, md->jac_size, hipMemcpyHostToDevice));

}

void get_guess_helper_from_gpu(N_Vector y_n, N_Vector y_n1,
        N_Vector hf, void *solver_data, N_Vector tmp1,
        N_Vector corr){

  //HANDLE_ERROR(hipMemcpy(mGPU->state, J, md->jac_size, hipMemcpyHostToDevice));


}

/** \brief Fusion deriv data calculated from CPU and GPU
 * (Calculations from CPU & GPU or GPU async case)
 *
 * \param md Pointer to the model data
 * \param deriv NVector to hold the calculated vector
 * \param time_step Current model time step (s)
 */
void rxn_fusion_deriv_gpu(ModelData *md, N_Vector deriv) {

  // Get a pointer to the derivative data
  realtype *deriv_data = N_VGetArrayPointer(deriv);

  hipDeviceSynchronize();
  //HANDLE_ERROR(hipMemsetAsync(md->deriv_data_gpu, 0.0,
  //        md->deriv_size, md->stream_gpu[STREAM_DERIV_GPU]));

  if (md->small_data){
  }
  else {
    for (int i = 0; i < NV_LENGTH_S(deriv); i++) {  // NV_LENGTH_S(deriv)
      //Add to deriv the auxiliar contributions from gpu
      deriv_data[i] += md->deriv_aux[i];
    }
  }

}

#ifdef PMC_USE_GPU
#else
void rxn_calc_deriv_cpu(ModelData *md, double *deriv_data,
                    double time_step) {

  //clock_t t = clock();

  // Get the number of reactions
  int n_rxn = md->n_rxn;

  // Loop through the reactions advancing the rxn_data pointer each time
  for (int i_rxn = 0; i_rxn < n_rxn; i_rxn++) {
    // Get pointers to the reaction data
    int *rxn_int_data =
        &(md->rxn_int_data[md->rxn_int_indices[i_rxn]]);
    double *rxn_float_data =
        &(md->rxn_float_data[md->rxn_float_indices[i_rxn]]);
    double *rxn_env_data =
        &(md->grid_cell_rxn_env_data[md->rxn_env_idx[i_rxn]]);

    // Get the reaction type
    int rxn_type = *(rxn_int_data++);

    // Call the appropriate function
    switch (rxn_type) {
      case RXN_AQUEOUS_EQUILIBRIUM:
        rxn_gpu_aqueous_equilibrium_calc_deriv_contrib(md, deriv_data,
                                                   rxn_int_data, rxn_float_data,
                                                   rxn_env_data, time_step);
        break;
      case RXN_ARRHENIUS:
        rxn_gpu_arrhenius_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                         rxn_float_data, rxn_env_data,
                                         time_step);
        break;
      case RXN_CMAQ_H2O2:
        rxn_gpu_CMAQ_H2O2_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                         rxn_float_data, rxn_env_data,
                                         time_step);
        break;
      case RXN_CMAQ_OH_HNO3:
        rxn_gpu_CMAQ_OH_HNO3_calc_deriv_contrib(md, deriv_data,
                                            rxn_int_data, rxn_float_data,
                                            rxn_env_data, time_step);
        break;
      case RXN_CONDENSED_PHASE_ARRHENIUS:
        rxn_gpu_condensed_phase_arrhenius_calc_deriv_contrib(
            md, deriv_data, rxn_int_data, rxn_float_data, rxn_env_data,
            time_step);
        break;
      case RXN_EMISSION:
        rxn_gpu_emission_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                        rxn_float_data, rxn_env_data,
                                        time_step);
        break;
      case RXN_FIRST_ORDER_LOSS:
        rxn_gpu_first_order_loss_calc_deriv_contrib(md, deriv_data,
                                                rxn_int_data, rxn_float_data,
                                                rxn_env_data, time_step);
        break;
      case RXN_HL_PHASE_TRANSFER:
        //rxn_gpu_HL_phase_transfer_calc_deriv_contrib(md, deriv_data,
        //                                         rxn_int_data, rxn_float_data,
        //                                         rxn_env_data, time_step);
        break;
      case RXN_PHOTOLYSIS:
        rxn_gpu_photolysis_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                          rxn_float_data, rxn_env_data,
                                          time_step);
        break;
      case RXN_SIMPOL_PHASE_TRANSFER:
        //rxn_gpu_SIMPOL_phase_transfer_calc_deriv_contrib(
        //   md, deriv_data, rxn_int_data, rxn_float_data, rxn_env_data,
        //    time_step);
        break;
      case RXN_TROE:
        rxn_gpu_troe_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                    rxn_float_data, rxn_env_data, time_step);
        break;
      case RXN_WET_DEPOSITION:
        rxn_gpu_wet_deposition_calc_deriv_contrib(md, deriv_data,
                                              rxn_int_data, rxn_float_data,
                                              rxn_env_data, time_step);
        break;
    }
  }

  //timeDeriv += (clock()- t);

}
#endif

/** \brief GPU function: Solve jacobian
 *
 * \param state_init Pointer to first value of state array
 * \param jac_init Pointer to first value of jacobian array
 * \param time_step Current time step being computed (s)
 * \param jac_length_cell jacobian length for one cell
 * \param md->state_size_cell jacobian length for one cell
 * \param n_rxn Number of reactions to include
 * \param n_cells_gpu Number of cells to compute
 * \param md->rxn_int Pointer to integer reaction data
 * \param md->rxn_double Pointer to double reaction data
 * \param rxn_env_data_init Pointer to first value of reaction rates
 */
__global__ void solveJacobian(double *state_init, double *jac_init,
                              double time_step, int jac_length_cell, int state_size_cell,
                              int n_rxn_env_data_cell, int n_rxn,
                              int n_cells, int *rxn_int, double *rxn_double,
                              double *rxn_env_data_init, int *rxn_env_data_idx) //Interface CPU/GPU
{
  //Get thread id
  /*int index = blockIdx.x * blockDim.x + threadIdx.x;

  //Maximum number of threads to compute all reactions
  if(index < n_rxn*n_cells){

    //Thread index for jac and state,
    // till we don't finish all reactions of a cell, we stay on same index
    int i_cell=index/n_rxn;
    int i_rxn=index%n_rxn;

    //Get indices of each reaction
    int *int_data = (int *) &(((int *) rxn_int)[i_rxn]); //Same indices for each cell
    double *float_data = (double *) &(((double *) rxn_double)[i_rxn]);
    int rxn_type = int_data[0];
    int *rxn_int_data = (int *) &(int_data[1*n_rxn]);

    //Get indices for concentrations
    double *jac_data = &( jac_init[jac_length_cell*i_cell]);
    double *state = &( state_init[state_size_cell*i_cell]);

    //Get indices for rates
    double *rxn_env_data = &(rxn_env_data_init
    [n_rxn_env_data_cell*i_cell+rxn_env_data_idx[i_rxn]]);

    switch (rxn_type) {
      case RXN_AQUEOUS_EQUILIBRIUM :
        //rxn_gpu_aqueous_equilibrium_calc_jac_contrib(rxn_env_data,
        //        state, jac_data, rxn_int_data, rxn_float_data, time_step, n_rxn);
        break;
      case RXN_ARRHENIUS :
        rxn_gpu_arrhenius_calc_jac_contrib(rxn_env_data,
                                           state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_CMAQ_H2O2 :
        rxn_gpu_CMAQ_H2O2_calc_jac_contrib(rxn_env_data,
                                           state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_CMAQ_OH_HNO3 :
        rxn_gpu_CMAQ_OH_HNO3_calc_jac_contrib(rxn_env_data,
                                              state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_CONDENSED_PHASE_ARRHENIUS :
        //rxn_gpu_condensed_phase_arrhenius_calc_jac_contrib(rxn_env_data,
        //        state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_EMISSION :
        rxn_gpu_emission_calc_jac_contrib(rxn_env_data,
                                          state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_FIRST_ORDER_LOSS :
        rxn_gpu_first_order_loss_calc_jac_contrib(rxn_env_data,
                                                  state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_HL_PHASE_TRANSFER :
        //rxn_gpu_HL_phase_transfer_calc_jac_contrib(rxn_env_data,
        //        state, jac_data, rxn_int_data, rxn_float_data, time_step, n_rxn);
        break;
      case RXN_PHOTOLYSIS :
        rxn_gpu_photolysis_calc_jac_contrib(rxn_env_data,
                                            state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_SIMPOL_PHASE_TRANSFER :
        //rxn_gpu_SIMPOL_phase_transfer_calc_jac_contrib(rxn_env_data,
        //        state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
      case RXN_TROE :
        rxn_gpu_troe_calc_jac_contrib(rxn_env_data,
                                      state, jac_data, rxn_int_data, rxn_float_data, time_step, n_rxn);
        break;
      case RXN_WET_DEPOSITION :
        rxn_gpu_wet_deposition_calc_jac_contrib(rxn_env_data,
                                                state, jac_data, rxn_int_data, rxn_float_data, time_step,n_rxn);
        break;
    }
    __syncthreads();
  }
   */

}


/** \brief Calculate the Jacobian on GPU
 *
 * \param md Pointer to the model data
 * \param J Jacobian to be calculated
 * \param time_step Current model time step (s)
 */

void rxn_calc_jac_gpu(SolverData *sd, SUNMatrix jac, double time_step) {

  //TODO

}

/** \brief Free GPU data structures
 */
void free_gpu_cu(SolverData *sd) {

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

#ifdef PMC_DEBUG_GPU

  printf("timeDerivKernel %lf\n", md->timeDerivKernel/1000);

#endif

  //for (int i = 0; i < n_streams; ++i)
  //  HANDLE_ERROR( hipStreamDestroy(md->stream_gpu[i]) );
/*

  */
  //free(md->jac_aux);
  HANDLE_ERROR(hipFree(mGPU->rxn_int));
  HANDLE_ERROR(hipFree(mGPU->rxn_double));
  HANDLE_ERROR(hipFree(mGPU->deriv_data));
  //HANDLE_ERROR(hipFree(J_solver_gpu));

  if(md->small_data){
  }
  else{
    free(md->deriv_aux);
    HANDLE_ERROR(hipFree(mGPU->state));
    HANDLE_ERROR(hipFree(mGPU->env));
    HANDLE_ERROR(hipFree(mGPU->rxn_env_data));
    HANDLE_ERROR(hipFree(mGPU->rxn_env_data_idx));

  }

}

/* Auxiliar functions */

void bubble_sort_gpu(unsigned int *n_zeros, unsigned int *rxn_position, int n_rxn){

  int tmp,s=1,i_rxn=n_rxn;

  while(s){
    s=0;
    for (int i = 1; i < i_rxn; i++) {
      //Few zeros go first
      if (n_zeros[i] < n_zeros[i - 1]) {
        //Swap positions
        tmp = rxn_position[i];
        rxn_position[i] = rxn_position[i - 1];
        rxn_position[i - 1] = tmp;

        tmp = n_zeros[i];
        n_zeros[i] = n_zeros[i - 1];
        n_zeros[i - 1] = tmp;
        s=1;
      }
    }
    i_rxn--;
  }

}

/* Prints */

void print_gpu_specs() {

  printf("GPU specifications \n");

  int nDevices;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  maxGridSize: %d\n", prop.maxGridSize[1]);
    printf("  maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
    printf("  maxThreadsDim: %d\n", prop.maxThreadsDim[1]);
    printf("  totalGlobalMem: %zu\n", prop.totalGlobalMem);
    printf("  sharedMemPerBlock: %zu\n", prop.sharedMemPerBlock); //bytes
    printf("  multiProcessorCount: %d\n", prop.multiProcessorCount);
  }

}

// Old code (Not used now, but could be useful)
/*
 //use this instead of normal update_model_state? is less code
int camp_solver_update_model_state_cpu(N_Vector solver_state, ModelData *md,
                                       realtype threshhold, realtype replacement_value)
{
  int status = CAMP_SOLVER_FAIL;
  int n_cells = md->n_cells;
  int n_state_var = md->n_per_cell_state_var;
  int n_dep_var = md->n_per_cell_dep_var;
  int n_threads = n_state_var*n_cells;
  int n_blocks = ((n_threads + md->max_n_gpu_thread - 1) / md->max_n_gpu_thread);
  int *var_type = md->var_type;
  double *state = md->total_state;
  double *y = NV_DATA_S(solver_state);
  int *map_state_deriv = md->map_state_deriv;

  for(int i_dep_var = 0; i_dep_var < n_dep_var*n_cells; i_dep_var++)
  {
    if (NV_DATA_S(solver_state)[i_dep_var] > -SMALL) {
      md->total_state[map_state_deriv[i_dep_var]] =
              NV_DATA_S(solver_state)[i_dep_var] > threshhold
              ? NV_DATA_S(solver_state)[i_dep_var] : replacement_value;
      status = CAMP_SOLVER_SUCCESS;
    } else { //error
#ifdef FAILURE_DETAIL
      printf("\nFailed model state update: [spec %d] = %le", i_spec,
                 NV_DATA_S(solver_state)[i_dep_var]);
#endif
      status = CAMP_SOLVER_FAIL;
      break;
    }
  }
  return status;
}
*/
}

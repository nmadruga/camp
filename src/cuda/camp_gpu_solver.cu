#include "hip/hip_runtime.h"
/* Copyright (C) 2019 Christian Guzman
 * Licensed under the GNU General Public License version 1 or (at your
 * option) any later version. See the file COPYING for details.
 *
 * Interface Host-Device (CPU-GPU) to compute reaction-specific functions on GPU
 *
 */

#include "itsolver_gpu.h"

extern "C" {
#include "camp_gpu_solver.h"
#include "rxns_gpu.h"
#include "aeros/aero_rep_gpu_solver.h"
#include "time_derivative_gpu.h"
#include "Jacobian_gpu.h"
}


// Reaction types (Must match parameters defined in pmc_rxn_factory)
#define RXN_ARRHENIUS 1
#define RXN_TROE 2
#define RXN_CMAQ_H2O2 3
#define RXN_CMAQ_OH_HNO3 4
#define RXN_PHOTOLYSIS 5
#define RXN_HL_PHASE_TRANSFER 6
#define RXN_AQUEOUS_EQUILIBRIUM 7
#define RXN_SIMPOL_PHASE_TRANSFER 10
#define RXN_CONDENSED_PHASE_ARRHENIUS 11
#define RXN_FIRST_ORDER_LOSS 12
#define RXN_EMISSION 13
#define RXN_WET_DEPOSITION 14

#define STREAM_RXN_ENV_GPU 0
#define STREAM_ENV_GPU 1
#define STREAM_DERIV_GPU 2

// Status codes for calls to camp_solver functions
#define CAMP_SOLVER_SUCCESS 0
#define CAMP_SOLVER_FAIL 1

//GPU async stream related variables to ensure robustness
//int n_solver_objects=0; //Number of solver_new_gpu calls
//hipStream_t *stream_gpu; //GPU streams to async computation/data movement
//int n_streams = 16;

//Gpu hardware info
//int md->max_n_gpu_thread;
//int md->max_n_gpu_blocks;

static void HandleError(hipError_t err,
                        const char *file,
                        int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err),
           file, line);
    exit(EXIT_FAILURE);
  }
}

/** \brief Allocate GPU solver variables
 *
 * \param n_dep_var number of solver variables per grid cell
 * \param n_state_var Number of variables on the state array per grid cell
 * \param n_rxn Number of reactions to include
 * \param n_rxn_int_param Total number of integer reaction parameters
 * \param n_rxn_float_param Total number of floating-point reaction parameters
 * \param n_cells Number of grid cells to solve simultaneously
 */
void solver_new_gpu_cu(SolverData *sd, int n_dep_var,
                       int n_state_var, int n_rxn,
                       int n_rxn_int_param, int n_rxn_float_param, int n_rxn_env_param,
                       int n_cells) {
  //TODO: Select what % of data we want to compute on GPU simultaneously with CPU remaining %
  //Lengths
  ModelData *md = &(sd->model_data);
  md->state_size = n_state_var * n_cells * sizeof(double);
  md->deriv_size = n_dep_var * n_cells * sizeof(double);
  md->env_size = PMC_NUM_ENV_PARAM_ * n_cells * sizeof(double); //Temp and pressure
  md->rxn_env_data_size = n_rxn_env_param * n_cells * sizeof(double);
  md->rxn_env_data_idx_size = (n_rxn+1) * sizeof(int);
  md->map_state_deriv_size = n_dep_var * n_cells * sizeof(int);
  md->small_data = 0;

  //Allocate streams array and update variables related to streams
  //md->md_id = n_solver_objects;
  //if(n_solver_objects==0){
    //stream_gpu = (hipStream_t *)malloc(n_streams_limit * sizeof(hipStream_t));
      //md->stream_gpu = (hipStream_t *)malloc(n_streams * sizeof(hipStream_t));
  //}
  //n_solver_objects++;

  //Detect if we are working with few data values
  //todo check if it's worth to maintain this case (we will use small_data?)
  if (n_dep_var*n_cells < DATA_SIZE_LIMIT_OPT){
    md->small_data = 0;//1;
  }

  //Set working GPU: we have 4 gpu available on power9. as default, it should be assign to gpu 0
  int device=0;
  hipSetDevice(device);

  //Set GPU properties
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  //Set max threads without triggering too many resources error
  md->max_n_gpu_thread = prop.maxThreadsPerBlock/2;
  md->max_n_gpu_blocks = prop.maxGridSize[1];
  int n_blocks = (n_rxn + md->max_n_gpu_thread - 1) / md->max_n_gpu_thread;

  //GPU allocation
  ModelDataGPU *mGPU = &sd->mGPU;
  HANDLE_ERROR(hipMalloc((void **) &mGPU->deriv_data, md->deriv_size));
  mGPU->n_rxn=md->n_rxn;
  //printf("md->n_rxn %d\n",md->n_rxn);
  mGPU->n_rxn_env_data=md->n_rxn_env_data;
  mGPU->n_aero_phase=md->n_aero_phase;
  mGPU->n_added_aero_phases=md->n_added_aero_phases;
  mGPU->n_aero_rep=md->n_added_aero_reps;
  mGPU->n_aero_rep_env_data=md->n_aero_rep_env_data;

  hipMalloc((void **) &mGPU->state, md->state_size);
  hipMalloc((void **) &mGPU->env, md->env_size);
  hipMalloc((void **) &mGPU->rxn_env_data, md->rxn_env_data_size);
  hipMalloc((void **) &mGPU->rxn_env_data_idx, md->rxn_env_data_idx_size);
  HANDLE_ERROR(hipMalloc((void **) &mGPU->map_state_deriv, md->map_state_deriv_size));


  time_derivative_initialize_gpu(sd);

  //Mapping state-deriv
  md->map_state_deriv = (int *)malloc(md->map_state_deriv_size);
  int i_dep_var = 0;
  for (int i_cell = 0; i_cell < n_cells; i_cell++) {
    for (int i_spec = 0; i_spec < n_state_var; i_spec++) {
      if (md->var_type[i_spec] == CHEM_SPEC_VARIABLE) {
        md->map_state_deriv[i_dep_var] = i_spec + i_cell * n_state_var;
        //printf("%d %d, %d %d %d\n", md->map_state_deriv_size/sizeof(int),
        //       md->map_state_deriv[i_dep_var],n_state_var, i_spec, i_cell, i_dep_var);
        i_dep_var++;
      }
    }
  }

  HANDLE_ERROR(hipMemcpy(mGPU->map_state_deriv, md->map_state_deriv,
                          md->map_state_deriv_size, hipMemcpyHostToDevice));

  //HANDLE_ERROR(hipMemcpy(md->int_pointer_gpu, int_pointer, rxn_int_length*sizeof(int), hipMemcpyHostToDevice));

  //GPU allocation few data on pinned memory
  if(md->small_data){
    //Notice auxiliar variables are created because we
    // can't pin directly variables initialized before
    hipHostMalloc((void**)&md->deriv_aux, md->deriv_size);
  }
  else{
    md->deriv_aux = (realtype *)malloc(md->deriv_size);
  }

  //printf("small_data:%d\n", md->small_data);
  //printf("threads_per_block :%d\n", md->max_n_gpu_thread);

  //GPU create streams
  //for (int i = 0; i < n_streams; ++i)
  //  HANDLE_ERROR( hipStreamCreate(&md->stream_gpu[i]) );

  // Warning if exceeding GPU limits
  if( n_blocks > md->max_n_gpu_blocks){
    printf("\nWarning: More blocks assigned: %d than maximum block numbers: %d",
           n_blocks, md->max_n_gpu_blocks);
  }

  if(n_dep_var<32 && sd->use_cpu==0) {
    printf("CAMP ERROR: TOO FEW SPECIES FOR GPU (Species < 32),"
           " use CPU case instead (More info: https://earth.bsc.es/gitlab/ac/PartMC/-/issues/65 \n");
    exit(0);
  }

#ifdef PMC_DEBUG_PRINT_GPU_SPECS
  print_gpu_specs();
#endif

#ifdef PMC_DEBUG_GPU

  md->timeDerivKernel=0.0;
  hipEventCreate(&md->startDerivKernel);
  hipEventCreate(&md->stopDerivKernel);

#endif

}

/** \brief Set reaction data on GPU prepared structure. RXN data is divided
 * into two different matrix, per double and int data respectively. Matrix are
 * reversed to improve memory access on GPU.
 *
 * \param md Pointer to the model data
 */

void set_reverse_int_double_rxn(
  int n_rxn, int rxn_env_data_idx_size,
  int *rxn_int_data, double *rxn_float_data,
  int *rxn_int_indices, int *rxn_float_indices,
  int *rxn_env_idx,
  SolverData *sd
) {

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

  unsigned int int_max_length = 0;
  unsigned int double_max_length = 0;

  //RXN lengths
  unsigned int int_lengths[n_rxn];
  unsigned int double_lengths[n_rxn];

  //Position on the matrix for each row
  unsigned int rxn_position[n_rxn];

  //Get lengths for int and double arrays
  for (int i_rxn = 0; i_rxn < n_rxn; i_rxn++) {

    //Get RXN lengths
    int_lengths[i_rxn] = rxn_int_indices[i_rxn+1] - rxn_int_indices[i_rxn];
    double_lengths[i_rxn] = rxn_float_indices[i_rxn+1] - rxn_float_indices[i_rxn];

    //Update max size
    if(int_lengths[i_rxn]>int_max_length) int_max_length=int_lengths[i_rxn];
    if(double_lengths[i_rxn]>double_max_length) double_max_length=double_lengths[i_rxn];

    //Set initial position
    rxn_position[i_rxn] = i_rxn;

  }

  //Total lengths of rxn structure
  unsigned int rxn_int_length=n_rxn*int_max_length;
  unsigned int rxn_double_length=n_rxn*double_max_length;

  //Allocate int and double rxn data separately
  //Add -1 to avoid access and have a square matrix
  int *rxn_int = (int *) malloc(rxn_int_length * sizeof(int));
  memset(rxn_int, -1, rxn_int_length * sizeof(int));

  //Add 0 to avoid access and have a square matrix
  double *rxn_double = (double*)calloc(rxn_double_length, sizeof(double));

  int rxn_env_data_idx_aux[n_rxn];

  for (int i_rxn = 0; i_rxn < n_rxn; i_rxn++) {
    int i_pos=rxn_position[i_rxn];//i_rxn;//rxn_position[i_rxn];//for bubblesort
    for (int j = 0; j < int_lengths[i_pos]; j++){
      int *rxn_int_data_aux = &(rxn_int_data[rxn_int_indices[i_pos]]);
      rxn_int[n_rxn*j + i_rxn] = rxn_int_data_aux[j];
    }
    for (int j = 0; j < double_lengths[i_pos]; j++) {
      double *rxn_float_data_aux = &(rxn_float_data[rxn_float_indices[i_pos]]);
      rxn_double[n_rxn*j + i_rxn] = rxn_float_data_aux[j];
    }
    //Reorder the rate indices
    //Todo update on main code the rxn_env_data to read consecutively in cpu
    rxn_env_data_idx_aux[i_rxn] = rxn_env_idx[i_pos];
  }

  //GPU allocation
  hipMalloc((void **) &mGPU->rxn_int, rxn_int_length * sizeof(int));
  hipMalloc((void **) &mGPU->rxn_double, rxn_double_length * sizeof(double));

  //Save data to GPU
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_int, rxn_int, rxn_int_length*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_double, rxn_double, rxn_double_length*sizeof(double), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_env_data_idx, rxn_env_data_idx_aux, rxn_env_data_idx_size, hipMemcpyHostToDevice));

  free(rxn_int);
  free(rxn_double);

}

void set_int_double_rxn(
        int n_rxn, int rxn_env_data_idx_size,
        int *rxn_int_data, double *rxn_float_data,
        int *rxn_int_indices, int *rxn_float_indices,
        int *rxn_env_idx,
        SolverData *sd
) {

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

  //GPU allocation
  hipMalloc((void **) &mGPU->rxn_int, (md->n_rxn_int_param + md->n_rxn)*sizeof(int));
  hipMalloc((void **) &mGPU->rxn_double, md->n_rxn_float_param*sizeof(double));
#ifdef REVERSE_INT_FLOAT_MATRIX
#else
  hipMalloc((void **) &mGPU->rxn_int_indices, (md->n_rxn+1)*sizeof(int));
  hipMalloc((void **) &mGPU->rxn_float_indices, (md->n_rxn+1)*sizeof(int));
#endif

  //Save data to GPU
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_int, rxn_int_data,(md->n_rxn_int_param + md->n_rxn)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_double, rxn_float_data, md->n_rxn_float_param*sizeof(double), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_env_data_idx, rxn_env_idx, rxn_env_data_idx_size, hipMemcpyHostToDevice));
#ifdef REVERSE_INT_FLOAT_MATRIX
#else
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_int_indices, md->rxn_int_indices,(md->n_rxn+1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_float_indices, md->rxn_float_indices,(md->n_rxn+1)*sizeof(int), hipMemcpyHostToDevice));
#endif

}

void set_int_double_aero(
        SolverData *sd
) {

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

  //GPU allocation
  hipMalloc((void **) &mGPU->aero_phase_int_indices, (md->n_aero_phase + 1) * sizeof(int));
  hipMalloc((void **) &mGPU->aero_phase_float_indices, (md->n_aero_phase + 1) * sizeof(int));
  hipMalloc((void **) &mGPU->aero_phase_int_data, md->n_aero_phase_int_param * sizeof(int));
  hipMalloc((void **) &mGPU->aero_phase_float_data, md->n_aero_phase_float_param * sizeof(double));

  hipMalloc((void **) &mGPU->aero_rep_int_indices, (md->n_aero_rep + 1) * sizeof(int));
  hipMalloc((void **) &mGPU->aero_rep_float_indices, (md->n_aero_rep + 1) * sizeof(int));
  hipMalloc((void **) &mGPU->aero_rep_env_idx, (md->n_aero_rep + 1) * sizeof(int));
  hipMalloc((void **) &mGPU->aero_rep_int_data, (md->n_aero_rep_int_param + md->n_aero_rep) * sizeof(int));
  hipMalloc((void **) &mGPU->aero_rep_float_data, md->n_aero_rep_float_param * sizeof(double));
  //hipMalloc((void **) &mGPU->grid_cell_aero_rep_env_data, (md->n_aero_rep_env_data*md->n_cells) * sizeof(double));
  hipMalloc((void **) &mGPU->aero_rep_env_data, (md->n_aero_rep_env_data*md->n_cells) * sizeof(double));

  //Save data to GPU
  HANDLE_ERROR(hipMemcpy(mGPU->aero_phase_int_indices, md->aero_phase_int_indices, (md->n_aero_phase + 1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_phase_float_indices, md->aero_phase_float_indices, (md->n_aero_phase + 1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_phase_int_data, md->aero_phase_int_data, md->n_aero_phase_int_param*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_phase_float_data, md->aero_phase_float_data, md->n_aero_phase_float_param*sizeof(double), hipMemcpyHostToDevice));

  HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_int_indices, md->aero_rep_int_indices, (md->n_aero_rep + 1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_float_indices, md->aero_rep_float_indices, (md->n_aero_rep + 1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_env_idx, md->aero_rep_env_idx, (md->n_aero_rep + 1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_int_data, md->aero_rep_int_data, (md->n_aero_rep_int_param + md->n_aero_rep)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_float_data, md->aero_rep_float_data, md->n_aero_rep_float_param*sizeof(double), hipMemcpyHostToDevice));
  //HANDLE_ERROR(hipMemcpy(mGPU->grid_cell_aero_rep_env_data, md->grid_cell_aero_rep_env_data, (md->n_aero_rep_env_data*md->n_cells)*sizeof(double), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_env_data, md->aero_rep_env_data, (md->n_aero_rep_env_data*md->n_cells)*sizeof(double), hipMemcpyHostToDevice));

}

void solver_init_int_double_gpu(SolverData *sd) {

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

#ifdef REVERSE_INT_FLOAT_MATRIX

  set_reverse_int_double_rxn(
          md->n_rxn, md->rxn_env_data_idx_size,
          md->rxn_int_data, md->rxn_float_data,
          md->rxn_int_indices, md->rxn_float_indices, md->rxn_env_idx,
          sd
  );

#else

  set_int_double_rxn(
          md->n_rxn, md->rxn_env_data_idx_size,
          md->rxn_int_data, md->rxn_float_data,
          md->rxn_int_indices, md->rxn_float_indices, md->rxn_env_idx,
          sd
  );

#endif


/*
  set_int_double_aero(
          sd
  );

 */

}

void init_jac_gpu(SolverData *sd, double *J){

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

  //todo reduce allocations (use tmp pointers from cvode for j_tmp)

  md->jac_size = md->n_per_cell_solver_jac_elem * md->n_cells * sizeof(double);
  md->nnz_J_solver = SM_NNZ_S(md->J_solver);
  md->nrows_J_solver = SM_NP_S(md->J_solver);

  //mGPU->n_per_cell_solver_jac_elem = md->n_per_cell_solver_jac_elem;
  hipMalloc((void **) &mGPU->J, md->jac_size);
  hipMalloc((void **) &mGPU->J_solver, md->jac_size);
  hipMalloc((void **) &mGPU->jJ_solver, md->nnz_J_solver*sizeof(int));
  hipMalloc((void **) &mGPU->iJ_solver, (md->nrows_J_solver+1)*sizeof(int));
  hipMalloc((void **) &mGPU->J_state, md->deriv_size);
  hipMalloc((void **) &mGPU->J_deriv, md->deriv_size);
  hipMalloc((void **) &mGPU->J_tmp, md->deriv_size);
  hipMalloc((void **) &mGPU->J_tmp2, md->deriv_size);
  //JacMap *jac_map_ptr = &mGPU->jac_map;
  //hipMalloc((void **) &jac_map_ptr, sizeof(JacMap)*md->n_mapped_values);
  hipMalloc((void **) &mGPU->jac_map, sizeof(JacMap)*md->n_mapped_values);
  hipMalloc((void **) &mGPU->J_rxn, sizeof(double)*SM_NNZ_S(md->J_rxn)*md->n_cells);//*md->n_mapped_values should be the same
  hipMalloc((void **) &mGPU->n_mapped_values, 1*sizeof(int));//*md->n_mapped_values should be the same

  //printf("md->n_per_cell_dep_var %d sd->jac.num_spec %d md->n_per_cell_solver_jac_elem %d\n",
  //       md->n_per_cell_dep_var,sd->jac.num_spec,md->n_per_cell_solver_jac_elem);

  double *J_solver = SM_DATA_S(md->J_solver);
  //Transfer sunindextype to int
  int *jJ_solver=(int*)malloc(sizeof(int)*md->nnz_J_solver);
  int *iJ_solver=(int*)malloc(sizeof(int)*md->nrows_J_solver+1);
  for(int i=0;i<md->nnz_J_solver;i++)
    jJ_solver[i]=SM_INDEXVALS_S(md->J_solver)[i];
  for(int i=0;i<=md->nrows_J_solver;i++)
    iJ_solver[i]=SM_INDEXPTRS_S(md->J_solver)[i];
  double *J_state = N_VGetArrayPointer(md->J_state);
  double *J_deriv = N_VGetArrayPointer(md->J_deriv);
  double *J_tmp = N_VGetArrayPointer(md->J_tmp);
  double *J_tmp2 = N_VGetArrayPointer(md->J_tmp2);

  HANDLE_ERROR(hipMemcpy(mGPU->J, J, md->jac_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_solver, J_solver, md->jac_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->jJ_solver, jJ_solver, md->nnz_J_solver*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->iJ_solver, iJ_solver, (md->nrows_J_solver+1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_state, J_state, md->deriv_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_deriv, J_deriv, md->deriv_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_tmp, J_tmp, md->deriv_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemset(mGPU->J_tmp2, 0.0, md->deriv_size));
  HANDLE_ERROR(hipMemcpy(mGPU->jac_map, md->jac_map, sizeof(JacMap)*md->n_mapped_values, hipMemcpyHostToDevice));
  double *J_data = SM_DATA_S(md->J_rxn);
  HANDLE_ERROR(hipMemcpy(mGPU->J_rxn, J_data, sizeof(double)*SM_NNZ_S(md->J_rxn)*md->n_cells, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->n_mapped_values, &md->n_mapped_values, 1*sizeof(int), hipMemcpyHostToDevice));

  jacobian_initialize_gpu(sd);

  if(md->small_data){
    hipHostMalloc((void**)&md->jac_aux, md->jac_size);
  }

}

void set_jac_data_gpu(SolverData *sd, double *J){

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

  double *J_solver = SM_DATA_S(md->J_solver);
  double *J_state = N_VGetArrayPointer(md->J_state);
  double *J_deriv = N_VGetArrayPointer(md->J_deriv);
  HANDLE_ERROR(hipMemcpy(mGPU->J, J, md->jac_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_solver, J_solver, md->jac_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_state, J_state, md->deriv_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->J_deriv, J_deriv, md->deriv_size, hipMemcpyHostToDevice));

  //HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_float_data, md->aero_rep_float_data, md->n_aero_rep_float_param*sizeof(double), hipMemcpyHostToDevice));

  itsolver *bicg = &(sd->bicg);
  hipMemcpy(mGPU->djA,bicg->jA,mGPU->nnz*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(mGPU->diA,bicg->iA,(mGPU->nrows+1)*sizeof(int),hipMemcpyHostToDevice);

}

void update_aero_contrib_gpu(SolverData *sd){

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

  HANDLE_ERROR(hipMemcpy(mGPU->state, md->total_state, md->state_size, hipMemcpyHostToDevice));
  //HANDLE_ERROR(hipMemcpy(mGPU->aero_rep_float_data, md->aero_rep_float_data, md->n_aero_rep_float_param*sizeof(double), hipMemcpyHostToDevice));

  int num_spec = md->n_per_cell_dep_var*md->n_cells;
  HANDLE_ERROR(hipMemcpy(mGPU->production_rates, sd->time_deriv.production_rates, num_spec*sizeof(mGPU->production_rates), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->loss_rates, sd->time_deriv.loss_rates, num_spec*sizeof(mGPU->loss_rates), hipMemcpyHostToDevice));

}


void rxn_update_env_state_gpu(SolverData *sd){

  ModelData *md = &(sd->model_data);
  int n_cells = md->n_cells;
  int n_rxn = md->n_rxn;
  int n_threads = n_rxn*n_cells; //Reaction group per number of repetitions/cells
  double *rxn_env_data = md->rxn_env_data;
  double *env = md->total_env;
  int n_blocks = ((n_threads + md->max_n_gpu_thread - 1) / md->max_n_gpu_thread);
  ModelDataGPU *mGPU = &sd->mGPU;

  //Faster, use for few values
  if (md->small_data){
    //This method of passing them as a function parameter has a theoric maximum of 4kb of data
    mGPU->rxn_env_data = rxn_env_data;
    mGPU->env = env;
  }
  //Slower, use for large values
  else{
    //Async memcpy
    //HANDLE_ERROR(hipMemcpyAsync(md->rxn_env_data_gpu, rxn_env_data,
    //        md->rxn_env_data_size, hipMemcpyHostToDevice, md->stream_gpu[STREAM_RXN_ENV_GPU]));
    //HANDLE_ERROR(hipMemcpyAsync(md->env_gpu, env, md->env_size,
    //        hipMemcpyHostToDevice, md->stream_gpu[STREAM_ENV_GPU]));


    HANDLE_ERROR(hipMemcpy(mGPU->rxn_env_data, rxn_env_data, md->rxn_env_data_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(mGPU->env, env, md->env_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(mGPU->state, md->total_state, md->state_size, hipMemcpyHostToDevice));

  }
}

__device__
void cudaDevicecamp_solver_check_model_state0(double *state, double *y,
                                        int *map_state_deriv, double threshhold, double replacement_value, int *flag,
                                        int deriv_length_cell, int n_cells)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int active_threads = n_cells*deriv_length_cell;
  //extern __shared__ int flag_shr[];
  __shared__ int flag_shr[1];
  flag_shr[0] = 0;

  if(tid<active_threads) {

    if (y[tid] < threshhold) {

      //*flag = CAMP_SOLVER_FAIL;
      flag_shr[0] = CAMP_SOLVER_FAIL;
#ifdef FAILURE_DETAIL
      printf("\nFailed model state update gpu (Negative value on 'y'):[spec %d] = %le",tid,y[tid]);
#endif

    } else {
      state[map_state_deriv[tid]] =
              y[tid] <= threshhold ?
              replacement_value : y[tid];

      //state_init[map_state_deriv[tid]] = 0.1;
      //printf("tid %d map_state_deriv %d\n", tid, map_state_deriv[tid]);

    }

    /*
    if (y[tid] > -SMALL) {
      state_init[map_state_deriv[tid]] =
      y[tid] > threshhold ?
      y[tid] : replacement_value;

      //state_init[map_state_deriv[tid]] = 0.1;
      //printf("tid %d map_state_deriv %d\n", tid, map_state_deriv[tid]);
    } else {
      *status = CAMP_SOLVER_FAIL;
#ifdef FAILURE_DETAIL
      printf("\nFailed model state update gpu (Negative value on 'y'):[spec %d] = %le",tid,y[tid]);
#endif
    }
     */
  }

  __syncthreads();
  *flag = flag_shr[0];
  return;

}

__global__
void camp_solver_check_model_state_cuda(double *state_init, double *y,
        int *map_state_deriv, double threshhold, double replacement_value, int *flag,
        int deriv_length_cell, int n_cells)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int active_threads = n_cells*deriv_length_cell;

  if(tid<active_threads) {

    if (y[tid] < threshhold) {

      *flag = CAMP_SOLVER_FAIL;
#ifdef FAILURE_DETAIL
      printf("\nFailed model state update gpu (Negative value on 'y'):[spec %d] = %le",tid,y[tid]);
#endif

    } else {
      state_init[map_state_deriv[tid]] =
              y[tid] <= threshhold ?
              replacement_value : y[tid];

      //state_init[map_state_deriv[tid]] = 0.1;
      //printf("tid %d map_state_deriv %d\n", tid, map_state_deriv[tid]);

    }

    /*
    if (y[tid] > -SMALL) {
      state_init[map_state_deriv[tid]] =
      y[tid] > threshhold ?
      y[tid] : replacement_value;

      //state_init[map_state_deriv[tid]] = 0.1;
      //printf("tid %d map_state_deriv %d\n", tid, map_state_deriv[tid]);
    } else {
      *status = CAMP_SOLVER_FAIL;
#ifdef FAILURE_DETAIL
      printf("\nFailed model state update gpu (Negative value on 'y'):[spec %d] = %le",tid,y[tid]);
#endif
    }
     */
  }

}

int camp_solver_check_model_state_gpu(N_Vector solver_state, SolverData *sd,
                                      double threshhold0, double replacement_value0)
{
  ModelData *md = &(sd->model_data);
  itsolver *bicg = &(sd->bicg);
  int flag = CAMP_SOLVER_SUCCESS; //0
  int n_cells = md->n_cells;
  int n_state_var = md->n_per_cell_state_var;
  int n_dep_var = md->n_per_cell_dep_var;
  int n_threads = n_dep_var*n_cells;
  int n_blocks = ((n_threads + md->max_n_gpu_thread - 1) / md->max_n_gpu_thread);
  int *var_type = md->var_type;
  double *state = md->total_state;
  double *y = NV_DATA_S(solver_state);
  ModelDataGPU *mGPU = &sd->mGPU;

/*
  //HANDLE_ERROR(hipMemcpy(md->deriv_aux, mGPU->dcv_y, md->deriv_size, hipMemcpyDeviceToHost));
  if(sd->counterDerivCPU<=5){
    printf("counterDeriv2 %d \n", sd->counterDerivCPU);
    for (int i = 0; i < NV_LENGTH_S(solver_state); i++) {
        //printf("(%d) %-le ", i + 1, NV_DATA_S(deriv)[i]);
      if(y[i]!=md->deriv_aux[i]) {
        printf("(%d) dy %-le y %-le\n", i + 1, md->deriv_aux[i], y[i]);
      }
    }
  }
*/

  double replacement_value = TINY;
  double threshhold = -SMALL;

  camp_solver_check_model_state_cuda << < n_blocks, md->max_n_gpu_thread >> >
   (mGPU->state, mGPU->dcv_y, mGPU->map_state_deriv,
   threshhold, replacement_value, &flag, n_dep_var, n_cells);

  HANDLE_ERROR(hipMemcpy(md->total_state, mGPU->state, md->state_size, hipMemcpyDeviceToHost));


#ifdef DEBUG_CHECK_MODEL_STATE_CUDA
  for (int i_cell = 0; i_cell < n_cells; i_cell++) {
   for (int i_dep_var = 0; i_dep_var < n_dep_var; i_dep_var++) {

     printf("(%d) %-le \n", i_dep_var+1,
            md->total_state[md->map_state_deriv[i_dep_var]]);
   }
}
#endif


  //printf("camp_solver_check_model_state_gpu flag %d\n",flag);

  return flag;
}


void camp_solver_update_model_state_gpu(N_Vector solver_state, SolverData *sd,
                                       double threshhold, double replacement_value)
{
  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;
  HANDLE_ERROR(hipMemcpy(mGPU->state, md->total_state, md->state_size, hipMemcpyHostToDevice));

}

__device__ void solveRXN0(
#ifdef BASIC_CALC_DERIV
        double *deriv_data,
#else
        TimeDerivativeGPU deriv_data,
#endif
       double time_step,
       ModelDataGPU *md
)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

#ifdef REVERSE_INT_FLOAT_MATRIX

  double *rxn_float_data = &( md->rxn_double[md->i_rxn]);
  int *int_data = &(md->rxn_int[md->i_rxn]);
  int rxn_type = int_data[0];
  int *rxn_int_data = (int *) &(int_data[1*md->n_rxn]);

#else

  double *rxn_float_data = (double *)&( md->rxn_double[md->rxn_float_indices[md->i_rxn]]);
  int *int_data = (int *)&(md->rxn_int[md->rxn_int_indices[md->i_rxn]]);

  //double *rxn_float_data = &( md->rxn_double[md->i_rxn]);
  //int *int_data = &(md->rxn_int[md->i_rxn]);


  int rxn_type = int_data[0];
  int *rxn_int_data = (int *) &(int_data[1]);

#endif

  //Get indices for rates
  double *rxn_env_data = &(md->rxn_env_data
  [md->n_rxn_env_data*md->i_cell+md->rxn_env_data_idx[md->i_rxn]]);

#ifdef DEBUG_DERIV_GPU
  if(tid==0){
    printf("[DEBUG] GPU solveRXN tid %d, \n", tid);
  }
#endif

  switch (rxn_type) {
    //case RXN_AQUEOUS_EQUILIBRIUM :
    //fix run-time error
    //rxn_gpu_aqueous_equilibrium_calc_deriv_contrib(md, deriv_data, rxn_int_data,
    //                                               rxn_float_data, rxn_env_data,time_step);
    //break;
    case RXN_ARRHENIUS :
      rxn_gpu_arrhenius_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                           rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CMAQ_H2O2 :
      rxn_gpu_CMAQ_H2O2_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                          rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CMAQ_OH_HNO3 :
      rxn_gpu_CMAQ_OH_HNO3_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                           rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CONDENSED_PHASE_ARRHENIUS :
      //rxn_gpu_condensed_phase_arrhenius_calc_deriv_contrib(md, deriv_data, rxn_int_data,
      //                                     rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_EMISSION :
      printf("RXN_EMISSION");
      //rxn_gpu_emission_calc_deriv_contrib(md, deriv_data, rxn_int_data,
      //                                     rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_FIRST_ORDER_LOSS :
      //rxn_gpu_first_order_loss_calc_deriv_contrib(md, deriv_data, rxn_int_data,
      //                                     rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_HL_PHASE_TRANSFER :
      //rxn_gpu_HL_phase_transfer_calc_deriv_contrib(md, deriv_data, rxn_int_data,
      //                                             rxn_float_data, rxn_env_data,time_stepn);
      break;
    case RXN_PHOTOLYSIS :
      rxn_gpu_photolysis_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                           rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_SIMPOL_PHASE_TRANSFER :
      //rxn_gpu_SIMPOL_phase_transfer_calc_deriv_contrib(md, deriv_data,
      //        rxn_int_data, rxn_float_data, rxn_env_data, time_step);
      break;
    case RXN_TROE :
#ifdef BASIC_CALC_DERIV
#else
      rxn_gpu_troe_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                      rxn_float_data, rxn_env_data,time_step);
#endif
      break;
    case RXN_WET_DEPOSITION :
      printf("RXN_WET_DEPOSITION");
      //rxn_gpu_wet_deposition_calc_deriv_contrib(md, deriv_data, rxn_int_data,
      //                                     rxn_float_data, rxn_env_data,time_step);
      break;
  }
/*
*/

}

__device__ void cudaDevicecalc_deriv0(
#ifdef PMC_DEBUG_GPU
        int counterDeriv2,
#endif
        //check_model_state
        //double threshhold, double replacement_value, int *flag,
        //f_gpu
        double time_step, int deriv_length_cell, int state_size_cell,
        int n_cells,
        int i_kernel, int threads_block, int n_shr_empty, double *y,
        ModelDataGPU md_object
) //Interface CPU/GPU
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int tid_cell=tid%deriv_length_cell;
  int active_threads = n_cells*deriv_length_cell;
  ModelDataGPU *md = &md_object;

#ifdef DEBUG_DERIV_GPU
  if(tid==0){
    printf("[DEBUG] GPU solveDerivative tid %d, \n", tid);
  }__syncthreads();
#endif

  if(tid<active_threads){

#ifdef DEBUG_solveDerivative_J_DERIV_IN_CPU
#else

    //N_VLinearSum(1.0, y, -1.0, md->J_state, md->J_tmp);
  cudaDevicezaxpby(1.0, y, -1.0, md->J_state, md->J_tmp, active_threads);
  //SUNMatMatvec(md->J_solver, md->J_tmp, md->J_tmp2);
  cudaDeviceSpmvCSC_block(md->J_tmp2, md->J_tmp, active_threads, md->J_solver, md->jJ_solver, md->iJ_solver, 0);
  //N_VLinearSum(1.0, md->J_deriv, 1.0, md->J_tmp2, md->J_tmp);
  cudaDevicezaxpby(1.0, md->J_deriv, 1.0, md->J_tmp2, md->J_tmp, active_threads);
  cudaDevicesetconst(md->J_tmp2, 0.0, active_threads); //Reset for next iter


#endif

    //Debug
    /*
    if(counterDeriv2<=1){
      printf("(%d) y %-le J_state %-le J_solver %-le J_tmp %-le J_tmp2 %-le J_deriv %-le\n",tid+1,
             y[tid], md->J_state[tid], md->J_solver[tid], md->J_tmp[tid], md->J_tmp2[tid], md->J_deriv[tid]);
      //printf("gpu threads %d\n", active_threads);
    }
*/

#ifdef BASIC_CALC_DERIV
    md->i_rxn=tid%n_rxn;
    double *deriv_init = md->deriv_data;
    md->deriv_data = &( md->deriv_init[deriv_length_cell*md->i_cell]);
    if(tid < n_rxn*n_cells){
        solveRXN(deriv_data, time_step, md);
    }
#else
    TimeDerivativeGPU deriv_data;
    deriv_data.num_spec = deriv_length_cell*n_cells;

#ifdef AEROS_CPU
#else
    deriv_data.production_rates = md->production_rates;
    deriv_data.loss_rates = md->loss_rates;
    time_derivative_reset_gpu(deriv_data);
    __syncthreads();
#endif

    int i_cell = tid/deriv_length_cell;
    md->i_cell = i_cell;
    deriv_data.production_rates = &( md->production_rates[deriv_length_cell*i_cell]);
    deriv_data.loss_rates = &( md->loss_rates[deriv_length_cell*i_cell]);

    md->grid_cell_state = &( md->state[state_size_cell*i_cell]);
    md->grid_cell_env = &( md->env[PMC_NUM_ENV_PARAM_*i_cell]);

    /*
    md->grid_cell_aero_rep_env_data =
    &(md->aero_rep_env_data[md->i_cell*md->n_aero_rep_env_data]);

    //Filter threads for n_aero_rep
    int n_aero_rep = md->n_aero_rep;
    if( tid_cell < n_aero_rep) {
      int n_iters = n_aero_rep / deriv_length_cell;
      //Repeat if there are more reactions than species
      for (int i = 0; i < n_iters; i++) {
        md->i_aero_rep = tid_cell + i*deriv_length_cell;

        aero_rep_gpu_update_state(md);
      }

      //Limit tid to pending rxns to compute
      int residual=n_aero_rep-(deriv_length_cell*n_iters);
      if(tid_cell < residual){
        md->i_aero_rep = tid_cell + deriv_length_cell*n_iters;

        aero_rep_gpu_update_state(md);
      }
    }
     */

    //Filter threads for n_rxn
    int n_rxn = md->n_rxn;
    if( tid_cell < n_rxn) {
      int n_iters = n_rxn / deriv_length_cell;
      //Repeat if there are more reactions than species
      for (int i = 0; i < n_iters; i++) {
        md->i_rxn = tid_cell + i*deriv_length_cell;

        solveRXN0(deriv_data, time_step, md);
      }

      //Limit tid to pending rxns to compute
      int residual=n_rxn-(deriv_length_cell*n_iters);
      if(tid_cell < residual){
        md->i_rxn = tid_cell + deriv_length_cell*n_iters;

        solveRXN0(deriv_data, time_step, md);
      }
    }
    __syncthreads();

    /*if(tid==0){
      printf("tid %d time_deriv.production_rates %-le time_deriv.loss_rates %-le\n",
              tid, deriv_data.production_rates[tid],
             deriv_data.loss_rates[tid]);
    }*/

    deriv_data.production_rates = md->production_rates;
    deriv_data.loss_rates = md->loss_rates;
    __syncthreads();
    time_derivative_output_gpu(deriv_data, md->deriv_data, md->J_tmp,0);
#endif

    /*
    if(tid<deriv_data.num_spec && tid>1022){
      //if(tid<1){
      //deriv_init[tid] = deriv_data.production_rates[tid];
      //deriv_init[tid] = deriv_data.loss_rates[tid];
      printf("tid %d time_deriv.production_rates %-le time_deriv.loss_rates %-le"
             "deriv_init %-le\n",
             tid, deriv_data.production_rates[tid],
             deriv_data.loss_rates[tid],
             //deriv_data.loss_rates[tid]);
             deriv_init[tid]);
    }*/

  }

}

__device__
void cudaDevicef0(
#ifdef PMC_DEBUG_GPU
        int counterDeriv2,
#endif
        //check_model_state
        double threshhold, double replacement_value, int *flag,
        //f_gpu
        double time_step, int deriv_length_cell, int state_size_cell,
        int n_cells,
        int i_kernel, int threads_block, int n_shr_empty, double *y,
        ModelDataGPU md_object
) //Interface CPU/GPU
{

  ModelDataGPU *md = &md_object;

  cudaDevicecamp_solver_check_model_state0(md->state, y,
                                          md->map_state_deriv, threshhold, replacement_value,
                                          flag, deriv_length_cell, n_cells);

  //__syncthreads();
  //study flag block effect: flag is global for all threads or for only the block?
  if(*flag==CAMP_SOLVER_FAIL)
    return;

  cudaDevicecalc_deriv0(
#ifdef PMC_DEBUG_GPU
           counterDeriv2,
#endif
        //check_model_state          md->map_state_deriv, threshhold, replacement_value, flag,
          //f_gpu
        time_step, deriv_length_cell, state_size_cell,
           n_cells, i_kernel, threads_block, n_shr_empty, y,
           md_object
          );
}

__global__
void cudaGlobalf(
#ifdef PMC_DEBUG_GPU
        int counterDeriv2,
#endif
        //check_model_state
        double threshhold, double replacement_value, int *flag,
        //f_gpu
        double time_step, int deriv_length_cell, int state_size_cell,
        int n_cells,
        int i_kernel, int threads_block, int n_shr_empty, double *y,
        ModelDataGPU md_object
) //Interface CPU/GPU
{

  cudaDevicef0(
#ifdef PMC_DEBUG_GPU
          counterDeriv2,
#endif
          //check_model_state
                threshhold, replacement_value, flag,
                //f_gpu
          time_step, deriv_length_cell, state_size_cell,
          n_cells, i_kernel, threads_block, n_shr_empty, y,
          md_object
  );
}



/** Old routine
 */
__global__ void solveDerivative(
#ifdef PMC_DEBUG_GPU
        int counterDeriv2,
#endif
        double time_step, int deriv_length_cell, int state_size_cell,
        int n_cells,
        int i_kernel, int threads_block, int n_shr_empty, double *y,
        double threshhold, double replacement_value, ModelDataGPU md_object
) //Interface CPU/GPU
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int tid_cell=tid%deriv_length_cell;
  int active_threads = n_cells*deriv_length_cell;
  ModelDataGPU *md = &md_object;

#ifdef DEBUG_DERIV_GPU
  if(tid==0){
    printf("[DEBUG] GPU solveDerivative tid %d, \n", tid);
  }__syncthreads();
#endif

  if(tid<active_threads){

#ifdef DEBUG_solveDerivative_J_DERIV_IN_CPU
#else

    //N_VLinearSum(1.0, y, -1.0, md->J_state, md->J_tmp);
    cudaDevicezaxpby(1.0, y, -1.0, md->J_state, md->J_tmp, active_threads);
    //SUNMatMatvec(md->J_solver, md->J_tmp, md->J_tmp2);
    cudaDeviceSpmvCSC_block(md->J_tmp2, md->J_tmp, active_threads, md->J_solver, md->jJ_solver, md->iJ_solver, 0);
    //N_VLinearSum(1.0, md->J_deriv, 1.0, md->J_tmp2, md->J_tmp);
    cudaDevicezaxpby(1.0, md->J_deriv, 1.0, md->J_tmp2, md->J_tmp, active_threads);
    cudaDevicesetconst(md->J_tmp2, 0.0, active_threads); //Reset for next iter


#endif

    //Debug
    //printf("HOLA\n");
    /*
    if(counterDeriv2<=1){
      printf("(%d) y %-le J_state %-le J_solver %-le J_tmp %-le J_tmp2 %-le J_deriv %-le\n",tid+1,
             y[tid], md->J_state[tid], md->J_solver[tid], md->J_tmp[tid], md->J_tmp2[tid], md->J_deriv[tid]);
      //printf("gpu threads %d\n", active_threads);
    }
*/

#ifdef BASIC_CALC_DERIV
    md->i_rxn=tid%n_rxn;
    double *deriv_init = md->deriv_data;
    md->deriv_data = &( md->deriv_init[deriv_length_cell*md->i_cell]);
    if(tid < n_rxn*n_cells){
        solveRXN0(deriv_data, time_step, md);
    }
#else
    TimeDerivativeGPU deriv_data;
    deriv_data.num_spec = deriv_length_cell*n_cells;

#ifdef AEROS_CPU
#else
    deriv_data.production_rates = md->production_rates;
    deriv_data.loss_rates = md->loss_rates;
    time_derivative_reset_gpu(deriv_data);
    __syncthreads();
#endif

    int i_cell = tid/deriv_length_cell;
    md->i_cell = i_cell;
    deriv_data.production_rates = &( md->production_rates[deriv_length_cell*i_cell]);
    deriv_data.loss_rates = &( md->loss_rates[deriv_length_cell*i_cell]);

    md->grid_cell_state = &( md->state[state_size_cell*i_cell]);
    md->grid_cell_env = &( md->env[PMC_NUM_ENV_PARAM_*i_cell]);

    //Filter threads for n_rxn
    int n_rxn = md->n_rxn;
    if( tid_cell < n_rxn) {
      int n_iters = n_rxn / deriv_length_cell;
      //Repeat if there are more reactions than species
      for (int i = 0; i < n_iters; i++) {
        md->i_rxn = tid_cell + i*deriv_length_cell;

        solveRXN0(deriv_data, time_step, md);
      }

      //Limit tid to pending rxns to compute
      int residual=n_rxn-(deriv_length_cell*n_iters);
      if(tid_cell < residual){
        md->i_rxn = tid_cell + deriv_length_cell*n_iters;

        solveRXN0(deriv_data, time_step, md);
      }
    }
    __syncthreads();

    /*if(tid==0){
      printf("tid %d time_deriv.production_rates %-le time_deriv.loss_rates %-le\n",
              tid, deriv_data.production_rates[tid],
             deriv_data.loss_rates[tid]);
    }*/

    deriv_data.production_rates = md->production_rates;
    deriv_data.loss_rates = md->loss_rates;
    __syncthreads();
    time_derivative_output_gpu(deriv_data, md->deriv_data, md->J_tmp,0);
#endif

    /*
    if(tid<deriv_data.num_spec && tid>1022){
      //if(tid<1){
      //deriv_init[tid] = deriv_data.production_rates[tid];
      //deriv_init[tid] = deriv_data.loss_rates[tid];
      printf("tid %d time_deriv.production_rates %-le time_deriv.loss_rates %-le"
             "deriv_init %-le\n",
             tid, deriv_data.production_rates[tid],
             deriv_data.loss_rates[tid],
             //deriv_data.loss_rates[tid]);
             deriv_init[tid]);
    }*/

  }

}


/** \brief Calculate the time derivative \f$f(t,y)\f$ on GPU
 *
 * \param md Pointer to the model data
 * \param deriv NVector to hold the calculated vector
 * \param time_step Current model time step (s)
 */
int rxn_calc_deriv_gpu(SolverData *sd, N_Vector y, N_Vector deriv, double time_step) {

  ModelData *md = &(sd->model_data);
  itsolver *bicg = &(sd->bicg);
  double *deriv_data = N_VGetArrayPointer(deriv);
  int n_cells = md->n_cells;
  int n_kernels = 1; // Divide load into multiple kernel calls
  //todo n_kernels case division left residual, an extra kernel computes remain residual
#ifdef BASIC_CALC_DERIV
  int total_threads = md->n_rxn*n_cells/n_kernels; //Reaction group per number of repetitions/cells
  int threads_block = md->max_n_gpu_thread;
#else
  int n_per_cell_dep_var = md->n_per_cell_dep_var;
  int total_threads = n_per_cell_dep_var * n_cells/n_kernels;
  int n_shr_empty = md->max_n_gpu_thread%n_per_cell_dep_var;
  int threads_block = md->max_n_gpu_thread - n_shr_empty; //last multiple of size_cell before max_threads
#endif
  int n_blocks = ((total_threads + threads_block - 1) / threads_block);
  double *J_tmp = N_VGetArrayPointer(md->J_tmp);
  ModelDataGPU *mGPU = &sd->mGPU;
  //Update state
  double replacement_value = TINY;
  double threshhold = -SMALL;
  int flag = CAMP_SOLVER_SUCCESS; //0

#ifdef DERIV_CPU_ON_GPU

  //Transfer cv_ftemp() not needed because mGPU->dftemp=md->deriv_data_gpu;
  //hipMemcpy(cv_ftemp_data,mGPU->dftemp,mGPU->nrows*sizeof(double),hipMemcpyDeviceToHost);

  HANDLE_ERROR(hipMemcpy(mGPU->deriv_data, deriv_data, md->deriv_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->state, md->total_state, md->state_size, hipMemcpyHostToDevice));

#else

#ifdef DEBUG_rxn_calc_deriv_gpu

  printf("rxn_calc_deriv_gpu start\n");

#endif

  if (camp_solver_check_model_state_gpu(y, sd, -SMALL, TINY) != CAMP_SOLVER_SUCCESS)
    return 1;

 //debug
 /*
  if(sd->counterDerivGPU<=0){
    printf("f_gpu start total_state [(id),conc], n_state_var %d, n_cells %d\n", md->n_per_cell_state_var, n_cells);
    printf("n_deriv %d\n", md->n_per_cell_dep_var);
    for (int i = 0; i < md->n_per_cell_state_var*n_cells; i++) {
      printf("(%d) %-le \n",i+1, md->total_state[i]);
    }
  }
  */

#ifdef BASIC_CALC_DERIV
  //Reset deriv gpu
  //check if cudamemset work fine with doubles
  HANDLE_ERROR(hipMemset(md->deriv_data_gpu, 0.0, md->deriv_size));
#endif

#ifdef PMC_DEBUG_GPU
  //timeDerivSend += (clock() - t1);
  //clock_t t2 = clock();

  hipEventRecord(md->startDerivKernel);

#endif

#ifdef AEROS_CPU

  update_aero_contrib_gpu(sd);

  //printf("hola");

#endif

#ifdef DEBUG_solveDerivative_J_DERIV_IN_CPU

/*
  if(sd->counterDerivGPU<=1 ){
    printf("f_gpu start J_TMP [(id),conc], n_state_var %d, n_cells %d\n", md->n_per_cell_state_var, n_cells);
    int size_j = NV_LENGTH_S(deriv);
    printf("length_deriv %d \n", size_j);
    for (int i = 0; i < 1; i++) {//n_cells
      printf("cell %d \n", i);
      for (int j = 0; j < size_j; j++) {  // NV_LENGTH_S(deriv)
        printf("(%d) %-le ", j + 1, NV_DATA_S(md->J_tmp)[j+i*size_j]);
      }
      printf("\n");
    }
  }*/

  HANDLE_ERROR(hipMemcpy(mGPU->J_tmp, J_tmp, md->deriv_size, hipMemcpyHostToDevice));

#endif

  //Loop to test multiple kernel executions
  for (int i_kernel=0; i_kernel<n_kernels; i_kernel++){
    //hipDeviceSynchronize();
    //solveDerivative << < (n_blocks), threads_block >> >(
    cudaGlobalf << < (n_blocks), threads_block >> >(
#ifdef PMC_DEBUG_GPU
    sd->counterDerivGPU,
#endif
    //update_state
    threshhold, replacement_value, &flag,
     //f_gpu
    time_step, md->n_per_cell_dep_var,
     md->n_per_cell_state_var,n_cells,
     i_kernel, threads_block,n_shr_empty, mGPU->dcv_y,
     sd->mGPU
     );
  }

  if(flag==CAMP_SOLVER_FAIL)
    return flag;

#ifdef PMC_DEBUG_GPU
  /*hipDeviceSynchronize();
  timeDerivKernel += (clock() - t2);
  t3 = clock();*/


  hipEventRecord(md->stopDerivKernel);
  hipEventSynchronize(md->stopDerivKernel);
  float msDerivKernel = 0.0;
  hipEventElapsedTime(&msDerivKernel, md->startDerivKernel, md->stopDerivKernel);
  md->timeDerivKernel+= msDerivKernel;


#endif

  //Use pinned memory for few values
  if (md->small_data){

    HANDLE_ERROR(hipMemcpy(md->deriv_aux, mGPU->deriv_data, md->deriv_size, hipMemcpyDeviceToHost));

    memcpy(deriv_data, md->deriv_aux, md->deriv_size);
  }
  else {
    //Async
    //HANDLE_ERROR(hipMemcpyAsync(md->deriv_aux, md->deriv_data_gpu,
    //md->deriv_size, hipMemcpyDeviceToHost, md->stream_gpu[STREAM_DERIV_GPU]));

    //Sync
    //HANDLE_ERROR(hipMemcpy(md->deriv_aux, md->deriv_data_gpu, md->deriv_size, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipMemcpy(deriv_data, mGPU->deriv_data, md->deriv_size, hipMemcpyDeviceToHost));


    //HANDLE_ERROR(hipMemcpy(mGPU->deriv_data, deriv_data, md->deriv_size, hipMemcpyHostToDevice));
    //HANDLE_ERROR(hipMemcpy(mGPU->dcv_y, deriv_data, md->deriv_size, hipMemcpyHostToDevice));
    //HANDLE_ERROR(hipMemcpy(mGPU->state, md->total_state, md->state_size, hipMemcpyHostToDevice));


  }

  //hipDeviceSynchronize();

 //debug
 /*
  if(sd->counterDerivGPU<=0 ){
    printf("f_gpu end deriv [(id),conc], n_state_var %d, n_cells %d\n", md->n_per_cell_state_var, n_cells);
    int size_j = NV_LENGTH_S(deriv);
    printf("length_deriv %d \n", size_j);
    for (int i = 0; i < 1; i++) {//n_cells
      printf("cell %d \n", i);
      for (int j = 0; j < size_j; j++) {  // NV_LENGTH_S(deriv)
        printf("(%d) %-le ", j + 1, NV_DATA_S(deriv)[j+i*size_j]);
      }
      printf("\n");
    }
  }
*/

#ifdef PMC_DEBUG_GPU
  /*timeDerivReceive += (clock() - t3);
  timeDeriv += (clock() - t1);
  t3 = clock();*/
#endif

#endif

  return 0;
}

void get_f_from_gpu(SolverData *sd){

  //HANDLE_ERROR(hipMemcpy(mGPU->state, J, md->jac_size, hipMemcpyHostToDevice));

}

void get_guess_helper_from_gpu(N_Vector y_n, N_Vector y_n1,
        N_Vector hf, void *solver_data, N_Vector tmp1,
        N_Vector corr){

  //HANDLE_ERROR(hipMemcpy(mGPU->state, J, md->jac_size, hipMemcpyHostToDevice));


}

/** \brief Fusion deriv data calculated from CPU and GPU
 * (Calculations from CPU & GPU or GPU async case)
 *
 * \param md Pointer to the model data
 * \param deriv NVector to hold the calculated vector
 * \param time_step Current model time step (s)
 */
void rxn_fusion_deriv_gpu(ModelData *md, N_Vector deriv) {

  // Get a pointer to the derivative data
  realtype *deriv_data = N_VGetArrayPointer(deriv);

  hipDeviceSynchronize();
  //HANDLE_ERROR(hipMemsetAsync(md->deriv_data_gpu, 0.0,
  //        md->deriv_size, md->stream_gpu[STREAM_DERIV_GPU]));

  if (md->small_data){
  }
  else {
    for (int i = 0; i < NV_LENGTH_S(deriv); i++) {  // NV_LENGTH_S(deriv)
      //Add to deriv the auxiliar contributions from gpu
      deriv_data[i] += md->deriv_aux[i];
    }
  }

}






__device__ void solveRXNJac0(
        JacobianGPU jac,
        double time_step,
        ModelDataGPU *md
)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

#ifdef REVERSE_INT_FLOAT_MATRIX

  double *rxn_float_data = &( md->rxn_double[md->i_rxn]);
  int *int_data = &(md->rxn_int[md->i_rxn]);
  int rxn_type = int_data[0];
  int *rxn_int_data = (int *) &(int_data[1*md->n_rxn]);

#else

  double *rxn_float_data = (double *)&( md->rxn_double[md->rxn_float_indices[md->i_rxn]]);
  int *int_data = (int *)&(md->rxn_int[md->rxn_int_indices[md->i_rxn]]);

  //double *rxn_float_data = &( md->rxn_double[md->i_rxn]);
  //int *int_data = &(md->rxn_int[md->i_rxn]);

  int rxn_type = int_data[0];
  int *rxn_int_data = (int *) &(int_data[1]);

#endif

  //Get indices for rates
  double *rxn_env_data = &(md->rxn_env_data
  [md->n_rxn_env_data*md->i_cell+md->rxn_env_data_idx[md->i_rxn]]);

#ifdef DEBUG_solveRXNJac0
  if(tid==0){
    printf("[DEBUG] GPU solveRXN tid %d, \n", tid);
  }
#endif

  switch (rxn_type) {
    //case RXN_AQUEOUS_EQUILIBRIUM :
    //fix run-time error
    //rxn_gpu_aqueous_equilibrium_calc_jac_contrib(md, jac, rxn_int_data,
    //                                               rxn_float_data, rxn_env_data,time_step);
    //break;
    case RXN_ARRHENIUS :
      rxn_gpu_arrhenius_calc_jac_contrib(md, jac, rxn_int_data,
                                           rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CMAQ_H2O2 :
      rxn_gpu_CMAQ_H2O2_calc_jac_contrib(md, jac, rxn_int_data,
                                           rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CMAQ_OH_HNO3 :
      rxn_gpu_CMAQ_OH_HNO3_calc_jac_contrib(md, jac, rxn_int_data,
                                              rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CONDENSED_PHASE_ARRHENIUS :
      //rxn_gpu_condensed_phase_arrhenius_calc_jac_contrib(md, jac, rxn_int_data,
      //                                     rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_EMISSION :
      //printf("RXN_EMISSION");
      //rxn_gpu_emission_calc_jac_contrib(md, jac, rxn_int_data,
      //                                     rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_FIRST_ORDER_LOSS :
      //rxn_gpu_first_order_loss_calc_jac_contrib(md, jac, rxn_int_data,
      //                                     rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_HL_PHASE_TRANSFER :
      //rxn_gpu_HL_phase_transfer_calc_jac_contrib(md, jac, rxn_int_data,
      //                                             rxn_float_data, rxn_env_data,time_stepn);
      break;
    case RXN_PHOTOLYSIS :
      rxn_gpu_photolysis_calc_jac_contrib(md, jac, rxn_int_data,
                                            rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_SIMPOL_PHASE_TRANSFER :
      //rxn_gpu_SIMPOL_phase_transfer_calc_jac_contrib(md, jac,
      //        rxn_int_data, rxn_float_data, rxn_env_data, time_step);
      break;
    case RXN_TROE :
      rxn_gpu_troe_calc_jac_contrib(md, jac, rxn_int_data,
                                      rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_WET_DEPOSITION :
      //printf("RXN_WET_DEPOSITION");
      //rxn_gpu_wet_deposition_calc_jac_contrib(md, jac, rxn_int_data,
      //                                     rxn_float_data, rxn_env_data,time_step);
      break;
  }
/*
*/

}

__device__ void cudaDevicecalc_Jac0(
#ifdef PMC_DEBUG_GPU
        int counterDeriv2,
#endif
        //check_model_state
        //double threshhold, double replacement_value, int *flag,
        //f_gpu
        double time_step, int deriv_length_cell, int state_size_cell,
        int n_cells, int i_kernel,
        int threads_block, int n_shr_empty, double *y,
        ModelDataGPU md_object
) //Interface CPU/GPU
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int tid_cell=tid%deriv_length_cell;
  int active_threads = n_cells*deriv_length_cell;
  ModelDataGPU *md = &md_object;

#ifdef DEBUG_cudaDeviceJac0
  if(tid==0){
    printf("[DEBUG] GPU solveDerivative tid %d, \n", tid);
  }__syncthreads();
#endif

  if(tid<active_threads){

    //Debug
    /*
    if(counterDeriv2<=1){
      printf("(%d) y %-le J_state %-le J_solver %-le J_tmp %-le J_tmp2 %-le J_deriv %-le\n",tid+1,
             y[tid], md->J_state[tid], md->J_solver[tid], md->J_tmp[tid], md->J_tmp2[tid], md->J_deriv[tid]);
      //printf("gpu threads %d\n", active_threads);
    }
*/

    JacobianGPU *jac = &md->jac;
    JacobianGPU jacBlock;

#ifdef DEV_JACOBIANGPUNUMSPEC
    jac->num_spec = state_size_cell;
    jacBlock.num_spec = state_size_cell;
#endif

#ifdef DEV_MULTICELLSGPU

    //todo not working

    jacBlock.num_elem[0] = jac->num_elem[0]*(blockDim.x/deriv_length_cell);

#else

    jacBlock.num_elem = jac->num_elem;

#endif

    //if(threadIdx.x==0) printf("*jac->num_elem %d\n",jac->num_elem[0]);
    //if(threadIdx.x==0) printf("deriv_length_cell %d\n",deriv_length_cell);
    //if(threadIdx.x==0) printf("state_size_cell %d\n",state_size_cell);

#ifdef AEROS_CPU
#else
#endif

    int i_cell = tid/deriv_length_cell;
    md->i_cell = i_cell;
    //jacBlock.production_partials = &( jac->production_partials[jac.num_elem[0]*i_cell]);
    //jacBlock.loss_partials = &( jac->loss_partials[jac.num_elem[0]*i_cell]);
    jacBlock.production_partials = &( jac->production_partials[jacBlock.num_elem[0]*blockIdx.x]);
    jacBlock.loss_partials = &( jac->loss_partials[jacBlock.num_elem[0]*blockIdx.x]);
    //jacBlock.col_ptrs = &( md->col_ptrs[algo*i_cell]);

    jacobian_reset_gpu(jacBlock);

    md->grid_cell_state = &( md->state[state_size_cell*i_cell]);
    md->grid_cell_env = &( md->env[PMC_NUM_ENV_PARAM_*i_cell]);

    /*
    md->grid_cell_aero_rep_env_data =
    &(md->aero_rep_env_data[md->i_cell*md->n_aero_rep_env_data]);

    //Filter threads for n_aero_rep
    int n_aero_rep = md->n_aero_rep;
    if( tid_cell < n_aero_rep) {
      int n_iters = n_aero_rep / deriv_length_cell;
      //Repeat if there are more reactions than species
      for (int i = 0; i < n_iters; i++) {
        md->i_aero_rep = tid_cell + i*deriv_length_cell;

        aero_rep_gpu_update_state(md);
      }

      //Limit tid to pending rxns to compute
      int residual=n_aero_rep-(deriv_length_cell*n_iters);
      if(tid_cell < residual){
        md->i_aero_rep = tid_cell + deriv_length_cell*n_iters;

        aero_rep_gpu_update_state(md);
      }
    }
     */

#ifdef DEBUG_cudaDevicecalc_Jac0

    if(tid==0)printf("cudaDevicecalc_Jac01\n");

    //if(threadIdx.x==0) {
    //  printf("jac.num_elem %d\n",jacBlock.num_elem);
    //  printf("*md->n_mapped_values %d\n",*md->n_mapped_values);
      //for (int i=0; i<*md->n_mapped_values; i++){
      //  printf("cudaDevicecalc_Jac0 jacBlock [%d]=%le\n",i,jacBlock.production_partials[i]);
      //}
    //}

#endif

    __syncthreads();
    //Filter threads for n_rxn
    int n_rxn = md->n_rxn;
    if( tid_cell < n_rxn) {
      int n_iters = n_rxn / deriv_length_cell;
      //Repeat if there are more reactions than species
      for (int i = 0; i < n_iters; i++) {
        md->i_rxn = tid_cell + i*deriv_length_cell;

        solveRXNJac0(jacBlock, time_step, md);
      }

      //Limit tid to pending rxns to compute
      int residual=n_rxn-(deriv_length_cell*n_iters);
      if(tid_cell < residual){
        md->i_rxn = tid_cell + deriv_length_cell*n_iters;

        solveRXNJac0(jacBlock, time_step, md);
      }
    }
    __syncthreads();


    jacobian_output_gpu(jacBlock, &(md->J_rxn[jacBlock.num_elem[0]*blockIdx.x]) );


#ifdef DEBUG_cudaDevicecalc_Jac0

    if(threadIdx.x==0) {
        printf("jac.num_elem %d\n",jacBlock.num_elem[0]);
        printf("*md->n_mapped_values %d\n",*md->n_mapped_values);
        for (int i=0; i<10; i++){//*md->n_mapped_values
          printf("cudaDevicecalc_Jac0 J_rxn [%d]=%le\n",i,md->J_rxn[i]);
        }
      }

#endif


#ifdef DEV_REMOVE_threadIdx0

    //todo use diA and djA pointers to better memory access
    JacMap *jac_map = md->jac_map;

    int nnz = md->n_mapped_values[0];
    int nnz_tid = nnz/active_threads;

    for (int n = i*nnz_tid; n < i*(nnz_tid+1); ++n) {

      md->J[jac_map[n].solver_id+nnz*blockIdx.x] = //+=0.;
              md->J_rxn[jac_map[n].rxn_id+jacBlock.num_elem[0]*blockIdx.x];

    }

    int nnz_left = nnz-(nnz_tid*active_threads);
    if(i<nnz_left)
      for (int n = nnz_tid; n < nnz_tid; ++n) {

        md->J[jac_map[n].solver_id+nnz*blockIdx.x] = //+=0.;
                md->J_rxn[jac_map[n].rxn_id+jacBlock.num_elem[0]*blockIdx.x];

      }

#else

    __syncthreads();
    if(threadIdx.x==0) {

      JacMap *jac_map = md->jac_map;
      int nnz = md->n_mapped_values[0];
      //int nnz = jac->num_elem;
      //SM_DATA_S(md->J_params)[0] = 1.0;  // dummy value for non-sub model calcs
      //for (int i_map = 0; i_map < md->n_mapped_values; ++i_map) {
      //for (int n = (nnz/gridDim.x)*blockIdx.x; n < (nnz/gridDim.x)*(blockIdx.x+1); n++) {

#ifdef DEV_MULTICELLSGPU

      //*(blockDim.x/deriv_length_cell)
      for (int j = 0; j < (blockDim.x/deriv_length_cell); j++) {
        for (int n = 0; n < nnz; n++) {
          //md->J[n] = 0.0;
          //md->J[n] = 0.0;

          //o = (j-1)*(I_E) + (i-1)
          //z = (k-1)*(I_E*I_N) + o
          int z = blockIdx.x*blockDim.x + j*nnz;
          int z_rxn = blockIdx.x*blockDim.x + j*jacBlock.num_elem[0];

          md->J[jac_map[n].solver_id + z] = //+=0.;
                  md->J_rxn[jac_map[n].rxn_id + jacBlock.num_elem[0] * blockIdx.x];
          //0.0;
          //* SM_DATA_S(md->J_params)[jac_map[i_map].param_id];

          //SM_DATA_S(J)
          //[i_cell * md->n_per_cell_solver_jac_elem + jac_map[i_map].solver_id] +=
          //        SM_DATA_S(md->J_rxn)[jac_map[i_map].rxn_id] *
          //0.0;
          //        SM_DATA_S(md->J_params)[jac_map[i_map].param_id];

        }
      }

#else

      for (int n = 0; n < nnz; n++) {
        md->J[jac_map[n].solver_id + nnz * blockIdx.x] = 0.0;

        md->J[jac_map[n].solver_id + nnz * blockIdx.x] = //+=0.;
                md->J_rxn[jac_map[n].rxn_id + jacBlock.num_elem[0] * blockIdx.x];
        //0.0;
        //* SM_DATA_S(md->J_params)[jac_map[i_map].param_id];

        //SM_DATA_S(J)
        //[i_cell * md->n_per_cell_solver_jac_elem + jac_map[i_map].solver_id] +=
        //        SM_DATA_S(md->J_rxn)[jac_map[i_map].rxn_id] *
        //0.0;
        //        SM_DATA_S(md->J_params)[jac_map[i_map].param_id];

      }

#endif

    }__syncthreads();

#endif

  }
}

__device__
void cudaDeviceJac0(
#ifdef PMC_DEBUG_GPU
        int counterDeriv2,
#endif
        //check_model_state
        double threshhold, double replacement_value, int *flag,
        //f_gpu
        double time_step, int deriv_length_cell, int state_size_cell,
        int n_cells, int i_kernel,
        int threads_block, int n_shr_empty, double *y,
        ModelDataGPU md_object, double *dftemp
) //Interface CPU/GPU
{

  int tid = threadIdx.x;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  int active_threads = n_cells*deriv_length_cell;


  ModelDataGPU *md = &md_object;

  cudaDevicef0(
#ifdef PMC_DEBUG_GPU
          counterDeriv2,
#endif
          //check_model_state
          threshhold, replacement_value, flag,
          //f_gpu
          time_step, deriv_length_cell, state_size_cell,
          n_cells, i_kernel, threads_block, n_shr_empty, y,
          md_object
  );



  //duplicated call to check_model_state (previous f funct already checks model_state)
  /*
  cudaDevicecamp_solver_check_model_state0(md->state, y,
                                           md->map_state_deriv, threshhold, replacement_value,
                                           flag, deriv_length_cell, n_cells);
*/


  //__syncthreads();
  //study flag block effect: flag is global for all threads or for only the block?
  if(*flag==CAMP_SOLVER_FAIL)
    return;



#ifdef DEV_RESET_JAC_GPU_TO_INIT
  //todo ensure dont needed
  SM_NNZ_S(J) = SM_NNZ_S(md->J_init);
  for (int i = 0; i <= SM_NP_S(J); i++) {
    (SM_INDEXPTRS_S(J))[i] = (SM_INDEXPTRS_S(md->J_init))[i];
    bicg->iA[i]=SM_INDEXPTRS_S(J)[i];
  }
  for (int i = 0; i < SM_NNZ_S(J); i++) {
    (SM_INDEXVALS_S(J))[i] = (SM_INDEXVALS_S(md->J_init))[i];
    bicg->jA[i]=SM_INDEXVALS_S(J)[i];
    (SM_DATA_S(J))[i] = (realtype)0.0;
  }
  hipMemcpy(mGPU->djA,bicg->jA,mGPU->nnz*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(mGPU->diA,bicg->iA,(mGPU->nrows+1)*sizeof(int),hipMemcpyHostToDevice);

#endif


  //if(tid==0)printf("cudaDeviceJac01\n");

  cudaDevicecalc_Jac0(
#ifdef PMC_DEBUG_GPU
          counterDeriv2,
#endif
          //check_model_state          md->map_state_deriv, threshhold, replacement_value, flag,
          //f_gpu
          time_step, deriv_length_cell, state_size_cell,
          n_cells, i_kernel, threads_block, n_shr_empty, y,
          md_object
  );

  //if(tid==0)printf("cudaDeviceJac0End\n");

#ifdef DEV_REMOVE_threadIdx0
#else
  if(i<active_threads){

    __syncthreads();
    if(threadIdx.x==0) {
      int nnz = md->n_mapped_values[0];
      for (int n = 0; n < nnz; n++) {
        md->J_solver[n]=md->J[n];
      }
    }__syncthreads();

    md->J_state[i]=y[i];
    md->J_deriv[i]=md->deriv_data[i];

  }

#endif

}

__global__
void cudaGlobalJac0(
#ifdef PMC_DEBUG_GPU
        int counterDeriv2,
#endif
        //check_model_state
        double threshhold, double replacement_value, int *flag,
        //f_gpu
        double time_step, int deriv_length_cell,
        int state_size_cell, int n_cells,
        int i_kernel, int threads_block, int n_shr_empty, double *y,
        ModelDataGPU md_object, double *dftemp
) //Interface CPU/GPU
{

  ModelDataGPU *md = &md_object;
  JacobianGPU *jac = &md->jac;

  //if(threadIdx.x==0)printf("cudaGlobalJac \n");
  //__syncthreads();

  cudaDeviceJac0(
#ifdef PMC_DEBUG_GPU
          counterDeriv2,
#endif
          //check_model_state
          threshhold, replacement_value, flag,
          //f_gpu
          time_step, deriv_length_cell, state_size_cell,
          n_cells, i_kernel, threads_block, n_shr_empty, y,
          md_object, dftemp
  );
}


/** \brief Calculate the Jacobian on GPU
 *
 * \param md Pointer to the model data
 * \param J Jacobian to be calculated
 * \param time_step Current model time step (s)
 */

int rxn_calc_jac_gpu(SolverData *sd, SUNMatrix J, double time_step, N_Vector deriv) {


  ModelData *md = &(sd->model_data);
  itsolver *bicg = &(sd->bicg);
  ModelDataGPU *mGPU = &sd->mGPU;
  double *deriv_data = N_VGetArrayPointer(deriv);
  int n_cells = md->n_cells;
  int n_kernels = 1; // Divide load into multiple kernel calls
  //todo n_kernels case division left residual, an extra kernel computes remain residual
#ifdef DEV_MULTICELLSGPU
  int total_threads = md->n_per_cell_dep_var * n_cells/n_kernels;
  int n_shr_empty = md->max_n_gpu_thread%md->n_per_cell_dep_var;
  int threads_block = md->max_n_gpu_thread - n_shr_empty; //last multiple of size_cell before max_threads
  int n_blocks = ((total_threads + threads_block - 1) / threads_block);
#else
  int threads_block = md->n_per_cell_dep_var;
  int n_blocks = mGPU->n_cells;
  //int n_shr = nextPowerOfTwo2(len_cell);
  //int n_shr_empty = n_shr-threads_block;
  int n_shr_empty = 0;
#endif

  //printf("threads_block %d n_blocks %d",total_threads,n_blocks);

  //Update state
  double replacement_value = TINY;
  double threshhold = -SMALL;
  int flag = CAMP_SOLVER_SUCCESS; //0

  //debug
  /*
   if(sd->counterDerivGPU<=0){
     printf("f_gpu start total_state [(id),conc], n_state_var %d, n_cells %d\n", md->n_per_cell_state_var, n_cells);
     printf("n_deriv %d\n", md->n_per_cell_dep_var);
     for (int i = 0; i < md->n_per_cell_state_var*n_cells; i++) {
       printf("(%d) %-le \n",i+1, md->total_state[i]);
     }
   }
   */

#ifdef DEV_RESET_JAC_GPU_TO_INIT

  //dont needed at first instance (only klu sparse resize the Jac, but our GPU ODE dont do that)
  SM_NNZ_S(J) = SM_NNZ_S(md->J_init);
  for (int i = 0; i <= SM_NP_S(J); i++) {
    (SM_INDEXPTRS_S(J))[i] = (SM_INDEXPTRS_S(md->J_init))[i];
    bicg->iA[i]=SM_INDEXPTRS_S(J)[i];
  }
  for (int i = 0; i < SM_NNZ_S(J); i++) {
    (SM_INDEXVALS_S(J))[i] = (SM_INDEXVALS_S(md->J_init))[i];
    bicg->jA[i]=SM_INDEXVALS_S(J)[i];
    (SM_DATA_S(J))[i] = (realtype)0.0;
  }
  hipMemcpy(mGPU->djA,bicg->jA,mGPU->nnz*sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(mGPU->diA,bicg->iA,(mGPU->nrows+1)*sizeof(int),hipMemcpyHostToDevice);

#endif

#ifdef DEBUG_rxn_calc_jac_gpu

  //hipDeviceSynchronize();
  for (int i=0; i<1; i++){//*md->n_mapped_values
    //printf("rxn_calc_jac_gpu J_rxn [%d]=%le\n",i,J_rxn_data[i]);
    printf("rxn_calc_jac_gpu start cudaGlobalJac \n");
  }

#endif

  //Loop to test multiple kernel executions
  for (int i_kernel=0; i_kernel<n_kernels; i_kernel++){
    //hipDeviceSynchronize();
    //solveDerivative << < (n_blocks), threads_block >> >(
    cudaGlobalJac0 << < (n_blocks), threads_block >> >(
#ifdef PMC_DEBUG_GPU
            sd->counterDerivGPU,
#endif
            //update_state
            threshhold, replacement_value, mGPU->flag,
            //f_gpu
            time_step, md->n_per_cell_dep_var,
            md->n_per_cell_state_var,md->n_cells,
            i_kernel, threads_block,n_shr_empty, mGPU->dcv_y,
            sd->mGPU, mGPU->dftemp
    );
  }

  hipMemcpy(&flag,mGPU->flag,1*sizeof(int),hipMemcpyDeviceToHost);

  //printf("rxn_calc_jac_gpu flag %d\n", flag);

#ifdef DEBUG_rxn_calc_jac_gpu

  hipDeviceSynchronize();
  for (int i=0; i<1; i++){//*md->n_mapped_values
    //printf("rxn_calc_jac_gpu J_rxn [%d]=%le\n",i,J_rxn_data[i]);
    printf("rxn_calc_jac_gpu end cudaGlobalJac \n");
  }

#endif

  if(flag==CAMP_SOLVER_FAIL)
    return flag;

  //Async
  //HANDLE_ERROR(hipMemcpyAsync(md->deriv_aux, md->deriv_data_gpu,
  //md->deriv_size, hipMemcpyDeviceToHost, md->stream_gpu[STREAM_DERIV_GPU]));

  double *J_data = SM_DATA_S(J);
  HANDLE_ERROR(hipMemcpy(J_data, mGPU->J, md->jac_size, hipMemcpyDeviceToHost));


#ifdef DEBUG_rxn_calc_jac_gpu

  for (int i=0; i<1; i++){//*md->n_mapped_values
    //printf("rxn_calc_jac_gpu J_rxn [%d]=%le\n",i,J_rxn_data[i]);
    printf("rxn_calc_jac_gpu J_data [%d]=%le\n",i,J_data[i]);
  }

#endif

  return flag;
}

/** \brief Free GPU data structures
 */
void free_gpu_cu(SolverData *sd) {

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU = &sd->mGPU;

  free(sd->flagCells);

#ifdef PMC_DEBUG_GPU

  printf("timeDerivKernel %lf\n", md->timeDerivKernel/1000);

#endif

  //for (int i = 0; i < n_streams; ++i)
  //  HANDLE_ERROR( hipStreamDestroy(md->stream_gpu[i]) );
/*

  */
  //free(md->jac_aux);
  HANDLE_ERROR(hipFree(mGPU->rxn_int));
  HANDLE_ERROR(hipFree(mGPU->rxn_double));
  HANDLE_ERROR(hipFree(mGPU->deriv_data));
  //HANDLE_ERROR(hipFree(J_solver_gpu));

  if(md->small_data){
  }
  else{
    free(md->deriv_aux);
    HANDLE_ERROR(hipFree(mGPU->state));
    HANDLE_ERROR(hipFree(mGPU->env));
    HANDLE_ERROR(hipFree(mGPU->rxn_env_data));
    HANDLE_ERROR(hipFree(mGPU->rxn_env_data_idx));

  }

}

/* Auxiliar functions */

void bubble_sort_gpu(unsigned int *n_zeros, unsigned int *rxn_position, int n_rxn){

  int tmp,s=1,i_rxn=n_rxn;

  while(s){
    s=0;
    for (int i = 1; i < i_rxn; i++) {
      //Few zeros go first
      if (n_zeros[i] < n_zeros[i - 1]) {
        //Swap positions
        tmp = rxn_position[i];
        rxn_position[i] = rxn_position[i - 1];
        rxn_position[i - 1] = tmp;

        tmp = n_zeros[i];
        n_zeros[i] = n_zeros[i - 1];
        n_zeros[i - 1] = tmp;
        s=1;
      }
    }
    i_rxn--;
  }

}

/* Prints */

void print_gpu_specs() {

  printf("GPU specifications \n");

  int nDevices;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  maxGridSize: %d\n", prop.maxGridSize[1]);
    printf("  maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
    printf("  maxThreadsDim: %d\n", prop.maxThreadsDim[1]);
    printf("  totalGlobalMem: %zu\n", prop.totalGlobalMem);
    printf("  sharedMemPerBlock: %zu\n", prop.sharedMemPerBlock); //bytes
    printf("  multiProcessorCount: %d\n", prop.multiProcessorCount);
  }

}

// Old code (Not used now, but could be useful)
/*
 //use this instead of normal update_model_state? is less code
int camp_solver_update_model_state_cpu(N_Vector solver_state, ModelData *md,
                                       realtype threshhold, realtype replacement_value)
{
  int status = CAMP_SOLVER_FAIL;
  int n_cells = md->n_cells;
  int n_state_var = md->n_per_cell_state_var;
  int n_dep_var = md->n_per_cell_dep_var;
  int n_threads = n_state_var*n_cells;
  int n_blocks = ((n_threads + md->max_n_gpu_thread - 1) / md->max_n_gpu_thread);
  int *var_type = md->var_type;
  double *state = md->total_state;
  double *y = NV_DATA_S(solver_state);
  int *map_state_deriv = md->map_state_deriv;

  for(int i_dep_var = 0; i_dep_var < n_dep_var*n_cells; i_dep_var++)
  {
    if (NV_DATA_S(solver_state)[i_dep_var] > -SMALL) {
      md->total_state[map_state_deriv[i_dep_var]] =
              NV_DATA_S(solver_state)[i_dep_var] > threshhold
              ? NV_DATA_S(solver_state)[i_dep_var] : replacement_value;
      status = CAMP_SOLVER_SUCCESS;
    } else { //error
#ifdef FAILURE_DETAIL
      printf("\nFailed model state update: [spec %d] = %le", i_spec,
                 NV_DATA_S(solver_state)[i_dep_var]);
#endif
      status = CAMP_SOLVER_FAIL;
      break;
    }
  }
  return status;
}
*/


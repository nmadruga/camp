/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
 * Illinois at Urbana-Champaign
 * SPDX-License-Identifier: MIT
 */

extern "C" {
#include "camp_gpu_solver.h"
}

void set_jac_data_gpu(SolverData *sd, double *J){
  ModelData *md = &(sd->model_data);
  ModelDataCPU *mCPU = &(sd->mCPU);
  ModelDataGPU *mGPU;
  mGPU = sd->mGPU;
  double *J_ptr = J;
  double *J_solver = SM_DATA_S(md->J_solver);
  double *J_state = N_VGetArrayPointer(md->J_state);
  double *J_deriv = N_VGetArrayPointer(md->J_deriv);
  hipMemcpy(mGPU->dA, J_ptr, mCPU->jac_size, hipMemcpyHostToDevice);
  hipMemcpy(mGPU->J_solver, J_solver, mCPU->jac_size, hipMemcpyHostToDevice);
  hipMemcpy(mGPU->J_state, J_state, mCPU->deriv_size, hipMemcpyHostToDevice);
  hipMemcpy(mGPU->J_deriv, J_deriv, mCPU->deriv_size, hipMemcpyHostToDevice);
  hipMemcpy(mGPU->djA, mCPU->jA, mGPU->nnz/mGPU->n_cells * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->diA, mCPU->iA, (mGPU->nrows/mGPU->n_cells + 1) * sizeof(int), hipMemcpyHostToDevice);
}

void camp_solver_update_model_state_gpu(N_Vector solver_state, SolverData *sd){
  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU;
  ModelDataCPU *mCPU = &(sd->mCPU);
  double *total_state = md->total_state;
  mGPU = sd->mGPU;
  hipMemcpy(mGPU->state, total_state, mCPU->state_size, hipMemcpyHostToDevice);
}

int rxn_calc_deriv_gpu(SolverData *sd, N_Vector y, N_Vector deriv, double time_step) {
  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU;
  ModelDataCPU *mCPU = &(sd->mCPU);
  double *total_state = md->total_state;
  double *deriv_data = N_VGetArrayPointer(deriv);
  if(sd->use_gpu_cvode==0){
    mGPU = sd->mGPU;
    hipMemcpy(mGPU->deriv_data, deriv_data, mCPU->deriv_size, hipMemcpyHostToDevice);
    hipMemcpy(mGPU->state, total_state, mCPU->state_size, hipMemcpyHostToDevice);
  }
  return 0;
}

void free_gpu_cu(SolverData *sd) {
  ModelDataGPU *mGPU = sd->mGPU;
  ModelDataCPU *mCPU = &(sd->mCPU);
  //printf("free_gpu_cu start\n");
  free(sd->flagCells);
  mGPU = sd->mGPU;
  //ModelDataGPU Start
  hipFree(mGPU->map_state_deriv);
  hipFree(mGPU->deriv_data);
  hipFree(mGPU->J_solver);
  hipFree(mGPU->J_state);
  hipFree(mGPU->J_deriv);
  hipFree(mGPU->J_tmp);
  hipFree(mGPU->J_tmp2);
  hipFree(mGPU->indexvals);
  hipFree(mGPU->indexptrs);
  hipFree(mGPU->rxn_int);
  hipFree(mGPU->rxn_double);
  hipFree(mGPU->state);
  hipFree(mGPU->env);
  hipFree(mGPU->rxn_env_data);
  hipFree(mGPU->rxn_env_data_idx);
  hipFree(mGPU->production_rates);
  hipFree(mGPU->loss_rates);
  hipFree(mGPU->rxn_int_indices);
  hipFree(mGPU->rxn_float_indices);
#ifdef DEV_AERO_REACTIONS
  hipFree(mGPU->aero_rep_int_indices);
  hipFree(mGPU->aero_rep_float_indices);
  hipFree(mGPU->aero_rep_env_idx);
  hipFree(mGPU->aero_rep_int_data);
  hipFree(mGPU->aero_rep_float_data);
  hipFree(mGPU->aero_rep_env_data);
#endif
  hipFree(mGPU->n_mapped_values);
  hipFree(mGPU->jac_map);
  hipFree(mGPU->yout);
  hipFree(mGPU->cv_Vabstol);
  hipFree(mGPU->grid_cell_state);
  hipFree(mGPU->grid_cell_env);
  hipFree(mGPU->grid_cell_aero_rep_env_data);
  hipFree(mGPU->cv_l);
  hipFree(mGPU->cv_tau);
  hipFree(mGPU->cv_tq);
  hipFree(mGPU->cv_last_yn);
  hipFree(mGPU->cv_acor_init);
  hipFree(mGPU->dA);
  hipFree(mGPU->djA);
  hipFree(mGPU->diA);
  hipFree(mGPU->dx);
  hipFree(mGPU->dtempv);
  hipFree(mGPU->ddiag);
  hipFree(mGPU->dr0);
  hipFree(mGPU->dr0h);
  hipFree(mGPU->dn0);
  hipFree(mGPU->dp0);
  hipFree(mGPU->dt);
  hipFree(mGPU->ds);
  hipFree(mGPU->dAx2);
  hipFree(mGPU->dy);
  hipFree(mGPU->dz);
  hipFree(mGPU->dftemp);
  hipFree(mGPU->dcv_y);
  hipFree(mGPU->dtempv1);
  hipFree(mGPU->dtempv2);
  hipFree(mGPU->flag);
  hipFree(mGPU->flagCells);
  hipFree(mGPU->cv_acor);
  hipFree(mGPU->dzn);
  hipFree(mGPU->dewt);
  hipFree(mGPU->dsavedJ);
  hipFree(mCPU->map_state_derivCPU);
  hipFree(mGPU->mdv);
  hipFree(mGPU->mdvo);
  hipFree(mGPU);
}

void print_gpu_specs() {
  printf("GPU specifications \n");
  int nDevicesMax;
  hipGetDeviceCount(&nDevicesMax);
  for (int i = 0; i < nDevicesMax; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  maxGridSize: %d\n", prop.maxGridSize[1]);
    printf("  maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
    printf("  maxThreadsDim: %d\n", prop.maxThreadsDim[1]);
    printf("  totalGlobalMem: %zu\n", prop.totalGlobalMem);
    printf("  sharedMemPerBlock: %zu\n", prop.sharedMemPerBlock); //bytes
    printf("  multiProcessorCount: %d\n", prop.multiProcessorCount);
  }
}
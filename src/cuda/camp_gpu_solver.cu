/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
 * Illinois at Urbana-Champaign
 * SPDX-License-Identifier: MIT
 */

#include "itsolver_gpu.h"

extern "C" {
#include "camp_gpu_solver.h"
#include "rxns_gpu.h"
#ifdef DEV_AERO_REACTIONS
#include "aeros/aero_rep_gpu_solver.h"
#endif
#include "time_derivative_gpu.h"
#include "Jacobian_gpu.h"
}


// Reaction types (Must match parameters defined in camp_rxn_factory)
#define RXN_ARRHENIUS 1
#define RXN_TROE 2
#define RXN_CMAQ_H2O2 3
#define RXN_CMAQ_OH_HNO3 4
#define RXN_PHOTOLYSIS 5
#define RXN_HL_PHASE_TRANSFER 6
#define RXN_AQUEOUS_EQUILIBRIUM 7
#define RXN_SIMPOL_PHASE_TRANSFER 10
#define RXN_CONDENSED_PHASE_ARRHENIUS 11
#define RXN_FIRST_ORDER_LOSS 12
#define RXN_EMISSION 13
#define RXN_WET_DEPOSITION 14

// Status codes for calls to camp_solver functions
#define CAMP_SOLVER_SUCCESS 0
#define CAMP_SOLVER_FAIL 1

//GPU async stream related variables to ensure robustness
//int n_solver_objects=0; //Number of solver_new_gpu calls
//hipStream_t *stream_gpu; //GPU streams to async computation/data movement
//int n_streams = 16;

static void HandleError(hipError_t err,
                        const char *file,
                        int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err),
           file, line);
    exit(EXIT_FAILURE);
  }
}

void set_jac_data_gpu(SolverData *sd, double *J){

  ModelData *md = &(sd->model_data);
  ModelDataCPU *mCPU = &(sd->mCPU);
  ModelDataGPU *mGPU;

  int offset_nnz_J_solver = 0;
  int offset_nrows = 0;
  for (int iDevice = sd->startDevice; iDevice < sd->endDevice; iDevice++) {
    hipSetDevice(iDevice);
    sd->mGPU = &(sd->mGPUs[iDevice]);
    mGPU = sd->mGPU;

    double *J_ptr = J+offset_nnz_J_solver;
    double *J_solver = SM_DATA_S(md->J_solver)+offset_nnz_J_solver;
    double *J_state = N_VGetArrayPointer(md->J_state)+offset_nrows;
    double *J_deriv = N_VGetArrayPointer(md->J_deriv)+offset_nrows;
    HANDLE_ERROR(hipMemcpy(mGPU->dA, J_ptr, mCPU->jac_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(mGPU->J_solver, J_solver, mCPU->jac_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(mGPU->J_state, J_state, mCPU->deriv_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(mGPU->J_deriv, J_deriv, mCPU->deriv_size, hipMemcpyHostToDevice));

    offset_nnz_J_solver += mCPU->nnz_J_solver;
    offset_nrows += md->n_per_cell_dep_var* mGPU->n_cells;
    hipMemcpy(mGPU->djA, mCPU->jA, mGPU->nnz/mGPU->n_cells * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(mGPU->diA, mCPU->iA, (mGPU->nrows/mGPU->n_cells + 1) * sizeof(int), hipMemcpyHostToDevice);
  }
}

void rxn_update_env_state_gpu(SolverData *sd) {

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU;
  ModelDataCPU *mCPU = &(sd->mCPU);
  double *rxn_env_data = md->rxn_env_data;
  double *env = md->total_env;
  double *total_state = md->total_state;

  for (int iDevice = sd->startDevice; iDevice < sd->endDevice; iDevice++) {
    hipSetDevice(iDevice);
    sd->mGPU = &(sd->mGPUs[iDevice]);
    mGPU = sd->mGPU;

    HANDLE_ERROR(hipMemcpy(mGPU->rxn_env_data, rxn_env_data, mCPU->rxn_env_data_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(mGPU->env, env, mCPU->env_size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(mGPU->state, total_state, mCPU->state_size, hipMemcpyHostToDevice));

    rxn_env_data += mGPU->n_rxn_env_data * mGPU->n_cells;
    env += CAMP_NUM_ENV_PARAM_ * mGPU->n_cells;
    total_state += mGPU->state_size_cell * mGPU->n_cells;

  }

}

void camp_solver_update_model_state_gpu(N_Vector solver_state, SolverData *sd,
                                       double threshhold, double replacement_value)
{
  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU;
  ModelDataCPU *mCPU = &(sd->mCPU);
  double *total_state = md->total_state;
  for (int iDevice = sd->startDevice; iDevice < sd->endDevice; iDevice++) {
    hipSetDevice(iDevice);
    sd->mGPU = &(sd->mGPUs[iDevice]);
    mGPU = sd->mGPU;
    HANDLE_ERROR(hipMemcpy(mGPU->state, total_state, mCPU->state_size, hipMemcpyHostToDevice));
    total_state += mGPU->state_size_cell * mGPU->n_cells;
  }
}


int rxn_calc_deriv_gpu(SolverData *sd, N_Vector y, N_Vector deriv, double time_step) {

  ModelData *md = &(sd->model_data);
  ModelDataGPU *mGPU;
  ModelDataCPU *mCPU = &(sd->mCPU);

  double *total_state = md->total_state;
  double *deriv_data = N_VGetArrayPointer(deriv);
  if(sd->use_gpu_cvode==0){
    for (int iDevice = sd->startDevice; iDevice < sd->endDevice; iDevice++) {
      hipSetDevice(iDevice);
      sd->mGPU = &(sd->mGPUs[iDevice]);
      mGPU = sd->mGPU;

      HANDLE_ERROR(hipMemcpy(mGPU->deriv_data, deriv_data, mCPU->deriv_size, hipMemcpyHostToDevice));
      HANDLE_ERROR(hipMemcpy(mGPU->state, total_state, mCPU->state_size, hipMemcpyHostToDevice));

      total_state += mGPU->state_size_cell * mGPU->n_cells;
      deriv_data += mGPU->nrows;
    }
  }
  return 0;
}

void free_gpu_cu(SolverData *sd) {
  ModelDataGPU *mGPU = sd->mGPU;
  ModelDataCPU *mCPU = &(sd->mCPU);
  //printf("free_gpu_cu start\n");
  free(sd->flagCells);
  for (int iDevice = sd->startDevice; iDevice < sd->endDevice; iDevice++) {
    hipSetDevice(iDevice);
    sd->mGPU = &(sd->mGPUs[iDevice]);
    mGPU = sd->mGPU;
    //hipStreamDestroy(mCPU->streams[iDevice]);
    //ModelDataGPU Start
    hipFree(mGPU->map_state_deriv);
    hipFree(mGPU->deriv_data);
    hipFree(mGPU->J_solver);
    hipFree(mGPU->J_state);
    hipFree(mGPU->J_deriv);
    hipFree(mGPU->J_tmp);
    hipFree(mGPU->J_tmp2);
    hipFree(mGPU->indexvals);
    hipFree(mGPU->indexptrs);
    hipFree(mGPU->rxn_int);
    hipFree(mGPU->rxn_double);
    hipFree(mGPU->state);
    hipFree(mGPU->env);
    hipFree(mGPU->rxn_env_data);
    hipFree(mGPU->rxn_env_data_idx);
    hipFree(mGPU->production_rates);
    hipFree(mGPU->loss_rates);
    hipFree(mGPU->rxn_int_indices);
    hipFree(mGPU->rxn_float_indices);
#ifdef DEV_AERO_REACTIONS
    hipFree(mGPU->aero_rep_int_indices);
    hipFree(mGPU->aero_rep_float_indices);
    hipFree(mGPU->aero_rep_env_idx);
    hipFree(mGPU->aero_rep_int_data);
    hipFree(mGPU->aero_rep_float_data);
    hipFree(mGPU->aero_rep_env_data);
#endif
    hipFree(mGPU->n_mapped_values);
    hipFree(mGPU->jac_map);
    hipFree(mGPU->yout);
    hipFree(mGPU->cv_Vabstol);
    hipFree(mGPU->grid_cell_state);
    hipFree(mGPU->grid_cell_env);
    hipFree(mGPU->grid_cell_aero_rep_env_data);
    hipFree(mGPU->cv_l);
    hipFree(mGPU->cv_tau);
    hipFree(mGPU->cv_tq);
    hipFree(mGPU->cv_last_yn);
    hipFree(mGPU->cv_acor_init);
    hipFree(mGPU->dA);
    hipFree(mGPU->djA);
    hipFree(mGPU->diA);
    hipFree(mGPU->dx);
    hipFree(mGPU->dtempv);
    hipFree(mGPU->ddiag);
    hipFree(mGPU->dr0);
    hipFree(mGPU->dr0h);
    hipFree(mGPU->dn0);
    hipFree(mGPU->dp0);
    hipFree(mGPU->dt);
    hipFree(mGPU->ds);
    hipFree(mGPU->dAx2);
    hipFree(mGPU->dy);
    hipFree(mGPU->dz);
    hipFree(mGPU->dftemp);
    hipFree(mGPU->dcv_y);
    hipFree(mGPU->dtempv1);
    hipFree(mGPU->dtempv2);
    hipFree(mGPU->flag);
    hipFree(mGPU->flagCells);
    hipFree(mGPU->cv_acor);
    hipFree(mGPU->dzn);
    hipFree(mGPU->dewt);
    hipFree(mGPU->dsavedJ);
    hipFree(mCPU->map_state_derivCPU);
    hipFree(mGPU->mdv);
    hipFree(mGPU->mdvo);
    hipFree(mGPU);
  }
}

void print_gpu_specs() {

  printf("GPU specifications \n");

  int nDevicesMax;
  hipGetDeviceCount(&nDevicesMax);
  for (int i = 0; i < nDevicesMax; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  maxGridSize: %d\n", prop.maxGridSize[1]);
    printf("  maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
    printf("  maxThreadsDim: %d\n", prop.maxThreadsDim[1]);
    printf("  totalGlobalMem: %zu\n", prop.totalGlobalMem);
    printf("  sharedMemPerBlock: %zu\n", prop.sharedMemPerBlock); //bytes
    printf("  multiProcessorCount: %d\n", prop.multiProcessorCount);
  }



}


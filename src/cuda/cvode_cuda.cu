#include "hip/hip_runtime.h"
/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
* Illinois at Urbana-Champaign
* SPDX-License-Identifier: MIT
*/

#include "cvode_cuda.h"
extern "C" {
#include "new.h"
}

__device__
void time_derivative_add_value_gpu(TimeDerivativeGPU time_deriv, unsigned int spec_id,
                               double rate_contribution) {
  if (rate_contribution > 0.0) {
    atomicAdd_block(&(time_deriv.production_rates[spec_id]),rate_contribution);
  } else {
    atomicAdd_block(&(time_deriv.loss_rates[spec_id]),-rate_contribution);
  }
}

__device__
void rxn_gpu_first_order_loss_calc_deriv_contrib(ModelDataGPU *model_data, TimeDerivativeGPU time_deriv, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *state = model_data->grid_cell_state;
  double rate = rxn_env_data[0] * state[int_data[1]-1];
  if (int_data[2] >= 0) time_derivative_add_value_gpu(time_deriv, int_data[2], -rate);
}

__device__
void rxn_gpu_first_order_loss_calc_jac_contrib(ModelDataGPU *model_data, JacobianGPU jac, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  if (int_data[3] >= 0) atomicAdd_block(&(jac.loss_partials[int_data[3]]),rxn_env_data[0]);
}

__device__
void rxn_gpu_CMAQ_H2O2_calc_deriv_contrib(ModelDataGPU *model_data, TimeDerivativeGPU time_deriv, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  double rate = rxn_env_data[0];
  for (int i_spec=0; i_spec<int_data[0]; i_spec++) rate *= state[int_data[(2 + i_spec)]-1];
  if (rate!=0.) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<int_data[0]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0]+int_data[1]+i_dep_var)] < 0) continue;
      time_derivative_add_value_gpu(time_deriv, int_data[(2 + int_data[0]+int_data[1]+i_dep_var)], -rate);
    }
    for (int i_spec=0; i_spec<int_data[1]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0]+int_data[1]+i_dep_var)] < 0) continue;
      if (-rate*float_data[(7 + i_spec)]*time_step <= state[int_data[(2 + int_data[0]+ i_spec)]-1]) {
        time_derivative_add_value_gpu(time_deriv, int_data[(2 + int_data[0]+int_data[1]+i_dep_var)],rate*float_data[(7 + i_spec)]);
      }
    }
  }
}

__device__
void rxn_gpu_CMAQ_H2O2_calc_jac_contrib(ModelDataGPU *model_data, JacobianGPU jac, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  int i_elem = 0;
  for (int i_ind = 0; i_ind < int_data[0]; i_ind++) {
    double rate = rxn_env_data[0];
    for (int i_spec = 0; i_spec < int_data[0]; i_spec++)
      if (i_ind != i_spec) rate *= state[int_data[(2 + i_spec)]-1];
    for (int i_dep = 0; i_dep < int_data[0]; i_dep++, i_elem++) {
      if (int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)] < 0) continue;
      int elem_id = int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)];
      atomicAdd_block(&(jac.loss_partials[elem_id]),rate);
    }
    for (int i_dep = 0; i_dep < int_data[1]; i_dep++, i_elem++) {
      if (int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)] < 0) continue;
      if (-rate * state[int_data[(2 + i_ind)]-1] * float_data[(7 + i_dep)] * time_step <=
          state[int_data[(2 + int_data[0]+ i_dep)]-1]) {
        int elem_id=int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)];
        atomicAdd_block(&(jac.production_partials[elem_id]),float_data[(7 + i_dep)] * rate);
      }
    }
  }
}

__device__
void rxn_gpu_CMAQ_OH_HNO3_calc_deriv_contrib(ModelDataGPU *model_data, TimeDerivativeGPU time_deriv, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  double rate = rxn_env_data[0];
  for (int i_spec=0; i_spec<int_data[0]; i_spec++) rate *= state[int_data[(2 + i_spec)]-1];
  if (rate!=0.) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<int_data[0]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0] + int_data[1] + i_dep_var)] < 0) continue;
      time_derivative_add_value_gpu(time_deriv, int_data[(2 + int_data[0] + int_data[1] + i_dep_var)], -rate);
    }
    for (int i_spec=0; i_spec<int_data[1]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0] + int_data[1] + i_dep_var)] < 0) continue;
      if (-rate*float_data[(11 + i_spec)]*time_step <= state[int_data[(2 + int_data[0] + i_spec)]-1]) {
        time_derivative_add_value_gpu(time_deriv, int_data[(2 + int_data[0] + int_data[1] + i_dep_var)],rate*float_data[(11 + i_spec)]);
      }
    }
  }
}

__device__
void rxn_gpu_CMAQ_OH_HNO3_calc_jac_contrib(ModelDataGPU *model_data, JacobianGPU jac, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  int i_elem = 0;
  for (int i_ind = 0; i_ind < int_data[0]; i_ind++) {
    double rate = rxn_env_data[0];
    for (int i_spec = 0; i_spec < int_data[0]; i_spec++)
      if (i_ind != i_spec) rate *= state[int_data[(2 + i_spec)]-1];
    for (int i_dep = 0; i_dep < int_data[0]; i_dep++, i_elem++) {
      if (int_data[(2 + 2*(int_data[0]+int_data[1]) + i_elem)] < 0) continue;
      int elem_id = int_data[(2 + 2*(int_data[0]+int_data[1]) + i_elem)];
      atomicAdd_block(&(jac.loss_partials[elem_id]),rate);
    }
    for (int i_dep = 0; i_dep < int_data[1]; i_dep++, i_elem++) {
      if (int_data[(2 + 2*(int_data[0]+int_data[1]) + i_elem)] < 0) continue;
      if (-rate * state[int_data[(2 + i_ind)]-1] * float_data[(11 + i_dep)] * time_step <=
          state[int_data[(2 + int_data[0] + i_dep)]-1]) {
        int elem_id=int_data[(2 + 2*(int_data[0]+int_data[1]) + i_elem)];
        atomicAdd_block(&(jac.production_partials[elem_id]), float_data[(11 + i_dep)] * rate);
      }
    }
  }
}

__device__
void rxn_gpu_arrhenius_calc_deriv_contrib(ModelDataGPU *model_data, TimeDerivativeGPU time_deriv,
                                int *rxn_int_data, double *rxn_float_data,
                                double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  double rate = rxn_env_data[0];
  for (int i_spec=0; i_spec<int_data[0]; i_spec++)
    rate *= state[int_data[(2 + i_spec)]-1];
  if (rate!=0.) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<int_data[0]; i_spec++, i_dep_var++) {
      if (int_data[2 + int_data[0] + int_data[1] + i_dep_var] < 0) continue;
      time_derivative_add_value_gpu(time_deriv, int_data[2 + int_data[0] + int_data[1] + i_dep_var], -rate);
    }
    for (int i_spec=0; i_spec<int_data[1]; i_spec++, i_dep_var++) {
      if (int_data[2 + int_data[0] + int_data[1] + i_dep_var] < 0) continue;
      if (-rate*float_data[6+i_spec]*time_step <= state[int_data[(2 + int_data[0] + i_spec)]-1]) {
        time_derivative_add_value_gpu(time_deriv, int_data[2 + int_data[0] + int_data[1] + i_dep_var],rate*float_data[6+i_spec]);
      }
    }
  }
}

__device__
void rxn_gpu_arrhenius_calc_jac_contrib(ModelDataGPU *model_data, JacobianGPU jac, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  int i_elem = 0;
  for (int i_ind = 0; i_ind < int_data[0]; i_ind++) {
    double rate = rxn_env_data[0];
    for (int i_spec = 0; i_spec < int_data[0]; i_spec++)
      if (i_spec != i_ind) rate *= state[int_data[(2 + i_spec)]-1];
    for (int i_dep = 0; i_dep < int_data[0]; i_dep++, i_elem++) {
      if (int_data[2 + 2*(int_data[0]+int_data[1]) + i_elem] < 0) continue;
      int elem_id = int_data[2 + 2*(int_data[0]+int_data[1]) + i_elem];
      atomicAdd_block(&(jac.loss_partials[elem_id]),rate);
    }
    for (int i_dep = 0; i_dep < int_data[1]; i_dep++, i_elem++) {
      if (int_data[2 + 2*(int_data[0]+int_data[1]) + i_elem] < 0) continue;
      if (-rate * state[int_data[(2 + i_ind)]-1] * float_data[6+i_dep] * time_step <=
        state[int_data[(2 + int_data[0] + i_dep)]-1]) {
        int elem_id=int_data[2 + 2*(int_data[0]+int_data[1]) + i_elem];
        atomicAdd_block(&(jac.production_partials[elem_id]), float_data[6+i_dep] * rate);
      }
    }
  }
}

__device__
void rxn_gpu_troe_calc_deriv_contrib(ModelDataGPU *model_data, TimeDerivativeGPU time_deriv, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  double rate = rxn_env_data[0];
  for (int i_spec=0; i_spec<int_data[0]; i_spec++)
          rate *= state[int_data[(2 + i_spec)]-1];
  if (rate!=ZERO) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<int_data[0]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0]+int_data[1]+i_dep_var)] < 0) continue;
      time_derivative_add_value_gpu(time_deriv, int_data[(2 + int_data[0]+int_data[1]+i_dep_var)], -rate);
    }
    for (int i_spec=0; i_spec<int_data[1]; i_spec++, i_dep_var++) {
      if (int_data[(2 + int_data[0]+int_data[1]+i_dep_var)] < 0) continue;
      if (-rate * float_data[(10 + i_spec)] * time_step <= state[int_data[(2 + int_data[0]+ i_spec)]-1]) {
        time_derivative_add_value_gpu(time_deriv, int_data[(2 + int_data[0]+int_data[1]+i_dep_var)],rate*float_data[(10 + i_spec)]);
      }
    }
  }
}

__device__
void rxn_gpu_photolysis_calc_deriv_contrib(ModelDataGPU *model_data, TimeDerivativeGPU time_deriv, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  double rate = rxn_env_data[0];
  for (int i_spec=0; i_spec<int_data[0]; i_spec++)
          rate *= state[int_data[(3 + i_spec)]-1];
  if (rate!=ZERO) {
    int i_dep_var = 0;
    for (int i_spec=0; i_spec<int_data[0]; i_spec++, i_dep_var++) {
      if (int_data[(3 + int_data[0]+int_data[1]+i_dep_var)] < 0) continue;
      time_derivative_add_value_gpu(time_deriv, int_data[(3 + int_data[0]+int_data[1]+i_dep_var)], -rate);
    }
    for (int i_spec=0; i_spec<int_data[1]; i_spec++, i_dep_var++) {
      if (int_data[(3 + int_data[0]+int_data[1]+i_dep_var)] < 0) continue;
        if (-rate * float_data[(1 + i_spec)] * time_step <= state[int_data[(3 + int_data[0]+ i_spec)]-1]){
        time_derivative_add_value_gpu(time_deriv, int_data[(3 + int_data[0]+int_data[1]+i_dep_var)],rate*float_data[(1 + i_spec)]);
      }
    }
  }
}

__device__
void rxn_gpu_troe_calc_jac_contrib(ModelDataGPU *model_data, JacobianGPU jac, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  int i_elem = 0;
  for (int i_ind = 0; i_ind < int_data[0]; i_ind++) {
    double rate = rxn_env_data[0];
    for (int i_spec = 0; i_spec < int_data[0]; i_spec++)
      if (i_ind != i_spec) rate *= state[int_data[(2 + i_spec)]-1];
    for (int i_dep = 0; i_dep < int_data[0]; i_dep++, i_elem++) {
      if (int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)] < 0) continue;
      int elem_id = int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)];
      atomicAdd_block(&(jac.loss_partials[elem_id]),rate);
    }
    for (int i_dep = 0; i_dep < int_data[1]; i_dep++, i_elem++) {
      if (int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)] < 0) continue;
      if (-rate * state[int_data[(2 + i_ind)]-1] * float_data[(10 + i_dep)] * time_step <=
        state[int_data[(2 + int_data[0]+ i_dep)]-1]) {
        int elem_id = (unsigned int) int_data[(2 + 2*(int_data[0]+int_data[1])+i_elem)];
        atomicAdd_block(&(jac.production_partials[elem_id]), float_data[(10 + i_dep)] * rate);
      }
    }
  }
}


__device__
void rxn_gpu_photolysis_calc_jac_contrib(ModelDataGPU *model_data, JacobianGPU jac, int *rxn_int_data,
          double *rxn_float_data, double *rxn_env_data, double time_step){
  int *int_data = rxn_int_data;
  double *float_data = rxn_float_data;
  double *state = model_data->grid_cell_state;
  int i_elem = 0;
  for (int i_ind = 0; i_ind < int_data[0]; i_ind++) {
    double rate = rxn_env_data[0];
    for (int i_spec = 0; i_spec < int_data[0]; i_spec++)
      if (i_spec != i_ind) rate *= state[int_data[(3 + i_spec)]-1];
    for (int i_dep = 0; i_dep < int_data[0]; i_dep++, i_elem++) {
      if (int_data[(3 + 2*(int_data[0]+int_data[1])+i_elem)] < 0) continue;
      int elem_id = int_data[(3 + 2*(int_data[0]+int_data[1])+i_elem)];
      atomicAdd_block(&(jac.loss_partials[elem_id]),rate);
    }
    for (int i_dep = 0; i_dep < int_data[1]; i_dep++, i_elem++) {
      if (int_data[(3 + 2*(int_data[0]+int_data[1])+i_elem)] < 0) continue;
      if (-rate * state[int_data[(3 + i_ind)]-1] * float_data[(1 + i_dep)] * time_step <=
          state[int_data[(3 + int_data[0]+ i_dep)]-1]) {
        int elem_id=int_data[(3 + 2*(int_data[0]+int_data[1])+i_elem)];
        atomicAdd_block(&(jac.production_partials[elem_id]),float_data[(1 + i_dep)] * rate);
      }
    }
  }
}

__device__ void cudaDevicemin_2(double *g_odata, double in, volatile double *sdata, int n_shr_empty){
  unsigned int tid = threadIdx.x;
  __syncthreads();
  sdata[tid] = in;
  __syncthreads();
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=sdata[tid];
  __syncthreads();
  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s){
      if(sdata[tid + s] < sdata[tid] ) sdata[tid]=sdata[tid + s];
    }
    __syncthreads();
  }
  *g_odata = sdata[0];
  __syncthreads();
}

#ifdef DEBUG_CVODE_GPU
__device__
void printmin(ModelDataGPU *md,double* y, const char *s) {
  __syncthreads();
  extern __shared__ double flag_shr2[];
  int tid= threadIdx.x + blockDim.x*blockIdx.x;
  __syncthreads();

  double min;
  cudaDevicemin_2(&min, y[tid], flag_shr2, md->n_shr_empty);
  __syncthreads();
  if(tid==0)printf("%s min %le\n",s,min);
  __syncthreads();

}
#endif

__device__ void cudaDeviceBCGprecond_2(double* dA, int* djA, int* diA, double* ddiag, double alpha){
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  int nnz=diA[blockDim.x];
  for(int j=diA[threadIdx.x];j<diA[threadIdx.x+1];j++){
    if(djA[j]==threadIdx.x){
      dA[j+nnz*blockIdx.x] = 1.0 + alpha*dA[j+nnz*blockIdx.x];
      if(dA[j+nnz*blockIdx.x]!=0.0){
        ddiag[row]= 1.0/dA[j+nnz*blockIdx.x];
       }else{
        ddiag[row]= 1.0;
      }
    }else{
      dA[j+nnz*blockIdx.x] = alpha*dA[j+nnz*blockIdx.x];
    }
  }
}

__device__ void cudaDeviceSpmv_2CSR(double* dx, double* db, double* dA, int* djA, int* diA){
  __syncthreads();
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  double sum = 0.0;
  int nnz=diA[blockDim.x];
  for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++){
    sum+= db[djA[j]+blockDim.x*blockIdx.x]*dA[j+nnz*blockIdx.x];
  }
  dx[row]=sum;
  __syncthreads();
}

__device__ void cudaDeviceSpmv_2CSC_block(double* dx, double* db, double* dA, int* djA, int* diA){
  int row = threadIdx.x + blockDim.x*blockIdx.x;
  __syncthreads();
  dx[row]=0.0;
  __syncthreads();
  int nnz=diA[blockDim.x];
  for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++){
    double mult = db[row]*dA[j+nnz*blockIdx.x];
    atomicAdd_block(&(dx[djA[j]+blockDim.x*blockIdx.x]),mult);
  }
  __syncthreads();
}

__device__ void cudaDeviceSpmv_2(double* dx, double* db, double* dA, int* djA, int* diA){
#ifndef USE_CSR_ODE_GPU
  cudaDeviceSpmv_2CSR(dx,db,dA,djA,diA);
#else
  cudaDeviceSpmv_2CSC_block(dx,db,dA,djA,diA);
#endif
}

__device__ void warpReduce_2(volatile double *sdata, unsigned int tid) {
  unsigned int blockSize = blockDim.x;
  if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
  if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
  if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
  if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
  if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
  if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

__device__ void cudaDevicedotxy_2(double *g_idata1, double *g_idata2,
                                 double *g_odata, int n_shr_empty){
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  __syncthreads();
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=0.;
  __syncthreads();
  sdata[tid] = g_idata1[i]*g_idata2[i];
  __syncthreads();
  unsigned int blockSize = blockDim.x+n_shr_empty;
  if ((blockSize >= 1024) && (tid < 512)) {
    sdata[tid] += sdata[tid + 512];
  }
  __syncthreads();
  if ((blockSize >= 512) && (tid < 256)) {
    sdata[tid] += sdata[tid + 256];
  }
  __syncthreads();
  if ((blockSize >= 256) && (tid < 128)) {
    sdata[tid] += sdata[tid + 128];
  }
  __syncthreads();
  if ((blockSize >= 128) && (tid < 64)) {
    sdata[tid] += sdata[tid + 64];
  }
  __syncthreads();
  if (tid < 32) warpReduce_2(sdata, tid);
  __syncthreads();
  *g_odata = sdata[0];
  __syncthreads();
}

__device__ void cudaDeviceVWRMS_Norm_2(double *g_idata1, double *g_idata2, double *g_odata, int n, int n_shr_empty){
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  __syncthreads();
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=0.;
  __syncthreads();
  sdata[tid] = g_idata1[i]*g_idata1[i]*g_idata2[i]*g_idata2[i];
  __syncthreads();
  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] += sdata[tid + s];

    __syncthreads();
  }
  g_odata[0] = sqrt(sdata[0]/n);
  __syncthreads();
}

__device__
void cudaDeviceJacCopy(int* Ap, double* Ax, double* Bx) {
  __syncthreads();
  int nnz=Ap[blockDim.x];
  for(int j=Ap[threadIdx.x]; j<Ap[threadIdx.x+1]; j++){
    Bx[j+nnz*blockIdx.x]=Ax[j+nnz*blockIdx.x];
  }
  __syncthreads();
}

__device__
int cudaDevicecamp_solver_check_model_state(ModelDataGPU *md, ModelDataVariable *dmdv, double *y, int *flag)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();
  extern __shared__ int flag_shr[];
  flag_shr[0] = 0;
  __syncthreads();
  if (y[tid] < -SMALL) {
    flag_shr[0] = CAMP_SOLVER_FAIL;
#ifdef DEBUG_cudaDevicecamp_solver_check_model_state
    printf("Failed model state update gpu:[spec %d] = %le flag_shr %d\n",tid,y[tid],flag_shr[0]);
#endif
  } else {
    md->state[md->map_state_deriv[tid]] =
            y[tid] <= -SMALL ?
            TINY : y[tid];
  }
  __syncthreads();
  *flag = (int)flag_shr[0];
  __syncthreads();
#ifdef DEBUG_printmin
  printmin(md,md->state,"cudaDevicecamp_solver_check_model_state end state");
#endif
#ifdef DEBUG_cudaDevicecamp_solver_check_model_state
  __syncthreads();if(tid==0)printf("flag %d flag_shr %d\n",*flag,flag_shr2[0]);
#endif
  return *flag;
}

__device__ void solveRXN(
        int i_rxn, int i_cell,TimeDerivativeGPU deriv_data,
        double time_step,
        ModelDataGPU *md, ModelDataVariable *dmdv
){
#ifdef REVERSE_INT_FLOAT_MATRIX
  double *rxn_float_data = &( md->rxn_double[i_rxn]);
  int *int_data = &(md->rxn_int[i_rxn]);
  int rxn_type = int_data[0];
  int *rxn_int_data = (int *) &(int_data[1*md->n_rxn]);
#else
  double *rxn_float_data = (double *)&( md->rxn_double[md->rxn_float_indices[i_rxn]]);
  int *int_data = (int *)&(md->rxn_int[md->rxn_int_indices[i_rxn]]);
  int rxn_type = int_data[0];
  int *rxn_int_data = (int *) &(int_data[1]);
#endif
  double *rxn_env_data = &(md->rxn_env_data
  [md->n_rxn_env_data*i_cell+md->rxn_env_data_idx[i_rxn]]);
#ifdef DEBUG_DERIV_GPU
  if(tid==0){
    printf("[DEBUG] GPU solveRXN tid %d, \n", tid);
  }
#endif
  switch (rxn_type) {
    case RXN_ARRHENIUS :
      rxn_gpu_arrhenius_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                           rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CMAQ_H2O2 :
      rxn_gpu_CMAQ_H2O2_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                           rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_CMAQ_OH_HNO3 :
      rxn_gpu_CMAQ_OH_HNO3_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                              rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_FIRST_ORDER_LOSS:
    rxn_gpu_first_order_loss_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                    rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_PHOTOLYSIS :
      rxn_gpu_photolysis_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                            rxn_float_data, rxn_env_data,time_step);
      break;
    case RXN_TROE :
      rxn_gpu_troe_calc_deriv_contrib(md, deriv_data, rxn_int_data,
                                      rxn_float_data, rxn_env_data,time_step);
      break;
  }
}

__device__ void cudaDevicecalc_deriv(double time_step, double *y,
        double *yout, ModelDataGPU *md, ModelDataVariable *dmdv)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int deriv_length_cell = md->nrows / md->n_cells;
  int tid_cell=i%deriv_length_cell;
  int state_size_cell = md->state_size_cell;
#ifdef DEBUG_DERIV_GPU
  if(i==0){
    printf("[DEBUG] GPU solveDerivative i %d, \n", i);
    printf("md->nrows %d, \n", md->nrows);
    printf("md->deriv_length_cell %d, \n", md->deriv_length_cell);
    printf("blockDim.x %d, \n", blockDim.x);
  }__syncthreads();
#endif
#ifdef DEBUG_printmin
  printmin(md,yout,"cudaDevicecalc_deriv start end yout");
  printmin(md,md->J_tmp,"cudaDevicecalc_deriv start end J_tmp");
  printmin(md,md->J_state,"cudaDevicecalc_deriv start end J_state");
#endif
  md->J_tmp[i]=y[i]-md->J_state[i];
  cudaDeviceSpmv_2(md->J_tmp2, md->J_tmp, md->J_solver, md->djA, md->diA);
  md->J_tmp[i]=md->J_deriv[i]+md->J_tmp2[i];
  md->J_tmp2[i]=0.0;
#ifdef DEBUG_printmin
    printmin(md,md->J_tmp,"cudaDevicecalc_deriv start end J_tmp");
    printmin(md,md->J_state,"cudaDevicecalc_deriv start end J_state");
#endif
    TimeDerivativeGPU deriv_data;
    deriv_data.num_spec = deriv_length_cell*gridDim.x;
#ifdef AEROS_CPU
#else
    deriv_data.production_rates = md->production_rates;
    deriv_data.loss_rates = md->loss_rates;
    if(i<deriv_data.num_spec){
      deriv_data.production_rates[i] = 0.0;
      deriv_data.loss_rates[i] = 0.0;
    }
    __syncthreads();
#endif
    int i_cell = i/deriv_length_cell;
    deriv_data.production_rates = &( md->production_rates[deriv_length_cell*i_cell]);
    deriv_data.loss_rates = &( md->loss_rates[deriv_length_cell*i_cell]);
    md->grid_cell_state = &( md->state[state_size_cell*i_cell]);
    md->grid_cell_env = &( md->env[CAMP_NUM_ENV_PARAM_*i_cell]);
    int n_rxn = md->n_rxn;
    if( tid_cell < n_rxn) {
      int n_iters = n_rxn / deriv_length_cell;
      for (int j = 0; j < n_iters; j++) {
        int i_rxn = tid_cell + j*deriv_length_cell;
        solveRXN(i_rxn, i_cell,deriv_data, time_step, md, dmdv);
      }
      int residual=n_rxn-(deriv_length_cell*n_iters);
      if(tid_cell < residual){
        int i_rxn = tid_cell + deriv_length_cell*n_iters;
        solveRXN(i_rxn, i_cell, deriv_data, time_step, md, dmdv);
      }
    }
    __syncthreads();
    deriv_data.production_rates = md->production_rates;
    deriv_data.loss_rates = md->loss_rates;
#ifdef DEBUG_printmin
    printmin(md,yout,"cudaDevicecalc_deriv start end yout");
#endif
    __syncthreads();
    double *J_tmp = md->J_tmp;
    if(i<deriv_data.num_spec){
        double *r_p = deriv_data.production_rates;
        double *r_l = deriv_data.loss_rates;
        if (r_p[i] + r_l[i] != 0.0) {
            double scale_fact;
            scale_fact = 1.0 / (r_p[i] + r_l[i]) /
                (1.0 / (r_p[i] + r_l[i]) + MAX_PRECISION_LOSS / fabs(r_p[i]- r_l[i]));
            yout[i] = scale_fact * (r_p[i] - r_l[i]) + (1.0 - scale_fact) * (J_tmp[i]);
        } else {
          yout[i] = 0.0;
        }
    }
#ifdef DEBUG_printmin
    printmin(md,yout,"cudaDevicecalc_deriv start end yout");
#endif
  __syncthreads();
}

__device__
int cudaDevicef(
        double time_step, double *y,
        double *yout, ModelDataGPU *md, ModelDataVariable *dmdv, int *flag
)
{
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  int clock_khz=md->clock_khz;
  clock_t start;
  start = clock();
#endif
#endif
#ifdef DEBUG_printmin
  printmin(md,y,"cudaDevicef Start y");
#endif
  time_step = time_step > 0. ? time_step : md->init_time_step;
#ifdef DEBUG_printmin
  printmin(md,md->state,"cudaDevicef start state");
#endif
  int checkflag=cudaDevicecamp_solver_check_model_state(md, dmdv, y, flag);
  __syncthreads();
  if(checkflag==CAMP_SOLVER_FAIL){
    *flag=CAMP_SOLVER_FAIL;
#ifdef DEBUG_printmin
    printmin(md,y,"cudaDevicef End y");
#endif
    __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadIdx.x==0) dmdv->timef += ((double)(int)(clock() - start))/(clock_khz*1000);
#endif
#endif
#ifdef DEBUG_cudaDevicef
    if(i==0)printf("cudaDevicef CAMP_SOLVER_FAIL %d\n",i);
#endif
    return CAMP_SOLVER_FAIL;
  }
#ifdef DEBUG_printmin
  printmin(md,yout,"cudaDevicef End yout");
#endif
  cudaDevicecalc_deriv(time_step, y,
          yout, md, dmdv
  );
  //printmin(md,yout,"cudaDevicef End yout");
  //printmin(md,y,"cudaDevicef End y");
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadIdx.x==0) dmdv->timef += ((double)(int)(clock() - start))/(clock_khz*1000);
#endif
#endif
  __syncthreads();
  *flag=0;
  __syncthreads();
  return 0;
}

__device__
int CudaDeviceguess_helper(double cv_tn, double cv_h, double* y_n,
                           double* y_n1, double* hf, double* dtempv1,
                           double* dtempv2, int *flag,
                           ModelDataGPU *md, ModelDataVariable *dmdv
) {
  extern __shared__ double flag_shr2[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;
  int n_shr_empty = md->n_shr_empty;
#ifdef DEBUG_CudaDeviceguess_helper
  if(i==0)printf("CudaDeviceguess_helper start gpu\n");
#endif
  __syncthreads();
  double min;
  cudaDevicemin_2(&min, y_n[i], flag_shr2, n_shr_empty);
#ifdef DEBUG_CudaDeviceguess_helper
  if(i==0)printf("min %le -SMALL %le\n",min, -SMALL);
#endif
  if(min>-SMALL){
#ifdef DEBUG_CudaDeviceguess_helper
    if(i==0)printf("Return 0 %le\n",y_n[i]);
#endif
    return 0;
  }
  __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  int clock_khz=md->clock_khz;
  clock_t start;
  start = clock();
#endif
#endif
  dtempv1[i]=y_n1[i];
  __syncthreads();
  if (cv_h > 0.) {
    dtempv2[i]=(1./cv_h)*hf[i];
  } else {
    dtempv2[i]=hf[i];
  }
  double t_0 = cv_h > 0. ? cv_tn - cv_h : cv_tn - 1.;
  double t_j = 0.;
  int GUESS_MAX_ITER = 5; //5 //reduce this to reduce time
  __syncthreads();
  for (int iter = 0; iter < GUESS_MAX_ITER && t_0 + t_j < cv_tn; iter++) {
    __syncthreads();
    double h_j = cv_tn - (t_0 + t_j);
    __syncthreads();
    double t_star;
    double h_j_init=h_j;
    if(dtempv2[i]==0){
      t_star=h_j;
    }else{
      t_star = -dtempv1[i] / dtempv2[i];
    }
    if( !(t_star > 0. || (t_star == 0. && dtempv2[i] < 0.)) ){//&&dtempv2[i]==0.)
      t_star=h_j;
    }
    __syncthreads();
    flag_shr2[tid]=h_j_init;
    cudaDevicemin_2(&h_j, t_star, flag_shr2, n_shr_empty);
    flag_shr2[0]=1;
    __syncthreads();
#ifdef DEBUG_CudaDeviceguess_helper
    //if(tid==0 && iter<=5) printf("CudaDeviceguess_helper h_j %le h_j_init %le t_star %le block %d iter %d\n",h_j,h_j_init,t_star,blockIdx.x,iter);
#endif
    if (cv_h > 0.)
      h_j *= 0.95 + 0.1 * iter / (double)GUESS_MAX_ITER;
    h_j = cv_tn < t_0 + t_j + h_j ? cv_tn - (t_0 + t_j) : h_j;
    __syncthreads();
    if (cv_h == 0. &&
        cv_tn - (h_j + t_j + t_0) > md->cv_reltol) {
#ifdef DEBUG_CudaDeviceguess_helper
      if(i==0)printf("CudaDeviceguess_helper small changes \n");
#endif
      __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) dmdv->timeguess_helper += ((double)(clock() - start))/(clock_khz*1000);
#endif
#endif
    return -1;
    }
    dtempv1[i]+=h_j*dtempv2[i];
    __syncthreads();
    t_j += h_j;
#ifdef DEBUG_CudaDeviceguess_helper
    //  printf("dcorr[%d] %le dhf %le dt_star %le dh_j %le dh_n %le\n",
    //         i,dtempv2[i],hf[i],t_star,h_j,cv_h);
    //if(i==0)
    //  for(int j=0;j<nrows;j++)
    //    printf("dcorr[%d] %le dtmp1 %le dhf %le dt_star %le dh_j %le dh_n %le\n",
    //           j,dtempv2[j],dtempv1[j],hf[j],t_star,h_j,cv_h);
#endif
#ifdef DEBUG_printmin
    printmin(md,md->state,"cudaDevicef start state");
#endif
    int aux_flag=0;
    int fflag=cudaDevicef(
            t_0 + t_j, dtempv1, dtempv2,md,dmdv,&aux_flag);
#ifdef DEBUG_printmin
    printmin(md,dtempv1,"cudaDevicef end dtempv1");
#endif
    __syncthreads();
    if (fflag == CAMP_SOLVER_FAIL) {
      dtempv2[i] = 0.;
#ifdef DEBUG_CudaDeviceguess_helper
      if(i==0)printf("CudaDeviceguess_helper df(t)\n");
#endif
      __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) dmdv->timeguess_helper += ((double)(clock() - start))/(clock_khz*1000);
#endif
#endif
     return -1;
    }
    if (iter == GUESS_MAX_ITER - 1 && t_0 + t_j < cv_tn) {
      if (cv_h == 0.){
        __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) dmdv->timeguess_helper += ((double)(clock() - start))/(clock_khz*1000);
#endif
#endif
        return -1;
      }
    }
    __syncthreads();
  }
  __syncthreads();
#ifdef DEBUG_CudaDeviceguess_helper
   if(i==0)printf("CudaDeviceguess_helper return 1\n");
#endif
  dtempv2[i]=dtempv1[i]-y_n[i];
  if (cv_h > 0.) dtempv2[i]=dtempv2[i]*0.999;
  hf[i]=dtempv1[i]-y_n1[i];
  __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0)  dmdv->timeguess_helper += ((double)(clock() - start))/(clock_khz*1000);
#endif
#endif
  __syncthreads();
  return 1;
}

__device__ void solveRXNJac(
        int i_rxn, int i_cell, JacobianGPU jac,
        ModelDataGPU *md, ModelDataVariable *dmdv
){
  double cv_next_h = dmdv->cv_next_h;
#ifdef REVERSE_INT_FLOAT_MATRIX
  double *rxn_float_data = &( md->rxn_double[i_rxn]);
  int *int_data = &(md->rxn_int[i_rxn]);
  int rxn_type = int_data[0];
  int *rxn_int_data = (int *) &(int_data[1*md->n_rxn]);
#else
  double *rxn_float_data = (double *)&( md->rxn_double[md->rxn_float_indices[i_rxn]]);
  int *int_data = (int *)&(md->rxn_int[md->rxn_int_indices[i_rxn]]);
  int rxn_type = int_data[0];
  int *rxn_int_data = (int *) &(int_data[1]);
#endif
  double *rxn_env_data = &(md->rxn_env_data
  [md->n_rxn_env_data*i_cell+md->rxn_env_data_idx[i_rxn]]);
#ifdef DEBUG_solveRXNJac
  if(tid==0){
    printf("[DEBUG] GPU solveRXN tid %d, \n", tid);
  }
#endif
  switch (rxn_type) {
    case RXN_ARRHENIUS :
      rxn_gpu_arrhenius_calc_jac_contrib(md, jac, rxn_int_data,
                                         rxn_float_data, rxn_env_data,cv_next_h);
      break;
    case RXN_CMAQ_H2O2 :
      rxn_gpu_CMAQ_H2O2_calc_jac_contrib(md, jac, rxn_int_data,
                                         rxn_float_data, rxn_env_data,cv_next_h);
      break;
    case RXN_CMAQ_OH_HNO3 :
      rxn_gpu_CMAQ_OH_HNO3_calc_jac_contrib(md, jac, rxn_int_data,
                                            rxn_float_data, rxn_env_data,cv_next_h);
      break;
  case RXN_FIRST_ORDER_LOSS :
    rxn_gpu_first_order_loss_calc_jac_contrib(md, jac, rxn_int_data,
                                        rxn_float_data, rxn_env_data,cv_next_h);
    break;
    case RXN_PHOTOLYSIS :
      rxn_gpu_photolysis_calc_jac_contrib(md, jac, rxn_int_data,
                                          rxn_float_data, rxn_env_data,cv_next_h);
      break;
    case RXN_TROE :
      rxn_gpu_troe_calc_jac_contrib(md, jac, rxn_int_data,
                                    rxn_float_data, rxn_env_data,cv_next_h);
      break;
  }
}

__device__ void cudaDevicecalc_Jac(double *y,ModelDataGPU *md, ModelDataVariable *dmdv
){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int deriv_length_cell = md->nrows / md->n_cells;
  int state_size_cell = md->state_size_cell;
  int tid_cell=tid%deriv_length_cell;
  int active_threads = md->nrows;
  __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  int clock_khz=md->clock_khz;
  clock_t start;
  start = clock();
#endif
#endif
#ifdef DEBUG_cudaDeviceJac
  if(tid==0){
    printf("[DEBUG] GPU solveDerivative tid %d, \n", tid);
  }__syncthreads();
#endif
  if(tid<active_threads){
    __syncthreads();
    JacobianGPU *jac = &md->jac;
    JacobianGPU jacBlock;
    __syncthreads();
    int i_cell = tid/deriv_length_cell;
    jacBlock.num_elem = jac->num_elem;
    jacBlock.production_partials = &( jac->production_partials[jacBlock.num_elem[0]*blockIdx.x]);
    jacBlock.loss_partials = &( jac->loss_partials[jacBlock.num_elem[0]*blockIdx.x]);
    __syncthreads();
    md->grid_cell_state = &( md->state[state_size_cell*i_cell]);
    md->grid_cell_env = &( md->env[CAMP_NUM_ENV_PARAM_*i_cell]);
#ifdef DEBUG_cudaDevicecalc_Jac
    if(tid==0)printf("cudaDevicecalc_Jac01\n");
#endif
    __syncthreads();
    int n_rxn = md->n_rxn;
    if( tid_cell < n_rxn) {
      int n_iters = n_rxn / deriv_length_cell;
      for (int j = 0; j < n_iters; j++) {
        int i_rxn = tid_cell + j*deriv_length_cell;
        solveRXNJac(i_rxn,i_cell,jacBlock, md, dmdv);
      }
      int residual=n_rxn-(deriv_length_cell*n_iters);
      if(tid_cell < residual){
        int i_rxn = tid_cell + deriv_length_cell*n_iters;
        solveRXNJac(i_rxn,i_cell,jacBlock, md, dmdv);
      }
    }
    __syncthreads();
  JacMap *jac_map = md->jac_map;
  int nnz = md->n_mapped_values[0];
  int n_iters = nnz / blockDim.x;
  for (int z = 0; z < n_iters; z++) {
    int j = threadIdx.x + z*blockDim.x;
    md->dA[jac_map[j].solver_id + nnz * blockIdx.x] =
    jacBlock.production_partials[jac_map[j].rxn_id] - jacBlock.loss_partials[jac_map[j].rxn_id];
    jacBlock.production_partials[jac_map[j].rxn_id] = 0.0;
    jacBlock.loss_partials[jac_map[j].rxn_id] = 0.0;
  }
  int residual=nnz-(blockDim.x*n_iters);
  if(threadIdx.x < residual){
    int j = threadIdx.x + n_iters*blockDim.x;
  md->dA[jac_map[j].solver_id + nnz * blockIdx.x] =
      jacBlock.production_partials[jac_map[j].rxn_id] - jacBlock.loss_partials[jac_map[j].rxn_id];
    jacBlock.production_partials[jac_map[j].rxn_id] = 0.0;
    jacBlock.loss_partials[jac_map[j].rxn_id] = 0.0;
  }
    __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) dmdv->timecalc_Jac += ((double)(clock() - start))/(clock_khz*1000);
#endif
#endif
  }
}

__device__
int cudaDeviceJac(int *flag, ModelDataGPU *md, ModelDataVariable *dmdv
) //Interface CPU/GPU
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double* dftemp = md->dftemp;
  double* dcv_y = md->dcv_y;
  __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  int clock_khz=md->clock_khz;
  clock_t start;
  start = clock();
#endif
#endif
#ifdef DEBUG_printmin
  printmin(md,md->state,"cudaDeviceJac start state");
#endif
  int aux_flag=0;
  int retval=cudaDevicef(
          dmdv->cv_next_h, dcv_y, dftemp,md,dmdv,&aux_flag
  );__syncthreads();
#ifdef DEBUG_cudaDevicef
  printmin(md,dftemp,"cudaDeviceJac dftemp");
#endif
  if(retval==CAMP_SOLVER_FAIL)
    return CAMP_SOLVER_FAIL;
#ifdef DEBUG_printmin
  printmin(md,dcv_y,"cudaDeviceJac dcv_y");
  printmin(md,md->state,"cudaDeviceJac start state");
#endif
  //debug
/*
  int checkflag=cudaDevicecamp_solver_check_model_state(md, dmdv, dcv_y, flag);
  __syncthreads();
  if(checkflag==CAMP_SOLVER_FAIL){
    *flag=CAMP_SOLVER_FAIL;
    //printf("cudaDeviceJac cudaDevicecamp_solver_check_model_state *flag==CAMP_SOLVER_FAIL\n");
    //printmin(md,dcv_y,"cudaDeviceJac end dcv_y");
    return CAMP_SOLVER_FAIL;
  }
*/
#ifdef DEBUG_printmin
  printmin(md,dcv_y,"cudaDeviceJac end dcv_y");
#endif
  //printmin(md,dftemp,"cudaDeviceJac end dftemp");
  cudaDevicecalc_Jac(dcv_y,md, dmdv);
  __syncthreads();
#ifdef DEBUG_printmin
 printmin(md,dftemp,"cudaDevicecalc_Jac end dftemp");
#endif
    __syncthreads();
  int nnz = md->n_mapped_values[0];
  int n_iters = nnz / blockDim.x;
  for (int z = 0; z < n_iters; z++) {
    int j = threadIdx.x + z*blockDim.x;
    md->J_solver[j]=md->dA[j];
  }
  int residual=nnz-(blockDim.x*n_iters);
  if(threadIdx.x < residual){
    int j = threadIdx.x + n_iters*blockDim.x;
    md->J_solver[j]=md->dA[j];
  }
    __syncthreads();
    md->J_state[tid]=dcv_y[tid];
    md->J_deriv[tid]=dftemp[tid];
  __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    //if(tid==0)printf("dmdv->timeJac %lf\n",dmdv->timeJac);
    if(threadIdx.x==0)  dmdv->timeJac += ((double)(clock() - start))/(clock_khz*1000);
#endif
#endif
  __syncthreads();
  *flag = 0;
  __syncthreads();
  return 0;
}

__device__
int cudaDevicelinsolsetup(
        ModelDataGPU *md, ModelDataVariable *dmdv,
        int convfail
) {
  extern __shared__ int flag_shr[];
  double* dA = md->dA;
  int* djA = md->djA;
  int* diA = md->diA;
  double* ddiag = md->ddiag;
  double* dsavedJ = md->dsavedJ;
  double dgamma;
  int jbad, jok;
#ifdef DEBUG_printmin
  printmin(md,dcv_y,"cudaDevicelinsolsetup Start dcv_y");
#endif
  dgamma = fabs((dmdv->cv_gamma / dmdv->cv_gammap) - 1.);//SUNRabs
  jbad = (dmdv->cv_nst == 0) ||
         (dmdv->cv_nst > dmdv->nstlj + CVD_MSBJ) ||
         ((convfail == CV_FAIL_BAD_J) && (dgamma < CVD_DGMAX)) ||
         (convfail == CV_FAIL_OTHER);
  jok = !jbad;
  if (jok==1) {
    __syncthreads();
    dmdv->cv_jcur = 0;
    cudaDeviceJacCopy(diA, dsavedJ, dA);
    __syncthreads();
  } else {
  __syncthreads();
    dmdv->nstlj = dmdv->cv_nst;
    dmdv->cv_jcur = 1;
  __syncthreads();
    int aux_flag=0;
    int guess_flag=cudaDeviceJac(&aux_flag,md,dmdv);
    __syncthreads();
    if (guess_flag < 0) {
      return -1;}
    if (guess_flag > 0) {
      return 1;}
   cudaDeviceJacCopy(diA, dA, dsavedJ);
  }
  __syncthreads();
  cudaDeviceBCGprecond_2(dA, djA, diA, ddiag, -dmdv->cv_gamma);
  __syncthreads();
  return 0;
}

__device__
void solveBcgCudaDeviceCVODE(ModelDataGPU *md, ModelDataVariable *dmdv)
{
#ifdef DEBUG_printmin
  printmin(md,dtempv,"solveBcgCudaDeviceCVODEStart dtempv");
#endif
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double* dA = md->dA;
  int* djA = md->djA;
  int* diA = md->diA;
  double* dx = md->dx;
  double* dtempv = md->dtempv;
  int n_shr_empty = md->n_shr_empty;
  int maxIt = md->maxIt;
  double tolmax = md->tolmax;
  double* ddiag = md->ddiag;
  double* dr0 = md->dr0;
  double* dr0h = md->dr0h;
  double* dn0 = md->dn0;
  double* dp0 = md->dp0;
  double* dt = md->dt;
  double* ds = md->ds;
  double* dy = md->dy;
  double alpha,rho0,omega0,beta,rho1,temp1,temp2;
  alpha=rho0=omega0=beta=rho1=temp1=temp2=1.0;
  dn0[i]=0.0;
  dp0[i]=0.0;
  cudaDeviceSpmv_2(dr0,dx,dA,djA,diA);
  dr0[i]=dtempv[i]-dr0[i];
  dr0h[i]=dr0[i];
  int it=0;
  do{
    cudaDevicedotxy_2(dr0, dr0h, &rho1, n_shr_empty);
    beta = (rho1 / rho0) * (alpha / omega0);
    dp0[i]=beta*dp0[i]+dr0[i]-dn0[i]*omega0*beta;
    dy[i]=ddiag[i]*dp0[i];
    cudaDeviceSpmv_2(dn0, dy, dA, djA, diA);
    cudaDevicedotxy_2(dr0h, dn0, &temp1, n_shr_empty);
    alpha = rho1 / temp1;
    ds[i]=dr0[i]-alpha*dn0[i];
    dx[i]+=alpha*dy[i];
    dy[i]=ddiag[i]*ds[i];
    cudaDeviceSpmv_2(dt, dy, dA, djA, diA);
    dr0[i]=ddiag[i]*dt[i];
    cudaDevicedotxy_2(dy, dr0, &temp1, n_shr_empty);
    cudaDevicedotxy_2(dr0, dr0, &temp2, n_shr_empty);
    omega0 = temp1 / temp2;
    dx[i]+=omega0*dy[i];
    dr0[i]=ds[i]-omega0*dt[i];
    dt[i]=0.0;
    cudaDevicedotxy_2(dr0, dr0, &temp1, n_shr_empty);
    temp1 = sqrtf(temp1);
    rho0 = rho1;
    it++;
  } while(it<maxIt && temp1>tolmax);//while(it<maxIt && temp1>tolmax);//while(0);
  __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  if(threadIdx.x==0) dmdv->counterBCGInternal += it;
  if(threadIdx.x==0) dmdv->counterBCG++;
#endif
#endif
}

__device__
int cudaDevicecvNewtonIteration(ModelDataGPU *md, ModelDataVariable *dmdv){
  extern __shared__ double flag_shr2[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int aux_flag=0;
  double* dx = md->dx;
  double* dtempv = md->dtempv;
  int nrows = md->nrows;
  double cv_tn = dmdv->cv_tn;
  double* dftemp = md->dftemp;
  double* dcv_y = md->dcv_y;
  double* dtempv1 = md->dtempv1;
  double* dtempv2 = md->dtempv2;
  double cv_next_h = dmdv->cv_next_h;
  int n_shr_empty = md->n_shr_empty;
  double* cv_acor = md->cv_acor;
  double* dzn = md->dzn;
  double* dewt = md->dewt;
  double del, delp, dcon, m;
  del = delp = 0.0;
  __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  int clock_khz=md->clock_khz;
  clock_t start;
#endif
#endif
#ifdef DEBUG_printmin
  printmin(md,dtempv,"cudaDevicecvNewtonIterationStart dtempv");
#endif
  for(;;) {
#ifdef DEBUG_printmin
    printmin(md,dftemp,"cudaDevicecvNewtonIteration dftemp");
#endif
    __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    start = clock();
#endif
#endif
    dtempv[i]=dmdv->cv_rl1*(dzn[i+nrows])+cv_acor[i];
    dtempv[i]=dmdv->cv_gamma*dftemp[i]-dtempv[i];
    solveBcgCudaDeviceCVODE(md, dmdv);
    __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) dmdv->dtBCG += ((double)(int)(clock() - start))/(clock_khz*1000);
#endif
#endif
    __syncthreads();
    dtempv[i] = dx[i];
    __syncthreads();
#ifdef DEBUG_printmin
    printmin(md,dcv_y,"cudaDevicecvNewtonIteration dcv_y");
    printmin(md,dtempv,"cudaDevicecvNewtonIteration dtempv");
#endif
    dftemp[i]=dcv_y[i]+dtempv[i];
#ifdef DEBUG_cudaDevicecvNewtonIteration
    //if(i==0)printf("cudaDevicecvNewtonIteration dftemp %le dtempv %le dcv_y %le it %d block %d\n",
    //               dftemp[(blockDim.x-1)*0],dtempv[(blockDim.x-1)*0],dcv_y[(blockDim.x-1)*0],it,blockIdx.x);
#endif
#ifdef DEBUG_printmin
    printmin(md,dftemp,"cudaDevicecvNewtonIteration dftemp");
#endif
    __syncthreads();
    int guessflag=CudaDeviceguess_helper(cv_tn, 0., dftemp,
                           dcv_y, dtempv, dtempv1,
                           dtempv2, &aux_flag, md, dmdv
    );
    __syncthreads();
    if (guessflag < 0) {
      if (!(dmdv->cv_jcur)) { //Bool set up during linsolsetup just before Jacobian
        return TRY_AGAIN;
      } else {
        return RHSFUNC_RECVR;
      }
    }
    dftemp[i]=dcv_y[i]+dtempv[i];
    double min;
    cudaDevicemin_2(&min, dftemp[i], flag_shr2, md->n_shr_empty);
    if (min < -CAMP_TINY) {
      //if (dftemp[i] < -CAMP_TINY) {
      return CONV_FAIL;
    }
    __syncthreads();
    cv_acor[i]+=dx[i];
    dcv_y[i]=dzn[i]+cv_acor[i];
    cudaDeviceVWRMS_Norm_2(dx, dewt, &del, nrows, n_shr_empty);
    if (m > 0) {
      dmdv->cv_crate = SUNMAX(0.3 * dmdv->cv_crate, del / delp);
    }
    dcon = del * SUNMIN(1.0, dmdv->cv_crate) / md->cv_tq[4+blockIdx.x*(NUM_TESTS + 1)];
    flag_shr2[0]=0;
    __syncthreads();
    if (dcon <= 1.0) {
      cudaDeviceVWRMS_Norm_2(cv_acor, dewt, &dmdv->cv_acnrm, nrows, n_shr_empty);
      __syncthreads();
      dmdv->cv_jcur = 0;
      __syncthreads();
      return CV_SUCCESS;
    }
    if ((m == md->cv_maxcor) || ((m >= 2) && (del > RDIV * delp))) {
      if (!(dmdv->cv_jcur)) {
        return TRY_AGAIN;
      } else {
        return RHSFUNC_RECVR;
      }
    }
    delp = del;
    __syncthreads();
#ifdef DEBUG_printmin
    printmin(md,md->state,"cudaDevicef start state");
#endif
    int retval=cudaDevicef(
            cv_next_h, dcv_y, dftemp, md, dmdv, &aux_flag
    );
    __syncthreads();
    cv_acor[i]=dcv_y[i]+dzn[i];
    if (retval < 0) {
      return CV_RHSFUNC_FAIL;
    }
    if (retval > 0) {
      if (!(dmdv->cv_jcur)) {
        return TRY_AGAIN;
      } else {
        return RHSFUNC_RECVR;
      }
    }
    __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) dmdv->dtPostBCG += ((double)(clock() - start))/(clock_khz*1000);
#endif
#endif
#ifdef DEBUG_cudaDevicecvNewtonIteration
    if(i==0)printf("cudaDevicecvNewtonIteration dzn[(blockDim.x*(blockIdx.x+1)-1)*0] %le it %d block %d\n",dzn[(blockDim.x*(blockIdx.x+1)-1)*0],it,blockIdx.x);
#endif
  }
}

__device__
int cudaDevicecvNlsNewton(int nflag,
        ModelDataGPU *md, ModelDataVariable *dmdv
) {
  extern __shared__ int flag_shr[];
  int flagDevice = 0;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double* dcv_y = md->dcv_y;
  double* cv_acor = md->cv_acor;
  double* dzn = md->dzn;
  double* dftemp = md->dftemp;
  double cv_tn = dmdv->cv_tn;
  double cv_h = dmdv->cv_h;
  double* dtempv = md->dtempv;
  double cv_next_h = dmdv->cv_next_h;
#ifdef DEBUG_printmin
  printmin(md,dtempv,"cudaDevicecvNlsNewtonStart dtempv");
#endif
  __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  int clock_khz=md->clock_khz;
  clock_t start;
#endif
#endif
  int convfail = ((nflag == FIRST_CALL) || (nflag == PREV_ERR_FAIL)) ?
                 CV_NO_FAILURES : CV_FAIL_OTHER;
  int dgamrat=fabs(dmdv->cv_gamrat - 1.);
  int callSetup = (nflag == PREV_CONV_FAIL) || (nflag == PREV_ERR_FAIL) ||
                  (dmdv->cv_nst == 0) ||
                  (dmdv->cv_nst >= dmdv->cv_nstlp + MSBP) ||
                  (dgamrat > DGMAX);
  dftemp[i]=dzn[i]+(-md->cv_last_yn[i]);
  __syncthreads();
  int guessflag=CudaDeviceguess_helper(cv_tn, cv_h, dzn,
             md->cv_last_yn, dftemp, dtempv,
             md->cv_acor_init,  &flagDevice,
             md, dmdv
  );
  __syncthreads();
#ifdef DEBUG_printmin
  printmin(md,dtempv,"cudaDevicecvSet after guess_helper dtempv");
#endif
  if(guessflag<0){
    return RHSFUNC_RECVR;
  }
  for(;;) {
    __syncthreads();
    dcv_y[i] = dzn[i];
#ifdef DEBUG_printmin
    //printmin(md,md->state,"cudaDevicef start state");
#endif
    int aux_flag=0;
    int retval=cudaDevicef(cv_next_h, dcv_y,
            dftemp,md,dmdv,&aux_flag
    );
    if (retval < 0) {
      return CV_RHSFUNC_FAIL;
    }
    if (retval> 0) {
      return RHSFUNC_RECVR;
    }
    if (callSetup==1) {
      __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
      start = clock();
#endif
#endif
      __syncthreads();
      int linflag=cudaDevicelinsolsetup(md, dmdv,convfail);
      __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
      if(threadIdx.x==0) dmdv->timelinsolsetup += ((double)(clock() - start))/(clock_khz*1000);
#endif
#endif
      callSetup = 0;
      dmdv->cv_gamrat = dmdv->cv_crate = 1.0;
      dmdv->cv_gammap = dmdv->cv_gamma;
      //if(threadIdx.x==0)
        dmdv->cv_nstlp = dmdv->cv_nst;
      __syncthreads();
      if (linflag < 0) {
        flag_shr[0] = CV_LSETUP_FAIL;
        break;
      }
      if (linflag > 0) {
        flag_shr[0] = CONV_FAIL;
        break;
      }
    }
    __syncthreads();
    cv_acor[i] = 0.0;
    __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    start = clock();
#endif
#endif
    __syncthreads();
    int nItflag=cudaDevicecvNewtonIteration(md, dmdv);
    __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0)  dmdv->timeNewtonIteration += ((double)(clock() - start))/(clock_khz*1000);
#endif
#endif
    if (nItflag != TRY_AGAIN) {
      return nItflag;
    }
    __syncthreads();
    callSetup = 1;
    __syncthreads();
    convfail = CV_FAIL_BAD_J;
    __syncthreads();
  } //for(;;)
  __syncthreads();
  return nflag;
}

__device__
void cudaDevicecvRescale(ModelDataGPU *md, ModelDataVariable *dmdv) {
  extern __shared__ double dzn[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double factor;
  __syncthreads();
  factor = dmdv->cv_eta;
  for (int j=1; j <= dmdv->cv_q; j++) {
    md->dzn[i+md->nrows*j]*=factor;
    __syncthreads();
    factor *= dmdv->cv_eta;
    __syncthreads();
  }
  dmdv->cv_h = dmdv->cv_hscale * dmdv->cv_eta;
  dmdv->cv_next_h = dmdv->cv_h;
  dmdv->cv_hscale = dmdv->cv_h;
  __syncthreads();
}

__device__
void cudaDevicecvRestore(ModelDataGPU *md, ModelDataVariable *dmdv, double saved_t) {
  extern __shared__ double dzn[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j, k;
  __syncthreads();
  dmdv->cv_tn=saved_t;
  for (k = 1; k <= dmdv->cv_q; k++){
    for (j = dmdv->cv_q; j >= k; j--) {
      md->dzn[i+md->nrows*(j-1)]-=md->dzn[i+md->nrows*j];
    }
  }
  md->dzn[i]=md->cv_last_yn[i];
  __syncthreads();
}

__device__
int cudaDevicecvHandleNFlag(ModelDataGPU *md, ModelDataVariable *dmdv, int *nflagPtr, double saved_t,
                             int *ncfPtr) {
  extern __shared__ int flag_shr[];
  if (*nflagPtr == CV_SUCCESS){
    return(DO_ERROR_TEST);
  }
  cudaDevicecvRestore(md, dmdv, saved_t);
  if (*nflagPtr == CV_LSETUP_FAIL)  return(CV_LSETUP_FAIL);
  if (*nflagPtr == CV_LSOLVE_FAIL)  return(CV_LSOLVE_FAIL);
  if (*nflagPtr == CV_RHSFUNC_FAIL) return(CV_RHSFUNC_FAIL);
  (*ncfPtr)++;
  dmdv->cv_etamax = 1.;
  __syncthreads();
  if ((fabs(dmdv->cv_h) <= dmdv->cv_hmin*ONEPSM) ||
      (*ncfPtr == dmdv->cv_maxncf)) {
    if (*nflagPtr == CONV_FAIL)     return(CV_CONV_FAILURE);
    if (*nflagPtr == RHSFUNC_RECVR) return(CV_REPTD_RHSFUNC_ERR);
  }
  __syncthreads();
  dmdv->cv_eta = SUNMAX(ETACF,
          dmdv->cv_hmin / fabs(dmdv->cv_h));
  __syncthreads();
  *nflagPtr = PREV_CONV_FAIL;
  cudaDevicecvRescale(md, dmdv);
  __syncthreads();
  return (PREDICT_AGAIN);
}

__device__
void cudaDevicecvSetTqBDFt(ModelDataGPU *md, ModelDataVariable *dmdv,
                           double hsum, double alpha0,
                           double alpha0_hat, double xi_inv, double xistar_inv) {
  extern __shared__ int flag_shr[];
  double A1, A2, A3, A4, A5, A6;
  double C, Cpinv, Cppinv;
  __syncthreads();
  A1 = 1. - alpha0_hat + alpha0;
  A2 = 1. + dmdv->cv_q * A1;
  md->cv_tq[2+blockIdx.x*(NUM_TESTS + 1)] = fabs(A1 / (alpha0 * A2));
  md->cv_tq[5+blockIdx.x*(NUM_TESTS + 1)] = fabs(A2 * xistar_inv / (md->cv_l[dmdv->cv_q+blockIdx.x*L_MAX] * xi_inv));
  if (dmdv->cv_qwait == 1) {
    if (dmdv->cv_q > 1) {
      C = xistar_inv / md->cv_l[dmdv->cv_q+blockIdx.x*L_MAX];
      A3 = alpha0 + 1. / dmdv->cv_q;
      A4 = alpha0_hat + xi_inv;
      Cpinv = (1. - A4 + A3) / A3;
      md->cv_tq[1+blockIdx.x*(NUM_TESTS + 1)] = fabs(C * Cpinv);
    }
    else md->cv_tq[1+blockIdx.x*(NUM_TESTS + 1)] = 1.;
    __syncthreads();
    hsum += md->cv_tau[dmdv->cv_q+blockIdx.x*(L_MAX + 1)];
    xi_inv = dmdv->cv_h / hsum;
    A5 = alpha0 - (1. / (dmdv->cv_q+1));
    A6 = alpha0_hat - xi_inv;
    Cppinv = (1. - A6 + A5) / A2;
    md->cv_tq[3+blockIdx.x*(NUM_TESTS + 1)] = fabs(Cppinv / (xi_inv * (dmdv->cv_q+2) * A5));
    __syncthreads();
  }
  md->cv_tq[4+blockIdx.x*(NUM_TESTS + 1)] = md->cv_nlscoef / md->cv_tq[2+blockIdx.x*(NUM_TESTS + 1)];
}

__device__
void cudaDevicecvSetBDF(ModelDataGPU *md, ModelDataVariable *dmdv) {
  extern __shared__ int flag_shr[];
  double alpha0, alpha0_hat, xi_inv, xistar_inv, hsum;
  int z,j;
  __syncthreads();
  md->cv_l[0+blockIdx.x*L_MAX] = md->cv_l[1+blockIdx.x*L_MAX] = xi_inv = xistar_inv = 1.;
  for (z=2; z <= dmdv->cv_q; z++) md->cv_l[z+blockIdx.x*L_MAX] = 0.;
  alpha0 = alpha0_hat = -1.;
  hsum = dmdv->cv_h;
  __syncthreads();
  if (dmdv->cv_q > 1) {
    for (j=2; j < dmdv->cv_q; j++) {
      hsum += md->cv_tau[j-1+blockIdx.x*(L_MAX + 1)];
      xi_inv = dmdv->cv_h / hsum;
      alpha0 -= 1. / j;
      for (z=j; z >= 1; z--) md->cv_l[z+blockIdx.x*L_MAX] += md->cv_l[z-1+blockIdx.x*L_MAX]*xi_inv;
    }
    __syncthreads();
    alpha0 -= 1. / dmdv->cv_q;
    xistar_inv = -md->cv_l[1+blockIdx.x*L_MAX] - alpha0;
    hsum += md->cv_tau[dmdv->cv_q-1+blockIdx.x*(L_MAX + 1)];
    xi_inv = dmdv->cv_h / hsum;
    alpha0_hat = -md->cv_l[1+blockIdx.x*L_MAX] - xi_inv;
    for (z=dmdv->cv_q; z >= 1; z--)
      md->cv_l[z+blockIdx.x*L_MAX] += md->cv_l[z-1+blockIdx.x*L_MAX]*xistar_inv;
  }
  __syncthreads();
  cudaDevicecvSetTqBDFt(md, dmdv, hsum, alpha0, alpha0_hat, xi_inv, xistar_inv);
}

__device__
void cudaDevicecvSet(ModelDataGPU *md, ModelDataVariable *dmdv) {
  extern __shared__ int flag_shr[];
#ifdef DEBUG_printmin
  printmin(md,md->dtempv,"cudaDevicecvSet Start dtempv");
#endif
  __syncthreads();
  cudaDevicecvSetBDF(md,dmdv);
  __syncthreads();
  dmdv->cv_rl1 = 1.0 / md->cv_l[1+blockIdx.x*L_MAX];
  dmdv->cv_gamma = dmdv->cv_h * dmdv->cv_rl1;
  __syncthreads();
  if (dmdv->cv_nst == 0){
    dmdv->cv_gammap = dmdv->cv_gamma;
  }
  __syncthreads();
  dmdv->cv_gamrat = (dmdv->cv_nst > 0) ?
                    dmdv->cv_gamma / dmdv->cv_gammap : 1.;  // protect x / x != 1.0
  __syncthreads();
}

__device__
void cudaDevicecvPredict(ModelDataGPU *md, ModelDataVariable *dmdv) {
  extern __shared__ double dzn[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j, k;
#ifdef DEBUG_printmin
  printmin(md,md->dtempv,"cudaDevicecvPredict start dtempv");
#endif
  __syncthreads();
  dmdv->cv_tn += dmdv->cv_h;
  __syncthreads();
  if (md->cv_tstopset) {
    if ((dmdv->cv_tn - md->cv_tstop)*dmdv->cv_h > 0.)
      dmdv->cv_tn = md->cv_tstop;
  }
  md->cv_last_yn[i]=md->dzn[i];
  for (k = 1; k <= dmdv->cv_q; k++){
    __syncthreads();
    for (j = dmdv->cv_q; j >= k; j--){
      __syncthreads();
      md->dzn[i+md->nrows*(j-1)]+=md->dzn[i+md->nrows*j];
    }
    __syncthreads();
  }
  __syncthreads();
}

__device__
void cudaDevicecvDecreaseBDF(ModelDataGPU *md, ModelDataVariable *dmdv) {
  extern __shared__ double dzn[];
  double hsum, xi;
  int z, j;
  for (z=0; z <= md->cv_qmax; z++) md->cv_l[z+blockIdx.x*L_MAX] = 0.;
  md->cv_l[2+blockIdx.x*L_MAX] = 1.;
  hsum = 0.;
  for (j=1; j <= dmdv->cv_q-2; j++) {
    hsum += md->cv_tau[j+blockIdx.x*(L_MAX + 1)];
    xi = hsum /dmdv->cv_hscale;
    for (z=j+2; z >= 2; z--)
      md->cv_l[z+blockIdx.x*L_MAX] = md->cv_l[z+blockIdx.x*L_MAX]*xi + md->cv_l[z-1+blockIdx.x*L_MAX];
  }
  for (j=2; j < dmdv->cv_q; j++){
    md->dzn[md->nrows*j]-=md->cv_l[j+blockIdx.x*L_MAX]*md->dzn[md->nrows*(dmdv->cv_q)];
  }
}

__device__
int cudaDevicecvDoErrorTest(ModelDataGPU *md, ModelDataVariable *dmdv,
                             int *nflagPtr,
                             double saved_t, int *nefPtr, double *dsmPtr) {
  extern __shared__ double dzn[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double dsm;
  double min_val;
  int retval;
  md->dftemp[i]=md->cv_l[blockIdx.x*L_MAX]*md->cv_acor[i]+md->dzn[i];
  cudaDevicemin_2(&min_val, md->dftemp[i], dzn, md->n_shr_empty);
  if (min_val < 0. && min_val > -CAMP_TINY) {
    md->dftemp[i]=fabs(md->dftemp[i]);
    md->dzn[i]=md->dftemp[i]-md->cv_l[0+blockIdx.x*L_MAX]*md->cv_acor[i];
    min_val = 0.;
  }
  dsm = dmdv->cv_acnrm * md->cv_tq[2+blockIdx.x*(NUM_TESTS + 1)];
  *dsmPtr = dsm;
  if (dsm <= 1. && min_val >= 0.) return(CV_SUCCESS);
  (*nefPtr)++;
  *nflagPtr = PREV_ERR_FAIL;
  cudaDevicecvRestore(md, dmdv, saved_t);
  __syncthreads();
  if ((fabs(dmdv->cv_h) <= dmdv->cv_hmin*ONEPSM) ||
      (*nefPtr == md->cv_maxnef)) return(CV_ERR_FAILURE);
  dmdv->cv_etamax = 1.;
  __syncthreads();
  if (*nefPtr <= MXNEF1) {
    dmdv->cv_eta = 1. / (pow(BIAS2*dsm,1./dmdv->cv_L) + ADDON);
    __syncthreads();
    dmdv->cv_eta = SUNMAX(ETAMIN, SUNMAX(dmdv->cv_eta,
                           dmdv->cv_hmin / fabs(dmdv->cv_h)));
    __syncthreads();
    if (*nefPtr >= SMALL_NEF)
      dmdv->cv_eta = SUNMIN(dmdv->cv_eta, ETAMXF);
    __syncthreads();

    cudaDevicecvRescale(md, dmdv);
    return(TRY_AGAIN);
  }
  __syncthreads();
  if (dmdv->cv_q > 1) {
    dmdv->cv_eta = SUNMAX(ETAMIN,
    dmdv->cv_hmin / fabs(dmdv->cv_h));
    cudaDevicecvDecreaseBDF(md, dmdv);
    dmdv->cv_L = dmdv->cv_q;
    dmdv->cv_q--;
    dmdv->cv_qwait = dmdv->cv_L;
    cudaDevicecvRescale(md, dmdv);
    __syncthreads();
    return(TRY_AGAIN);
  }
  __syncthreads();
  dmdv->cv_eta = SUNMAX(ETAMIN, dmdv->cv_hmin / fabs(dmdv->cv_h));
  __syncthreads();
  dmdv->cv_h *= dmdv->cv_eta;
  dmdv->cv_next_h = dmdv->cv_h;
  dmdv->cv_hscale = dmdv->cv_h;
  __syncthreads();
  dmdv->cv_qwait = 10;
#ifdef DEBUG_printmin
  printmin(md,md->state,"cudaDevicef start state");
#endif
  int aux_flag=0;
  retval=cudaDevicef(
          dmdv->cv_tn, md->dzn, md->dtempv,md,dmdv, &aux_flag
  );
  if (retval < 0)  return(CV_RHSFUNC_FAIL);
  if (retval > 0)  return(CV_UNREC_RHSFUNC_ERR);
  md->dzn[1*md->nrows+i]=dmdv->cv_h*md->dtempv[i];
  return(TRY_AGAIN);
}

__device__
void cudaDevicecvCompleteStep(ModelDataGPU *md, ModelDataVariable *dmdv) {
  extern __shared__ double dzn[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int z, j;
#ifdef DEBUG_printmin
  printmin(md,md->dtempv,"cudaDevicecvCompleteStep start dtempv");
#endif
  __syncthreads();
  if(threadIdx.x==0) dmdv->cv_nst++;
  __syncthreads();
  dmdv->cv_hu = dmdv->cv_h;
  for (z=dmdv->cv_q; z >= 2; z--)  md->cv_tau[z+blockIdx.x*(L_MAX + 1)] = md->cv_tau[z-1+blockIdx.x*(L_MAX + 1)];
  if ((dmdv->cv_q==1) && (dmdv->cv_nst > 1))
    md->cv_tau[2+blockIdx.x*(L_MAX + 1)] = md->cv_tau[1+blockIdx.x*(L_MAX + 1)];
  md->cv_tau[1+blockIdx.x*(L_MAX + 1)] = dmdv->cv_h;
  __syncthreads();
  for (j=0; j <= dmdv->cv_q; j++){
    md->dzn[i+md->nrows*j]+=md->cv_l[j+blockIdx.x*L_MAX]*md->cv_acor[i];
  }
  dmdv->cv_qwait--;
  if ((dmdv->cv_qwait == 1) && (dmdv->cv_q != md->cv_qmax)) {
    md->dzn[i+md->nrows*(md->cv_qmax)]=md->cv_acor[i];
    dmdv->cv_saved_tq5 = md->cv_tq[5+blockIdx.x*(NUM_TESTS + 1)];
    dmdv->cv_indx_acor = md->cv_qmax;
  }
}

__device__
void cudaDevicecvChooseEta(ModelDataGPU *md, ModelDataVariable *dmdv) {
  extern __shared__ double dzn[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double etam;
  etam = SUNMAX(dmdv->cv_etaqm1, SUNMAX(dmdv->cv_etaq, dmdv->cv_etaqp1));
  __syncthreads();
  if (etam < THRESH) {
    dmdv->cv_eta = 1.;
    dmdv->cv_qprime = dmdv->cv_q;
    return;
  }
  __syncthreads();
  if (etam == dmdv->cv_etaq) {
    dmdv->cv_eta = dmdv->cv_etaq;
    dmdv->cv_qprime = dmdv->cv_q;
  } else if (etam == dmdv->cv_etaqm1) {
    dmdv->cv_eta = dmdv->cv_etaqm1;
    dmdv->cv_qprime = dmdv->cv_q - 1;
  } else {
    dmdv->cv_eta = dmdv->cv_etaqp1;
    dmdv->cv_qprime = dmdv->cv_q + 1;
    __syncthreads();
    md->dzn[md->nrows*(md->cv_qmax)+i]=md->cv_acor[i];
  }
  __syncthreads();
}

__device__
void cudaDevicecvSetEta(ModelDataGPU *md, ModelDataVariable *dmdv) {
  extern __shared__ int flag_shr[];
  __syncthreads();
  if (dmdv->cv_eta < THRESH) {
    dmdv->cv_eta = 1.;
    dmdv->cv_hprime = dmdv->cv_h;
  } else {
    __syncthreads();
    dmdv->cv_eta = SUNMIN(dmdv->cv_eta, dmdv->cv_etamax);
    __syncthreads();
    dmdv->cv_eta /= SUNMAX(ONE,
            fabs(dmdv->cv_h)*md->cv_hmax_inv*dmdv->cv_eta);
    __syncthreads();
    dmdv->cv_hprime = dmdv->cv_h * dmdv->cv_eta;
    __syncthreads();
  }
  __syncthreads();
}

__device__
int cudaDevicecvPrepareNextStep(ModelDataGPU *md, ModelDataVariable *dmdv, double dsm) {
  extern __shared__ double sdata[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();
#ifdef DEBUG_printmin
  printmin(md,md->dtempv,"cudaDevicecvPrepareNextStep start dtempv");
#endif
  if (dmdv->cv_etamax == 1.) {
    dmdv->cv_qwait = SUNMAX(dmdv->cv_qwait, 2);
    dmdv->cv_qprime = dmdv->cv_q;
    dmdv->cv_hprime = dmdv->cv_h;
    dmdv->cv_eta = 1.;
    return 0;
  }
  __syncthreads();
  dmdv->cv_etaq = 1. /(pow(BIAS2*dsm,1./dmdv->cv_L) + ADDON);
  __syncthreads();
  if (dmdv->cv_qwait != 0) {
    dmdv->cv_eta = dmdv->cv_etaq;
    dmdv->cv_qprime = dmdv->cv_q;
    cudaDevicecvSetEta(md, dmdv);
    return 0;
  }
  __syncthreads();
  dmdv->cv_qwait = 2;
  double ddn;
  dmdv->cv_etaqm1 = 0.;
  __syncthreads();
  if (dmdv->cv_q > 1) {
    cudaDeviceVWRMS_Norm_2(&md->dzn[md->nrows*(dmdv->cv_q)],
                         md->dewt, &ddn, md->nrows, md->n_shr_empty);
    __syncthreads();
    ddn *= md->cv_tq[1+blockIdx.x*(NUM_TESTS + 1)];
    __syncthreads();
    dmdv->cv_etaqm1 = 1./(pow(BIAS1*ddn, 1./dmdv->cv_q) + ADDON);
  }
  double dup, cquot;
  dmdv->cv_etaqp1 = 0.;
  __syncthreads();
  if (dmdv->cv_q != md->cv_qmax && dmdv->cv_saved_tq5 != 0.) {
    cquot = (md->cv_tq[5+blockIdx.x*(NUM_TESTS + 1)] / dmdv->cv_saved_tq5) *
            pow(double(dmdv->cv_h/md->cv_tau[2+blockIdx.x*(L_MAX + 1)]), double(dmdv->cv_L));
    md->dtempv[i]=md->cv_acor[i]-cquot*md->dzn[i+md->nrows*md->cv_qmax];
    cudaDeviceVWRMS_Norm_2(md->dtempv, md->dewt, &dup, md->nrows, md->n_shr_empty);
    __syncthreads();
    dup *= md->cv_tq[3+blockIdx.x*(NUM_TESTS + 1)];
    __syncthreads();
    dmdv->cv_etaqp1 = 1. / (pow(BIAS3*dup, 1./(dmdv->cv_L+1)) + ADDON);
  }
  __syncthreads();
  cudaDevicecvChooseEta(md, dmdv);
  __syncthreads();
  cudaDevicecvSetEta(md, dmdv);
  __syncthreads();
  return CV_SUCCESS;
}

__device__
void cudaDevicecvIncreaseBDF(ModelDataGPU *md, ModelDataVariable *dmdv) {

  extern __shared__ double dzn[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;
  double alpha0, alpha1, prod, xi, xiold, hsum, A1;
  int z, j;
  for (z=0; z <= md->cv_qmax; z++) md->cv_l[z+blockIdx.x*L_MAX] = 0.;
  md->cv_l[2+blockIdx.x*L_MAX] = alpha1 = prod = xiold = 1.;
  alpha0 = -1.;
  hsum = dmdv->cv_hscale;
  if (dmdv->cv_q > 1) {
    for (j=1; j < dmdv->cv_q; j++) {
      hsum += md->cv_tau[j+1+blockIdx.x*(L_MAX + 1)];
      xi = hsum / dmdv->cv_hscale;
      prod *= xi;
      alpha0 -= 1. / (j+1);
      alpha1 += 1. / xi;
      for (z=j+2; z >= 2; z--)
        md->cv_l[z+blockIdx.x*L_MAX] = md->cv_l[z+blockIdx.x*L_MAX]*xiold + md->cv_l[z-1+blockIdx.x*L_MAX];
      xiold = xi;
    }
  }
  A1 = (-alpha0 - alpha1) / prod;
  dzn[tid]=md->dzn[md->nrows*(dmdv->cv_L)+i];
  dzn[tid]=A1*md->dzn[md->nrows*(dmdv->cv_indx_acor)+i];
  md->dzn[md->nrows*(dmdv->cv_L)+i]=dzn[tid];
  for (j=2; j <= dmdv->cv_q; j++){
    md->dzn[i+md->nrows*j]+=md->cv_l[j+blockIdx.x*L_MAX]*md->dzn[i+md->nrows*(dmdv->cv_L)];
  }
}

__device__
void cudaDevicecvAdjustParams(ModelDataGPU *md, ModelDataVariable *dmdv) {
  if (dmdv->cv_qprime != dmdv->cv_q) {
    int deltaq = dmdv->cv_qprime-dmdv->cv_q;
    switch(deltaq) {
      case 1:
        cudaDevicecvIncreaseBDF(md, dmdv);
        break;
      case -1:
        cudaDevicecvDecreaseBDF(md, dmdv);
        break;
    }
    dmdv->cv_q = dmdv->cv_qprime;
    dmdv->cv_L = dmdv->cv_q+1;
    dmdv->cv_qwait = dmdv->cv_L;
  }
  cudaDevicecvRescale(md, dmdv);
}

__device__
int cudaDevicecvStep(ModelDataGPU *md, ModelDataVariable *dmdv) {
  extern __shared__ double sdata[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double saved_t = dmdv->cv_tn;
  int ncf = 0;
  int nef = 0;
  int nflag=FIRST_CALL;
  double dsm;
  __syncthreads();
  if ((dmdv->cv_nst > 0) && (dmdv->cv_hprime != dmdv->cv_h)){
    cudaDevicecvAdjustParams(md, dmdv);
  }
  __syncthreads();
  for (;;) {
    __syncthreads();
    cudaDevicecvPredict(md, dmdv);
    __syncthreads();
    cudaDevicecvSet(md, dmdv);
    __syncthreads();
    nflag = cudaDevicecvNlsNewton(nflag,md, dmdv);
    __syncthreads();
#ifdef DEBUG_cudaDevicecvStep
    if(threadIdx.x==0)printf("DEBUG_cudaDevicecvStep nflag %d block %d\n",nflag, blockIdx.x);
#endif
    int kflag = cudaDevicecvHandleNFlag(md, dmdv, &nflag, saved_t, &ncf);
    __syncthreads();
#ifdef DEBUG_cudaDevicecvStep
    if(threadIdx.x==0)printf("DEBUG_cudaDevicecvStep kflag %d block %d\n",kflag, blockIdx.x);
#endif
    if (kflag == PREDICT_AGAIN) {
      continue;
    }
    if (kflag != DO_ERROR_TEST) {
      return (kflag);
    }
    __syncthreads();
    int eflag=cudaDevicecvDoErrorTest(md,dmdv,&nflag,saved_t,&nef,&dsm);
    __syncthreads();
#ifdef DEBUG_cudaDevicecvStep
    if(threadIdx.x==0)printf("DEBUG_cudaDevicecvStep nflag %d eflag %d block %d\n",nflag, eflag, blockIdx.x);    //if(i==0)printf("eflag %d\n", eflag);
#endif
    if (eflag == TRY_AGAIN){
      continue;
    }
    if (eflag != CV_SUCCESS){
      return (eflag);
    }
    break;
  }
  __syncthreads();
  cudaDevicecvCompleteStep(md, dmdv);
  __syncthreads();
  cudaDevicecvPrepareNextStep(md, dmdv, dsm);
  __syncthreads();
  dmdv->cv_etamax=10.;
  md->cv_acor[i]*=md->cv_tq[2+blockIdx.x*(NUM_TESTS + 1)];
  __syncthreads();
  return(CV_SUCCESS);
  }

__device__
int cudaDeviceCVodeGetDky(ModelDataGPU *md, ModelDataVariable *dmdv,
                           double t, int k, double *dky) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double s, c, r;
  double tfuzz, tp, tn1;
  int z, j;
  __syncthreads();
   tfuzz = FUZZ_FACTOR * md->cv_uround * (fabs(dmdv->cv_tn) + fabs(dmdv->cv_hu));
   if (dmdv->cv_hu < 0.) tfuzz = -tfuzz;
   tp = dmdv->cv_tn - dmdv->cv_hu - tfuzz;
   tn1 = dmdv->cv_tn + tfuzz;
   if ((t-tp)*(t-tn1) > 0.) {
     return(CV_BAD_T);
   }
  __syncthreads();
   s = (t - dmdv->cv_tn) / dmdv->cv_h;
   for (j=dmdv->cv_q; j >= k; j--) {
     c = 1.;
     for (z=j; z >= j-k+1; z--) c *= z;
     if (j == dmdv->cv_q) {
       dky[i]=c*md->dzn[i+md->nrows*j];
     } else {
        dky[i]=c*md->dzn[i+md->nrows*j]+s*dky[i];
     }
   }
  __syncthreads();
   if (k == 0) return(CV_SUCCESS);
  __syncthreads();
   r = pow(double(dmdv->cv_h),double(-k));
  __syncthreads();
   dky[i]=dky[i]*r;
   return(CV_SUCCESS);
}

__device__
int cudaDevicecvEwtSetSV(ModelDataGPU *md, ModelDataVariable *dmdv,
                         double *dzn, double *weight) {
  extern __shared__ double flag_shr2[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  md->dtempv[i]=fabs(dzn[i]);
  double min;
  md->dtempv[i]=md->cv_reltol*md->dtempv[i]+md->cv_Vabstol[i];
  cudaDevicemin_2(&min, md->dtempv[i], flag_shr2, md->n_shr_empty);
__syncthreads();
  if (min <= 0.) return(-1);
  weight[i]= 1./md->dtempv[i];
  return(0);
}

__device__
int cudaDeviceCVode(ModelDataGPU *md, ModelDataVariable *dmdv) {
  extern __shared__ int flag_shr[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int kflag2;
#ifdef DEBUG_printmin
  printmin(md,md->state,"cudaDeviceCVode start state");
#endif
  for(;;) {
    __syncthreads();
#ifdef CAMP_DEBUG_GPU
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    if(threadIdx.x==0) dmdv->countercvStep++;
#endif
#endif
    flag_shr[0] = 0;
    __syncthreads();
    dmdv->cv_next_h = dmdv->cv_h;
    dmdv->cv_next_q = dmdv->cv_q;
    int ewtsetOK = 0;
    if (dmdv->cv_nst > 0) {
      ewtsetOK = cudaDevicecvEwtSetSV(md, dmdv, md->dzn, md->dewt);
      if (ewtsetOK != 0) {
        dmdv->cv_tretlast = dmdv->tret = dmdv->cv_tn;
        md->yout[i] = md->dzn[i];
        if(i==0) printf("ERROR: ewtsetOK\n");
        return CV_ILL_INPUT;
      }
    }
    if ((md->cv_mxstep > 0) && (dmdv->nstloc >= md->cv_mxstep)) {
      dmdv->cv_tretlast = dmdv->tret = dmdv->cv_tn;
      md->yout[i] = md->dzn[i];
      if(i==0) printf("ERROR: cv_mxstep\n");
      return CV_TOO_MUCH_WORK;
    }

    double nrm;
    cudaDeviceVWRMS_Norm_2(md->dzn,
                         md->dewt, &nrm, md->nrows, md->n_shr_empty);
    dmdv->cv_tolsf = md->cv_uround * nrm;
    if (dmdv->cv_tolsf > 1.) {
      dmdv->cv_tretlast = dmdv->tret = dmdv->cv_tn;
      md->yout[i] = md->dzn[i];
      dmdv->cv_tolsf *= 2.;
      if(i==0) printf("ERROR: cv_tolsf\n");
      __syncthreads();
      return CV_TOO_MUCH_ACC;
    } else {
      dmdv->cv_tolsf = 1.;
    }
#ifdef ODE_WARNING
    if (dmdv->cv_tn + dmdv->cv_h == dmdv->cv_tn) {
      if(threadIdx.x==0) dmdv->cv_nhnil++;
      if ((dmdv->cv_nhnil <= dmdv->cv_mxhnil) ||
              (dmdv->cv_nhnil == dmdv->cv_mxhnil))
        if(i==0)printf("WARNING: h below roundoff level in tn");
    }
#endif

    kflag2 = cudaDevicecvStep(md, dmdv);

    __syncthreads();
#ifdef DEBUG_cudaDeviceCVode
    if(i==0){
      printf("DEBUG_cudaDeviceCVode%d thread %d\n", i);
      printf("dmdv->cv_tn %le md->tout %le dmdv->cv_h %le dmdv->cv_hprime %le\n",
             dmdv->cv_tn,md->tout,dmdv->cv_h,dmdv->cv_hprime);
    }
#endif
    if (kflag2 != CV_SUCCESS) {
      dmdv->cv_tretlast = dmdv->tret = dmdv->cv_tn;
      md->yout[i] = md->dzn[i];
      if(i==0) printf("ERROR: kflag != CV_SUCCESS\n");
      return kflag2;
    }
    dmdv->nstloc++;
    if ((dmdv->cv_tn - md->tout) * dmdv->cv_h >= 0.) {
      dmdv->cv_tretlast = dmdv->tret = md->tout;
      cudaDeviceCVodeGetDky(md, dmdv, md->tout, 0, md->yout);
      return CV_SUCCESS;
    }
    if (md->cv_tstopset) {//needed?
      double troundoff = FUZZ_FACTOR * md->cv_uround * (fabs(dmdv->cv_tn) + fabs(dmdv->cv_h));
      if (fabs(dmdv->cv_tn - md->cv_tstop) <= troundoff) {
        cudaDeviceCVodeGetDky(md, dmdv, md->cv_tstop, 0, md->yout);
        dmdv->cv_tretlast = dmdv->tret = md->cv_tstop;
        md->cv_tstopset = SUNFALSE;
        if(i==0) printf("ERROR: cv_tstopset\n");
        __syncthreads();
        return CV_TSTOP_RETURN;
      }
      if ((dmdv->cv_tn + dmdv->cv_hprime - md->cv_tstop) * dmdv->cv_h > 0.) {
        dmdv->cv_hprime = (md->cv_tstop - dmdv->cv_tn) * (1.0 - 4.0 * md->cv_uround);
        if(i==0) printf("ERROR: dmdv->cv_tn + dmdv->cv_hprime - dmdv->cv_tstop\n");
        dmdv->cv_eta = dmdv->cv_hprime / dmdv->cv_h;
      }
    }
  }
}

__global__
void cudaGlobalCVode(ModelDataGPU md_object) {
  ModelDataGPU *md = &md_object;
  extern __shared__ int flag_shr[];
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  ModelDataVariable *dmdv = &md->sCells[blockIdx.x];
  int active_threads = md->nrows;
  int istate;
  __syncthreads();
  if(tid<active_threads){
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
    int clock_khz=md->clock_khz;
    clock_t start;
    start = clock();
    __syncthreads();
#endif
    istate=cudaDeviceCVode(md,dmdv);
    __syncthreads();
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  if(threadIdx.x==0) dmdv->dtcudaDeviceCVode += ((double)(int)(clock() - start))/(clock_khz*1000);
  __syncthreads();
#endif
  }
  __syncthreads();
  if(threadIdx.x==0) md->flagCells[blockIdx.x]=istate;
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  ModelDataVariable *mdvo = md->mdvo;
  *mdvo = *dmdv;
#endif
}

int nextPowerOfTwoCVODE2(int v){
  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;
  return v;
}

void cvodeRun(ModelDataGPU *mGPU, hipStream_t stream){
  int len_cell = mGPU->nrows / mGPU->n_cells;
  int threads_block = len_cell;
  int blocks = mGPU->n_cells;
  int n_shr_memory = nextPowerOfTwoCVODE2(len_cell);
  int n_shr_empty = mGPU->n_shr_empty = n_shr_memory - threads_block;
  cudaGlobalCVode <<<blocks, threads_block, n_shr_memory * sizeof(double), stream>>>(*mGPU);
}

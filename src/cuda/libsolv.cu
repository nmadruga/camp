#include "hip/hip_runtime.h"
/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
 * Illinois at Urbana-Champaign
 * SPDX-License-Identifier: MIT
 */

#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

#include "libsolv.h"

//#include<hipblas.h>
//#include<hipblas.h>

using namespace std;

//
//dAthreads
//
// Para reservar memoria Double e Int
extern "C++" void cudaMallocDouble(double* &vector,int size)
{
	hipMalloc((void**)&vector,size*sizeof(double));
}

extern "C++" void cudaMallocInt(int* &vector,int size)
{
	hipMalloc((void**)&vector,size*sizeof(int));
}

// Para copiar a CPU->GPU Double e Int
extern "C++" void cudaMemcpyDToGpu(double* h_vect,double* d_vect,int size )
{
  hipMemcpy(d_vect,h_vect,size*sizeof(double),hipMemcpyHostToDevice);
}

extern "C++" void cudaMemcpyIToGpu(int* h_vect,int* d_vect,int size )
{
		hipMemcpy(d_vect,h_vect,size*sizeof(int),hipMemcpyHostToDevice);
}

// Para copiar a GPU->CPU Double e Int
extern "C++" void cudaMemcpyIToCpu(int* h_vect, int* d_vect,int size )
{
		hipMemcpy(h_vect,d_vect,size*sizeof(int),hipMemcpyDeviceToHost);
}

extern "C++" void cudaMemcpyDToCpu(double* h_vect, double* d_vect,int size )
{
  hipMemcpy(h_vect,d_vect,size*sizeof(double),hipMemcpyDeviceToHost);
}

// Para liberar memoria
extern "C++" void cudaFreeMem(void* vector)
{
	hipFree(vector);
}

extern "C++" void cudaGetLastErrorC(){
     hipError_t error;
     error=hipGetLastError();
     if(error!= hipSuccess)
     {
       cout<<" ERROR INSIDE A CUDA FUNCTION: "<<error<<" "<<hipGetErrorString(error)<<endl;
       exit(0);
     }
}

__global__ void cudamatScaleAddI(int nrows, double* dA, int* djA, int* diA, double alpha)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows)
  {
    int jstart = diA[row];
    int jend   = diA[row+1];
    for(int j=jstart; j<jend; j++)
    {
      if(djA[j]==row)
      {
        dA[j] = 1.0 + alpha*dA[j];
      }
      else{
        dA[j] = alpha*dA[j];
      }
    }
  }
}

// A = I - gamma*J
// dA  : Matrix values (nnz size)
// djA : Matrix columns (nnz size)
// diA : Matrix rows (nrows+1 size)
// alpha : Scale factor
extern "C++" void gpu_matScaleAddI(int nrows, double* dA, int* djA, int* diA, double alpha, int blocks, int threads)
{

   blocks = (nrows+threads-1)/threads;

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

  cudamatScaleAddI<<<dimGrid,dimBlock>>>(nrows, dA, djA, diA, alpha);
}

__global__
void check_input_gpud(double *x, int len, int var_id)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  printf("%d[%d]=%-le\n",var_id,i,x[i]);

}

// Diagonal precond
__global__ void cudadiagprecond(int nrows, double* dA, int* djA, int* diA, double* ddiag)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;

#ifdef DEBUG_cudadiagprecond


#endif

  if(row < nrows){
    int jstart=diA[row];
    int jend  =diA[row+1];
    for(int j=jstart;j<jend;j++){
      if(djA[j]==row){
        if(dA[j]!=0.0)
          ddiag[row]= 1.0/dA[j];
        else{
          //printf("cudadiagprecond else\n");
          ddiag[row]= 1.0;
        }
      }
    }
  }

}

extern "C++" void gpu_diagprecond(int nrows, double* dA, int* djA, int* diA, double* ddiag, int blocks, int threads)
{

  blocks = (nrows+threads-1)/threads;

  dim3 dimGrid(blocks,1,1);
  dim3 dimBlock(threads,1,1);

  cudadiagprecond<<<dimGrid,dimBlock>>>(nrows, dA, djA, diA, ddiag);
  //check_input_gpud<< < 1, 5>> >(ddiag,nrows,0);
}

// y = constant
__global__ void cudasetconst(double* dy,double constant,int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dy[row]=constant;
	}
}

extern "C++" void gpu_yequalsconst(double *dy, double constant, int nrows, int blocks, int threads)
{
   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   cudasetconst<<<dimGrid,dimBlock>>>(dy,constant,nrows);

}


// x=A*b
__global__ void cudaSpmvCSR(double* dx, double* db, int nrows, double* dA, int* djA, int* diA)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows)
  {
    int jstart = diA[row];
    int jend   = diA[row+1];
    double sum = 0.0;
    for(int j=jstart; j<jend; j++)
    {
      sum+= db[djA[j]]*dA[j];
    }
    dx[row]=sum;
	}

}

__global__ void cudaSpmvCSC(double* dx, double* db, int nrows, double* dA, int* djA, int* diA)
{
  double mult;
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows)
  {
    int jstart = diA[row];
    int jend   = diA[row+1];
    for(int j=jstart; j<jend; j++)
    {
      mult = db[row]*dA[j];
      atomicAdd(&(dx[djA[j]]),mult);
    }
	}
}

extern "C++" void gpu_spmv(double* dx ,double* db, int nrows, double* dA, int *djA,int *diA,int blocks,int  threads)
{
   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

#ifndef CSR_SPMV_CPU

   cudaSpmvCSR<<<dimGrid,dimBlock>>>(dx, db, nrows, dA, djA, diA);

#else
    cudasetconst<<<dimGrid,dimBlock>>>(dx, 0.0, nrows);
    cudaSpmvCSC<<<dimGrid,dimBlock>>>(dx, db, nrows, dA, djA, diA);
#endif

}

// y= a*x+ b*y
__global__ void cudaaxpby(double* dy,double* dx, double a, double b, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dy[row]= a*dx[row] + b*dy[row];
	}
}

extern "C++" void gpu_axpby(double* dy ,double* dx, double a, double b, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   cudaaxpby<<<dimGrid,dimBlock>>>(dy,dx,a,b,nrows);
}

// y = x
__global__ void cudayequalsx(double* dy,double* dx,int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dy[row]=dx[row];
	}
}

extern "C++" void gpu_yequalsx(double *dy, double* dx, int nrows, int blocks, int threads)
{
   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   cudayequalsx<<<dimGrid,dimBlock>>>(dy,dx,nrows);

}

__global__ void cudareducey(double *g_odata, unsigned int n)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;

  double mySum =  (tid < n) ? g_odata[tid] : 0;

  sdata[tid] = mySum;
  __syncthreads();

  for (unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void cudadotxy(double *g_idata1, double *g_idata2, double *g_odata, unsigned int n)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;//*2 because init blocks is half
  //unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;//*2 because init blocks is half

  double mySum = (i < n) ? g_idata1[i]*g_idata2[i] : 0;

  if (i + blockDim.x < n)
    mySum += g_idata1[i+blockDim.x]*g_idata2[i+blockDim.x];

  sdata[tid] = mySum;
  __syncthreads();

  //for (unsigned int s=(blockDim.x+1)/2; s>0; s>>=1)
  for (unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

//threads need to be pow of 2 //todo h_temp not needed
extern "C++" double gpu_dotxy(double* vec1, double* vec2, double* h_temp, double* d_temp, int nrows, int blocks,int threads)
{
  double sum;
  dim3 dimGrid(blocks,1,1);
  dim3 dimBlock(threads,1,1);

  //threads*sizeof(double)
  cudadotxy<<<dimGrid,dimBlock,threads*sizeof(double)>>>(vec1,vec2,d_temp,nrows);
  hipMemcpy(&sum, d_temp, sizeof(double), hipMemcpyDeviceToHost);
  //printf("rho1 %f", sum);

  int redsize= sqrt(blocks) +1;
  redsize=pow(2,redsize);

  dim3 dimGrid2(1,1,1);
  dim3 dimBlock2(redsize,1,1);

  cudareducey<<<dimGrid2,dimBlock2,redsize*sizeof(double)>>>(d_temp,blocks);
  hipMemcpy(&sum, d_temp, sizeof(double), hipMemcpyDeviceToHost);

  return sum;

/*
  hipMemcpy(h_temp, d_temp, blocks * sizeof(double), hipMemcpyDeviceToHost);
  double sum=0;
  for(int i=0;i<blocks;i++)
  {
    sum+=h_temp[i];
  }
  return sum;
*/
  /*dim3 dimGrid2(1,1,1);
  dim3 dimBlock2(blocks,1,1);

  //Cuda only sum kernel call
  //cudareducey<<<dimGrid2,dimBlock2,blocks*sizeof(double)>>>(d_temp,blocks); //Takes quasi WAY MORE than cpu calc

  hipMemcpy(h_temp, d_temp, sizeof(double), hipMemcpyDeviceToHost);
  return h_temp[0];*/
}

/*
extern "C++" double gpu_dotxy(double *dy, double* dx, int nrows)
{
   double dot=0.0;
   hipblasHandle_t hl;
   hipblasCreate(&hl);

   hipblasDdot(hl,nrows,dy,1,dx,1,&dot);

   hipblasDestroy(hl);
   return dot;
}
*/

// z= a*z + x + b*y
__global__ void cudazaxpbypc(double* dz, double* dx,double* dy, double a, double b, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dz[row]=a*dz[row]  + dx[row] + b*dy[row];
	}
}

extern "C++" void gpu_zaxpbypc(double* dz, double* dx ,double* dy, double a, double b, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   cudazaxpbypc<<<dimGrid,dimBlock>>>(dz,dx,dy,a,b,nrows);
}

// z= x*y
__global__ void cudamultxy(double* dz, double* dx,double* dy, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dz[row]=dx[row]*dy[row];
	}
}

extern "C++" void gpu_multxy(double* dz, double* dx ,double* dy, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   cudamultxy<<<dimGrid,dimBlock>>>(dz,dx,dy,nrows);
}

// a*x + b*y = z
//__global__ void cudazaxpby(double* dz, double* dx,double* dy, double a, double b, int nrows)
__global__ void cudazaxpby(double a, double* dx, double b, double* dy, double* dz, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dz[row]=a*dx[row] + b*dy[row];
	}
}

extern "C++" void gpu_zaxpby(double a, double* dx, double b, double* dy, double* dz, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

  cudazaxpby<<<dimGrid,dimBlock>>>(a,dx,b,dy,dz,nrows);
}

// y= a*x + y
__global__ void cudaaxpy(double* dy,double* dx, double a, int nrows)
{
	int row= threadIdx.x + blockDim.x*blockIdx.x;
   	if(row < nrows){
		dy[row]=a*dx[row] + dy[row];
	}
}

extern "C++" void gpu_axpy(double* dy, double* dx ,double a, int nrows, int blocks, int threads)
{

   dim3 dimGrid(blocks,1,1);
   dim3 dimBlock(threads,1,1);

   cudaaxpy<<<dimGrid,dimBlock>>>(dy,dx,a,nrows);
}

// sqrt(sum ( (x_i*y_i)^2)/n)
__global__ void cudaDVWRMS_Norm(double *g_idata1, double *g_idata2, double *g_odata, unsigned int n)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

  double mySum = (i < n) ? g_idata1[i]*g_idata1[i]*g_idata2[i]*g_idata2[i] : 0;

  if (i + blockDim.x < n)
    mySum += g_idata1[i+blockDim.x]*g_idata1[i+blockDim.x]*g_idata2[i+blockDim.x]*g_idata2[i+blockDim.x];

  sdata[tid] = mySum;
  __syncthreads();

  for (unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] = mySum = mySum + sdata[tid + s];

    __syncthreads();
  }

  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

extern "C++" double gpu_VWRMS_Norm(int n, double* vec1,double* vec2,double* h_temp,double* d_temp, int blocks,int threads)
{
  dim3 dimGrid(blocks,1,1);
  dim3 dimBlock(threads,1,1);

  cudaDVWRMS_Norm<<<dimGrid,dimBlock,threads*sizeof(double)>>>(vec1,vec2,d_temp,n);

  //hipMemcpy(h_temp, d_temp, blocks * sizeof(double), hipMemcpyDeviceToHost);

  int redsize= sqrt(blocks) +1;
  redsize=pow(2,redsize);

  dim3 dimGrid2(1,1,1);
  dim3 dimBlock2(redsize,1,1);

  cudareducey<<<dimGrid2,dimBlock2,redsize*sizeof(double)>>>(d_temp,blocks);

  double sum;
  hipMemcpy(&sum, d_temp, sizeof(double), hipMemcpyDeviceToHost);

  return sqrt(sum/n);

/*
  double sum=0;
  for(int i=0;i<blocks;i++)
  {
    sum+=h_temp[i];
  }
  return sqrt(sum/n);
  */
}

// y=alpha*y
__global__ void cudascaley(double* dy, double a, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  if(row < nrows){
    dy[row]=a*dy[row];
  }
}

extern "C++" void gpu_scaley(double* dy, double a, int nrows, int blocks, int threads)
{
  dim3 dimGrid(blocks,1,1);
  dim3 dimBlock(threads,1,1);

  cudascaley<<<dimGrid,dimBlock>>>(dy,a,nrows);
}

// Device functions (equivalent to global functions but in device to allow calls from gpu)
__device__ void cudaDeviceBCGprecond(double* dA, int* djA, int* diA, double* ddiag, double alpha){
int row= threadIdx.x + blockDim.x*blockIdx.x;
int nnz=diA[blockDim.x];
  for(int j=diA[threadIdx.x];j<diA[threadIdx.x+1];j++){
    if(djA[j]==threadIdx.x){
      dA[j+nnz*blockIdx.x] = 1.0 + alpha*dA[j+nnz*blockIdx.x];
      if(dA[j+nnz*blockIdx.x]!=0.0){
        ddiag[row]= 1.0/dA[j+nnz*blockIdx.x];
       }else{
        ddiag[row]= 1.0;
      }
    }else{
      dA[j+nnz*blockIdx.x] = alpha*dA[j+nnz*blockIdx.x];
    }
  }
}

// y = constant
__device__ void cudaDevicesetconst(double* dy,double constant){
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dy[row]=constant;
}

// x=A*b
__device__ void cudaDeviceSpmvCSR(double* dx, double* db, double* dA, int* djA, int* diA){
  __syncthreads();
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  double sum = 0.0;
  int nnz=diA[blockDim.x];
  for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++){
    sum+= db[djA[j]+blockDim.x*blockIdx.x]*dA[j+nnz*blockIdx.x];
  }
  dx[row]=sum;
  __syncthreads();
}

__device__ void cudaDeviceSpmvCSC_block(double* dx, double* db, double* dA, int* djA, int* diA){
  int row = threadIdx.x + blockDim.x*blockIdx.x;
  __syncthreads();
  dx[row]=0.0;
  __syncthreads(); //Multiple threads can save to the same row

  int nnz=diA[blockDim.x];
  for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++){
    double mult = db[row]*dA[j+nnz*blockIdx.x];
    atomicAdd_block(&(dx[djA[j]+blockDim.x*blockIdx.x]),mult);
  //		dx[djA[j]]+= db[row]*dA[j];
  }
  __syncthreads();
}

__device__ void cudaDeviceSpmv(double* dx, double* db, double* dA, int* djA, int* diA){
#ifndef USE_CSR_ODE_GPU
  cudaDeviceSpmvCSR(dx,db,dA,djA,diA);
#else
  cudaDeviceSpmvCSC_block(dx,db,dA,djA,diA);
#endif
}

// y= a*x+ b*y
__device__ void cudaDeviceaxpby(double* dy,double* dx, double a, double b, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dy[row]= a*dx[row] + b*dy[row];
}

// y = x
__device__ void cudaDeviceyequalsx(double* dy,double* dx,int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
    dy[row]=dx[row];
}

__device__ void cudaDevicemin(double *g_odata, double in, volatile double *sdata, int n_shr_empty)
{
  unsigned int tid = threadIdx.x;

  __syncthreads();

  sdata[tid] = in;

  __syncthreads();
  //first threads update empty positions
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=sdata[tid];
  __syncthreads(); //Not needed (should)

  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s){
      if(sdata[tid + s] < sdata[tid] ) sdata[tid]=sdata[tid + s];
    }
    __syncthreads();
  }

  __syncthreads();
  *g_odata = sdata[0];
  __syncthreads();

}

__device__ void cudaDevicemaxI(int *g_odata, int in, volatile double *sdata, int n_shr_empty)
{
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  __syncthreads();

  sdata[tid] = in;

  __syncthreads();
  //first threads update empty positions
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=sdata[tid];
  __syncthreads(); //Not needed (should)

  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s){
      if(sdata[tid + s] > sdata[tid] ) sdata[tid]=sdata[tid + s];
    }
    __syncthreads();
  }

  __syncthreads();
  *g_odata = sdata[0];
  __syncthreads();

}

__device__ void cudaDeviceaddI(int *g_odata, int in, volatile double *sdata, int n_shr_empty)
{
  //extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  __syncthreads();

  sdata[tid] = in;

  __syncthreads();

  //first threads update empty positions
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=sdata[tid];

  __syncthreads(); //Not needed (should)

  //if(blockIdx.x==0)printf("i %d in %le sdata[tid] %le\n",i,in,sdata[tid]);

  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s){//&& sdata[tid + s]!=0.
      //if(sdata[tid + s] < sdata[tid] ) sdata[tid]=sdata[tid + s];
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  __syncthreads();
  *g_odata = sdata[0];
  __syncthreads();

}

__device__ void warpReduce(volatile double *sdata, unsigned int tid) {
  unsigned int blockSize = blockDim.x;
  if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
  if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
  if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
  if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
  if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
  if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

__device__ void cudaDevicedotxy(double *g_idata1, double *g_idata2,
                                 double *g_odata, int n_shr_empty)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  __syncthreads();
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=0.;
  __syncthreads();
  sdata[tid] = g_idata1[i]*g_idata2[i];
  __syncthreads();
  unsigned int blockSize = blockDim.x+n_shr_empty;
  // do reduction in shared mem
  if ((blockSize >= 1024) && (tid < 512)) {
    sdata[tid] += sdata[tid + 512];
  }
  __syncthreads();
  if ((blockSize >= 512) && (tid < 256)) {
    sdata[tid] += sdata[tid + 256];
  }
  __syncthreads();
  if ((blockSize >= 256) && (tid < 128)) {
    sdata[tid] += sdata[tid + 128];
  }
  __syncthreads();
  if ((blockSize >= 128) && (tid < 64)) {
    sdata[tid] += sdata[tid + 64];
  }
  __syncthreads();
  if (tid < 32) warpReduce(sdata, tid);
  __syncthreads();//not needed?
  *g_odata = sdata[0];
  __syncthreads();
}

// z= a*z + x + b*y
__device__ void cudaDevicezaxpbypc(double* dz, double* dx,double* dy, double a, double b, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dz[row]=a*dz[row]  + dx[row] + b*dy[row];
}

// z= x*y
__device__ void cudaDevicemultxy(double* dz, double* dx,double* dy, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dz[row]=dx[row]*dy[row];
}

// z= a*x + b*y
__device__ void cudaDevicezaxpby(double a, double* dx, double b, double* dy, double* dz, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dz[row]=a*dx[row] + b*dy[row];
}

// y= a*x + y
__device__ void cudaDeviceaxpy(double* dy,double* dx, double a, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dy[row]=a*dx[row] + dy[row];
}

// sqrt(sum ( (x_i*y_i)^2)/n)
__device__ void cudaDeviceVWRMS_Norm(double *g_idata1, double *g_idata2, double *g_odata, int n, int n_shr_empty)
{
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  //unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

  __syncthreads();

  //first threads update empty positions
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=0.;

  __syncthreads(); //Not needed (should)

/*
  double mySum = (i < n) ? g_idata1[i]*g_idata1[i]*g_idata2[i]*g_idata2[i] : 0;
  if (i + blockDim.x < n)
    mySum += g_idata1[i+blockDim.x]*g_idata1[i+blockDim.x]*g_idata2[i+blockDim.x]*g_idata2[i+blockDim.x];
*/

  __syncthreads();
  sdata[tid] = g_idata1[i]*g_idata1[i]*g_idata2[i]*g_idata2[i];
  __syncthreads();

  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s)
      sdata[tid] += sdata[tid + s];

    __syncthreads();
  }

  //if (tid == 0) g_odata[blockIdx.x] = sdata[0];
  g_odata[0] = sqrt(sdata[0]/n);
  //*g_odata = sqrt(sdata[0]/n);
  __syncthreads();
}

// y=alpha*y
__device__ void cudaDevicescaley(double* dy, double a, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dy[row]=a*dy[row];
}


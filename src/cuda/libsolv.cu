#include "hip/hip_runtime.h"
/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
 * Illinois at Urbana-Champaign
 * SPDX-License-Identifier: MIT
 */

#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include "libsolv.h"

__device__ void cudaDeviceBCGprecond(double* dA, int* djA, int* diA, double* ddiag, double alpha){
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  int nnz=diA[blockDim.x];
  for(int j=diA[threadIdx.x];j<diA[threadIdx.x+1];j++){
    if(djA[j]==threadIdx.x){
      dA[j+nnz*blockIdx.x] = 1.0 + alpha*dA[j+nnz*blockIdx.x];
      if(dA[j+nnz*blockIdx.x]!=0.0){
        ddiag[row]= 1.0/dA[j+nnz*blockIdx.x];
       }else{
        ddiag[row]= 1.0;
      }
    }else{
      dA[j+nnz*blockIdx.x] = alpha*dA[j+nnz*blockIdx.x];
    }
  }
}

__device__ void cudaDevicesetconst(double* dy,double constant){
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dy[row]=constant;
}

__device__ void cudaDeviceSpmvCSR(double* dx, double* db, double* dA, int* djA, int* diA){
  __syncthreads();
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  double sum = 0.0;
  int nnz=diA[blockDim.x];
  for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++){
    sum+= db[djA[j]+blockDim.x*blockIdx.x]*dA[j+nnz*blockIdx.x];
  }
  dx[row]=sum;
  __syncthreads();
}

__device__ void cudaDeviceSpmvCSC_block(double* dx, double* db, double* dA, int* djA, int* diA){
  int row = threadIdx.x + blockDim.x*blockIdx.x;
  __syncthreads();
  dx[row]=0.0;
  __syncthreads();
  int nnz=diA[blockDim.x];
  for(int j=diA[threadIdx.x]; j<diA[threadIdx.x+1]; j++){
    double mult = db[row]*dA[j+nnz*blockIdx.x];
    atomicAdd_block(&(dx[djA[j]+blockDim.x*blockIdx.x]),mult);
  }
  __syncthreads();
}

__device__ void cudaDeviceSpmv(double* dx, double* db, double* dA, int* djA, int* diA){
#ifndef old_USE_CSR_ODE_GPU
  cudaDeviceSpmvCSR(dx,db,dA,djA,diA);
#else
  cudaDeviceSpmvCSC_block(dx,db,dA,djA,diA);
#endif
}

// y= a*x+ b*y
__device__ void cudaDeviceaxpby(double* dy,double* dx, double a, double b, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dy[row]= a*dx[row] + b*dy[row];
}

// y = x
__device__ void cudaDeviceyequalsx(double* dy,double* dx,int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
    dy[row]=dx[row];
}

__device__ void cudaDevicemin(double *g_odata, double in, volatile double *sdata, int n_shr_empty){
  unsigned int tid = threadIdx.x;
  __syncthreads();
  sdata[tid] = in;
  __syncthreads();
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=sdata[tid];
  __syncthreads();
  for (unsigned int s=(blockDim.x+n_shr_empty)/2; s>0; s>>=1)
  {
    if (tid < s){
      if(sdata[tid + s] < sdata[tid] ) sdata[tid]=sdata[tid + s];
    }
    __syncthreads();
  }
  *g_odata = sdata[0];
  __syncthreads();
}

__device__ void warpReduce(volatile double *sdata, unsigned int tid) {
  unsigned int blockSize = blockDim.x;
  if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
  if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
  if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
  if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
  if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
  if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

__device__ void cudaDevicedotxy(double *g_idata1, double *g_idata2,
                                 double *g_odata, int n_shr_empty){
  extern __shared__ double sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  __syncthreads();
  if(tid<n_shr_empty)
    sdata[tid+blockDim.x]=0.;
  __syncthreads();
  sdata[tid] = g_idata1[i]*g_idata2[i];
  __syncthreads();
  unsigned int blockSize = blockDim.x+n_shr_empty;
  if ((blockSize >= 1024) && (tid < 512)) {
    sdata[tid] += sdata[tid + 512];
  }
  __syncthreads();
  if ((blockSize >= 512) && (tid < 256)) {
    sdata[tid] += sdata[tid + 256];
  }
  __syncthreads();
  if ((blockSize >= 256) && (tid < 128)) {
    sdata[tid] += sdata[tid + 128];
  }
  __syncthreads();
  if ((blockSize >= 128) && (tid < 64)) {
    sdata[tid] += sdata[tid + 64];
  }
  __syncthreads();
  if (tid < 32) warpReduce(sdata, tid);
  __syncthreads();
  *g_odata = sdata[0];
  __syncthreads();
}

// z= a*z + x + b*y
__device__ void cudaDevicezaxpbypc(double* dz, double* dx,double* dy, double a, double b, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dz[row]=a*dz[row]  + dx[row] + b*dy[row];
}

// z= x*y
__device__ void cudaDevicemultxy(double* dz, double* dx,double* dy, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dz[row]=dx[row]*dy[row];
}

// z= a*x + b*y
__device__ void cudaDevicezaxpby(double a, double* dx, double b, double* dy, double* dz, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dz[row]=a*dx[row] + b*dy[row];
}

// y= a*x + y
__device__ void cudaDeviceaxpy(double* dy,double* dx, double a, int nrows)
{
  int row= threadIdx.x + blockDim.x*blockIdx.x;
  dy[row]+=a*dx[row];
}



#include "hip/hip_runtime.h"
/* Copyright (C) 2019 Matthew Dawson
 * Licensed under the GNU General Public License version 2 or (at your
 * option) any later version. See the file COPYING for details.
 *
 * Functions of the time derivative structure
 *
 */
/** \file
 * \brief Functions of the time derivative structure
 */

extern "C" {
#include "time_derivative_gpu.h"
#include <math.h>
#include <stdio.h>

int time_derivative_initialize_gpu(SolverData *sd) {

  ModelData *md = &(sd->model_data);

  int num_spec = md->n_per_cell_dep_var*md->n_cells;
  //Pre-allocation related arrays (internal struct arrays are mandatory)

  ModelDataGPU *mGPU = &sd->mGPU;

  hipMalloc((void **) &(mGPU->production_rates),num_spec*sizeof(mGPU->production_rates));
  hipMalloc((void **) &(mGPU->loss_rates),num_spec*sizeof(mGPU->loss_rates));

  return 1;
}

#ifdef __CUDA_ARCH__
__host__ __device__
#endif
void time_derivative_reset_gpu(TimeDerivativeGPU time_deriv) {

#ifdef __CUDA_ARCH__
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<time_deriv.num_spec){
    time_deriv.production_rates[i] = 0.0;
    time_deriv.loss_rates[i] = 0.0;
    //time_deriv.production_rates[i] = 0.00001;
    //time_deriv.loss_rates[i] = 0.00001;
  }
#else
  for (unsigned int i_spec = 0; i_spec < time_deriv.num_spec; ++i_spec) {
    time_deriv.production_rates[i_spec] = 0.0;
    time_deriv.loss_rates[i_spec] = 0.0;
  }
#endif

}

#ifdef __CUDA_ARCH__
__host__ __device__
#endif
void time_derivative_output_gpu(TimeDerivativeGPU time_deriv, double *dest_array,
                            double *deriv_est, unsigned int output_precision) {

#ifdef PMC_DEBUG
  time_deriv.last_max_loss_precision = 1.0;
#endif

#ifdef __CUDA_ARCH__

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<time_deriv.num_spec){

    double *r_p = time_deriv.production_rates;
    double *r_l = time_deriv.loss_rates;

    //dest_array[i] = 0.1;
    //dest_array[i] = r_p[i];
    if (r_p[i] + r_l[i] != 0.0) {
      if (deriv_est) {
        double scale_fact;
        scale_fact =
            1.0 / (r_p[i] + r_l[i]) /
            (1.0 / (r_p[i] + r_l[i]) + MAX_PRECISION_LOSS / fabs(r_p[i]- r_l[i]));
          dest_array[i] =
          scale_fact * (r_p[i] - r_l[i]) + (1.0 - scale_fact) * (deriv_est[i]);
        //dest_array[i] = 0.1;
      } else {
        dest_array[i] = r_p[i] - r_l[i];
        //dest_array[i] = 0.2;
      }
    } else {
      dest_array[i] = 0.0;
      //dest_array[i] = 0.000000003;
      //dest_array[i] = r_l[i];
      //dest_array[i] = r_p[i];
    }
    //dest_array[i] = r_p[i];
    //dest_array[i] = r_l[i];
  }

#else

  double *r_p = time_deriv.production_rates;
  double *r_l = time_deriv.loss_rates;

  for (unsigned int i_spec = 0; i_spec < time_deriv.num_spec; ++i_spec) {
    double prec_loss = 1.0;
    if (*r_p + *r_l != 0.0) {
      if (deriv_est) {
        double scale_fact;
        scale_fact =
            1.0 / (*r_p + *r_l) /
            (1.0 / (*r_p + *r_l) + MAX_PRECISION_LOSS / fabsl(*r_p - *r_l));
        *dest_array =
            scale_fact * (*r_p - *r_l) + (1.0 - scale_fact) * (*deriv_est);
      } else {
        *dest_array = *r_p - *r_l;
      }
#ifdef PMC_DEBUG
      if (*r_p != 0.0 && *r_l != 0.0) {
        prec_loss = *r_p > *r_l ? 1.0 - *r_l / *r_p : 1.0 - *r_p / *r_l;
        if (prec_loss < time_deriv.last_max_loss_precision)
          time_deriv.last_max_loss_precision = prec_loss;
      }
#endif
    } else {
      *dest_array = 0.0;
    }
    ++r_p;
    ++r_l;
    ++dest_array;
    if (deriv_est) ++deriv_est;
#ifdef PMC_DEBUG
    if (output_precision == 1) {
      printf("\nspec %d prec_loss %le", i_spec, -log(prec_loss) / log(2.0));
    }
#endif
  }

#endif

}

#ifdef __CUDA_ARCH__
__host__ __device__
#endif
void time_derivative_add_value_gpu(TimeDerivativeGPU time_deriv, unsigned int spec_id,
                               double rate_contribution) {
#ifdef __CUDA_ARCH__
  if (rate_contribution > 0.0) {
    atomicAdd_block(&(time_deriv.production_rates[spec_id]),rate_contribution);
    //atomicAdd(&(time_deriv.production_rates[spec_id]),rate_contribution);
    //atomicAdd(&(time_deriv.production_rates[spec_id]),0.2); //debug
  } else {
    atomicAdd_block(&(time_deriv.loss_rates[spec_id]),-rate_contribution);
    //atomicAdd(&(time_deriv.loss_rates[spec_id]),-rate_contribution);
    //atomicAdd(&(time_deriv.loss_rates[spec_id]),-0.1); //debug
  }
#else
  if (rate_contribution > 0.0) {
    time_deriv.production_rates[spec_id] += rate_contribution;
  } else {
    time_deriv.loss_rates[spec_id] += -rate_contribution;
  }
#endif
}

#ifdef PMC_DEBUG
double time_derivative_max_loss_precision(TimeDerivativeGPU time_deriv) {
  return -log(time_deriv.last_max_loss_precision) / log(2.0);
}
#endif

void time_derivative_free_gpu(TimeDerivativeGPU time_deriv) {
#ifdef __CUDA_ARCH__
  hipFree(time_deriv.production_rates);
  hipFree(time_deriv.loss_rates);
#else
  free(time_deriv.production_rates);
  free(time_deriv.loss_rates);
#endif
}

}

//todo merge with default time_derivative "class" using .cpp file
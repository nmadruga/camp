/* Copyright (C) 2021 Barcelona Supercomputing Center and University of
 * Illinois at Urbana-Champaign
 * SPDX-License-Identifier: MIT
 */

#include "cvode_cuda.h"
extern "C" {
#include "cvode_gpu.h"
}
#ifdef CAMP_USE_MPI
#include <mpi.h>
#endif

void constructor_cvode_gpu(SolverData *sd){
  CVodeMem cv_mem = (CVodeMem) sd->cvode_mem;
  ModelDataCPU *mCPU = &(sd->mCPU);
  ModelData *md = &(sd->model_data);
  CVDlsMem cvdls_mem = (CVDlsMem) cv_mem->cv_lmem;
  sd->mGPU = (ModelDataGPU *)malloc(sizeof(ModelDataGPU));
  ModelDataGPU *mGPU = sd->mGPU;
  int n_cells = md->n_cells_gpu;
  mGPU->n_cells= n_cells;
  sd->flagCells = (int *) malloc((n_cells) * sizeof(int));
  int n_dep_var = md->n_per_cell_dep_var;
  int n_state_var = md->n_per_cell_state_var;
  int n_rxn = md->n_rxn;
  size_t rxn_env_data_idx_size = (n_rxn+1) * sizeof(int);
  size_t map_state_deriv_size = n_dep_var * n_cells * sizeof(int);
  int nGPUs;
  hipGetDeviceCount(&nGPUs);
  int rank, size;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Comm_size(MPI_COMM_WORLD, &size);
  int iDevice = rank % nGPUs;
  hipSetDevice(iDevice);
  mGPU->n_rxn=md->n_rxn;
  mGPU->n_rxn_env_data=md->n_rxn_env_data;
  HANDLE_ERROR(hipMalloc((void **) &mGPU->state, n_state_var * n_cells * sizeof(double)));
  HANDLE_ERROR(hipMalloc((void **) &mGPU->env, CAMP_NUM_ENV_PARAM_ * n_cells * sizeof(double)));
  hipMalloc((void **) &mGPU->rxn_env_data, md->n_rxn_env_data * n_cells * sizeof(double));
  hipMalloc((void **) &mGPU->rxn_env_data_idx, rxn_env_data_idx_size);
  hipMalloc((void **) &mGPU->map_state_deriv, map_state_deriv_size);
  int num_spec = md->n_per_cell_dep_var*n_cells;
  hipMalloc((void **) &(mGPU->production_rates),num_spec*sizeof(mGPU->production_rates));
  hipMalloc((void **) &(mGPU->loss_rates),num_spec*sizeof(mGPU->loss_rates));
  int *map_state_derivCPU = (int *)malloc(map_state_deriv_size);
  int i_dep_var = 0;
  for (int i_cell = 0; i_cell < n_cells; i_cell++) {
    for (int i_spec = 0; i_spec < n_state_var; i_spec++) {
      if (md->var_type[i_spec] == CHEM_SPEC_VARIABLE) {
        map_state_derivCPU[i_dep_var] = i_spec + i_cell * n_state_var;
        i_dep_var++;
      }
    }
  }
  HANDLE_ERROR(hipMemcpy(mGPU->map_state_deriv, map_state_derivCPU,
                          map_state_deriv_size, hipMemcpyHostToDevice));
  free(map_state_derivCPU);
  if(n_dep_var<32) {
    printf("CAMP ERROR: TOO FEW SPECIES FOR GPU (Species < 32),"
           " use CPU case instead\n");
    exit(0);
  }
  size_t deriv_size = n_dep_var * n_cells * sizeof(double);
  size_t jac_size = md->n_per_cell_solver_jac_elem * n_cells * sizeof(double);
  hipMalloc((void **) &mGPU->dA, jac_size);
  hipMalloc((void **) &mGPU->J_solver, jac_size);
  hipMalloc((void **) &mGPU->J_state, deriv_size);
  hipMalloc((void **) &mGPU->J_deriv, deriv_size);
  hipMalloc((void **) &mGPU->J_tmp, deriv_size);
  hipMalloc((void **) &mGPU->J_tmp2, deriv_size);
  hipMalloc((void **) &mGPU->jac_map, sizeof(JacMap) * md->n_mapped_values);
  HANDLE_ERROR(hipMalloc((void **) &mGPU->n_mapped_values, 1 * sizeof(int)));
  double *J_solver = SM_DATA_S(md->J_solver);
  hipMemcpy(mGPU->J_solver, J_solver, jac_size, hipMemcpyHostToDevice);

  //todo j_solver is required to also copy in cvode_run?


  double *J_state = N_VGetArrayPointer(md->J_state);
  HANDLE_ERROR(hipMemcpy(mGPU->J_state, J_state, deriv_size, hipMemcpyHostToDevice));
  double *J_deriv = N_VGetArrayPointer(md->J_deriv);
  HANDLE_ERROR(hipMemcpy(mGPU->J_deriv, J_deriv, deriv_size, hipMemcpyHostToDevice));
  double *J_tmp2 = N_VGetArrayPointer(md->J_tmp2);
  HANDLE_ERROR(hipMemcpy(mGPU->J_tmp2, J_tmp2, deriv_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->jac_map, md->jac_map, sizeof(JacMap) * md->n_mapped_values, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->n_mapped_values, &md->n_mapped_values, 1 * sizeof(int), hipMemcpyHostToDevice));
  Jacobian *jac = &sd->jac;
  JacobianGPU *jacgpu = &(mGPU->jac);
  hipMalloc((void **) &jacgpu->num_elem, 1 * sizeof(jacgpu->num_elem));
  HANDLE_ERROR(hipMemcpy(jacgpu->num_elem, &jac->num_elem, 1 * sizeof(jacgpu->num_elem), hipMemcpyHostToDevice));
  int num_elem = jac->num_elem * n_cells;
  HANDLE_ERROR(hipMalloc((void **) &(jacgpu->production_partials), num_elem * sizeof(double)));
  HANDLE_ERROR(hipMalloc((void **) &(jacgpu->loss_partials), num_elem * sizeof(double)));
  double *aux=(double*)malloc(sizeof(double)*num_elem);
  for (int i = 0; i < num_elem; i++) {
    aux[i]=0.;
  }
  HANDLE_ERROR(hipMemcpy(jacgpu->production_partials, aux, num_elem * sizeof(double), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(jacgpu->loss_partials, aux, num_elem * sizeof(double), hipMemcpyHostToDevice));
  hipMalloc((void **) &mGPU->rxn_int, (md->n_rxn_int_param + md->n_rxn)*sizeof(int));
  hipMalloc((void **) &mGPU->rxn_double, md->n_rxn_float_param*sizeof(double));
  hipMalloc((void **) &mGPU->rxn_int_indices, (md->n_rxn+1)*sizeof(int));
  hipMalloc((void **) &mGPU->rxn_float_indices, (md->n_rxn+1)*sizeof(int));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_int, md->rxn_int_data,(md->n_rxn_int_param + md->n_rxn)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_double, md->rxn_float_data, md->n_rxn_float_param*sizeof(double), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_env_data_idx, md->rxn_env_idx, rxn_env_data_idx_size, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_int_indices, md->rxn_int_indices,(md->n_rxn+1)*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(mGPU->rxn_float_indices, md->rxn_float_indices,(md->n_rxn+1)*sizeof(int), hipMemcpyHostToDevice));
  int nrows = n_dep_var * n_cells;
  mGPU->nrows = nrows;
  double ** dr0 = &mGPU->dr0;
  double ** dr0h = &mGPU->dr0h;
  double ** dn0 = &mGPU->dn0;
  double ** dp0 = &mGPU->dp0;
  double ** dt = &mGPU->dt;
  double ** ds = &mGPU->ds;
  double ** dy = &mGPU->dy;
  double ** ddiag = &mGPU->ddiag;
  hipMalloc(dr0,nrows*sizeof(double));
  hipMalloc(dr0h,nrows*sizeof(double));
  hipMalloc(dn0,nrows*sizeof(double));
  hipMalloc(dp0,nrows*sizeof(double));
  hipMalloc(dt,nrows*sizeof(double));
  hipMalloc(ds,nrows*sizeof(double));
  hipMalloc(dy,nrows*sizeof(double));
  HANDLE_ERROR(hipMalloc(ddiag,nrows*sizeof(double)));
  SUNMatrix J = cvdls_mem->A;
  double *A = ((double *) SM_DATA_S(J));
  HANDLE_ERROR(hipMemcpy(mGPU->dA, A, jac_size, hipMemcpyHostToDevice));
  int nnz = SM_NNZ_S(J);
  //Translate from int64 (sunindextype) to int
  int *jA = (int *) malloc(sizeof(int) *nnz/n_cells);
  int *iA = (int *) malloc(sizeof(int) * (nrows/n_cells + 1));
  for (int i = 0; i < nnz/n_cells; i++)
    jA[i] = SM_INDEXVALS_S(J)[i];
  for (int i = 0; i <= nrows/n_cells; i++)
    iA[i] = SM_INDEXPTRS_S(J)[i];
  hipMalloc((void **) &mGPU->djA, nnz/n_cells * sizeof(int));
  hipMalloc((void **) &mGPU->diA, (nrows/n_cells + 1) * sizeof(int));
  hipMemcpy(mGPU->djA, jA, nnz/n_cells * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->diA, iA, (nrows/n_cells + 1) * sizeof(int), hipMemcpyHostToDevice);
  double *ewt = N_VGetArrayPointer(cv_mem->cv_ewt);
  double *tempv = N_VGetArrayPointer(cv_mem->cv_tempv);
  double *cv_last_yn = N_VGetArrayPointer(cv_mem->cv_last_yn);
  double *cv_acor_init = N_VGetArrayPointer(cv_mem->cv_acor_init);
  hipMalloc((void **) &mGPU->dftemp, deriv_size);
  hipMalloc((void **) &mGPU->sCells, sizeof(ModelDataVariable)*n_cells);
  hipMalloc((void **) &mGPU->flag, 1 * sizeof(int));
  hipMalloc((void **) &mGPU->flagCells, n_cells * sizeof(int));
  hipMalloc((void **) &mGPU->dsavedJ, nnz * sizeof(double));
  hipMalloc((void **) &mGPU->dewt, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_acor, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dtempv, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dtempv1, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dtempv2, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dzn, nrows * (cv_mem->cv_qmax + 1) * sizeof(double));
  hipMalloc((void **) &mGPU->dcv_y, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->dx, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_last_yn, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_acor_init, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_acor, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->yout, nrows * sizeof(double));
  hipMalloc((void **) &mGPU->cv_l, L_MAX * n_cells * sizeof(double));
  hipMalloc((void **) &mGPU->cv_tau, (L_MAX + 1) * n_cells * sizeof(double));
  hipMalloc((void **) &mGPU->cv_tq, (NUM_TESTS + 1) * n_cells * sizeof(double));
  hipMalloc((void **) &mGPU->cv_Vabstol, nrows * sizeof(double));
  HANDLE_ERROR(hipMemset(mGPU->flagCells, CV_SUCCESS, n_cells * sizeof(int)));
  hipMemcpy(mGPU->dsavedJ, A, nnz * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->dewt, ewt, nrows * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->cv_acor, ewt, nrows * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->dftemp, ewt, nrows * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->dx, tempv, nrows * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->cv_last_yn, cv_last_yn, nrows * sizeof(double), hipMemcpyHostToDevice);
  HANDLE_ERROR(hipMemcpy(mGPU->cv_acor_init, cv_acor_init, nrows * sizeof(double), hipMemcpyHostToDevice));
  mGPU->state_size_cell = md->n_per_cell_state_var;
  int flag = 999;
  hipMemcpy(mGPU->flag, &flag, 1 * sizeof(int), hipMemcpyHostToDevice);
  mCPU->mdvCPU.nstlj = 0;
#ifdef CAMP_DEBUG_GPU
  hipEventCreate(&mCPU->startcvStep);
  hipEventCreate(&mCPU->stopcvStep);
#ifdef CAMP_PROFILE_DEVICE_FUNCTIONS
  hipMalloc((void **) &mGPU->mdvo, sizeof(ModelDataVariable));
  hipDeviceGetAttribute(&mGPU->clock_khz, hipDeviceAttributeClockRate, 0);
  mCPU->mdvCPU.countercvStep=0;
  mCPU->mdvCPU.counterBCGInternal=0;
  mCPU->mdvCPU.counterBCG=0;
  mCPU->mdvCPU.timeNewtonIteration=0.;
  mCPU->mdvCPU.timeJac=0.;
  mCPU->mdvCPU.timelinsolsetup=0.;
  mCPU->mdvCPU.timecalc_Jac=0.;
  mCPU->mdvCPU.timef=0.;
  mCPU->mdvCPU.timeguess_helper=0.;
  mCPU->mdvCPU.dtBCG=0.;
  mCPU->mdvCPU.dtcudaDeviceCVode=0.;
  mCPU->mdvCPU.dtPostBCG=0.;
  HANDLE_ERROR(hipMemcpy(mGPU->mdvo, &mCPU->mdvCPU, sizeof(ModelDataVariable), hipMemcpyHostToDevice));
#endif
#endif
  for (int i = 0; i < n_cells; i++){
    hipMemcpy(&mGPU->sCells[i], &mCPU->mdvCPU, sizeof(ModelDataVariable), hipMemcpyHostToDevice);
  }
#ifdef IS_DEBUG_MODE_CSR_ODE_GPU
  int n_row=nrows/n_cells;
  int* Ap=iA;
  int* Aj=jA;
  double* Ax=A;
  nnz=nnz/n_cells;
  int* Bp=(int*)malloc((n_row+1)*sizeof(int));
  int* Bi=(int*)malloc(nnz*sizeof(int));
  double* Bx=(double*)malloc(nnz*sizeof(double));
  memset(Bp, 0, (n_row+1)*sizeof(int));
  for (int n = 0; n < nnz; n++){
   Bp[Aj[n]]++;
  }
  for(int col = 0, cumsum = 0; col < n_row; col++){
    int temp  = Bp[col];
    Bp[col] = cumsum;
    cumsum += temp;
  }
  Bp[n_row] = nnz;
  int *mapJSPMV= (int *)malloc(nnz * sizeof(int));
  for(int row = 0; row < n_row; row++){
    for(int jj = Ap[row]; jj < Ap[row+1]; jj++){
      int col  = Aj[jj];
      int dest = Bp[col];
      Bi[dest] = row;
      Bx[dest] = Ax[jj];
      mapJSPMV[jj]=dest;
      Bp[col]++;
    }
  }
  for(int col = 0, last = 0; col <= n_row; col++){
    int temp  = Bp[col];
    Bp[col] = last;
    last    = temp;
  }
  ModelData *md = &(sd->model_data);
  nnz=md->n_mapped_values;
  int *aux_solver_id= (int *)malloc(nnz * sizeof(int));
  for (int i = 0; i < nnz; i++){
    aux_solver_id[i]=mapJSPMV[md->jac_map[i].solver_id];
  }
  free(mapJSPMV);
  int *jac_solver_id= (int *)malloc(nnz * sizeof(int));
  for (int i = 0; i < nnz; i++){
    jac_solver_id[i]=aux_solver_id[i];
    aux_solver_id[i]=md->jac_map[i].solver_id;
    md->jac_map[i].solver_id=jac_solver_id[i];
  }
  hipMemcpy(mGPU->diA, Bp, (n_row + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->djA, Bi, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->dA, Bx, nnz * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mGPU->jac_map, md->jac_map, sizeof(JacMap) * md->n_mapped_values, hipMemcpyHostToDevice);
  free(Bp);
  free(Bi);
  free(Bx);
  free(jac_solver_id);
  free(aux_solver_id);
  nnz=nnz*n_cells;
#endif

  /*
#ifdef DEV_CPU_GPU
  //CPU cvode
  sd->cvode_mem2 = CVodeCreate(CV_BDF
#ifndef SUNDIALS_VERSION_MAJOR
#  error SUNDIALS_VERSION_MAJOR not defined
#elif SUNDIALS_VERSION_MAJOR < 4
      , CV_NEWTON
#endif
  );
  int n_state_var = sd->model_data.n_per_cell_state_var;
  int n_dep_var = sd->model_data.n_per_cell_dep_var;
  int var_type = sd->model_data.var_type;
  int n_cells = 1;
  flag = CVodeSetUserData(sd->cvode_mem2, sd);
  sd->y = N_VNew_Serial(n_dep_var);
  sd->deriv = N_VNew_Serial(n_dep_var);
  flag = CVodeInit(sd->cvode_mem, f, (realtype)0.0, sd->y2);
  sd->abs_tol_nv2 = N_VNew_Serial(n_dep_var);
  i_dep_var = 0;
  for (int i_spec = 0; i_spec < n_dep_var; i_spec++)
    NV_Ith_S(sd->abs_tol_nv2, i_spec) = md->abs_tol[i_spec];
  flag = CVodeSVtolerances(sd->cvode_mem2, rel_tol, sd->abs_tol_nv2);
  flag = CVodeSetMaxNumSteps(sd->cvode_mem2, max_steps);
  flag = CVodeSetMaxConvFails(sd->cvode_mem2, max_conv_fails);
  flag = CVodeSetMaxErrTestFails(sd->cvode_mem2, max_conv_fails);
  flag = CVodeSetMaxHnilWarns(sd->cvode_mem2, -1);
  //get_jac_init



  sd->model_data.J_init = SUNMatClone(sd->J);
  SUNMatCopy(sd->J, sd->model_data.J_init);
  sd->J_guess = SUNMatClone(sd->J);
  SUNMatCopy(sd->J, sd->J_guess);
  sd->ls = SUNKLU(sd->y, sd->J);
  flag = CVDlsSetLinearSolver(sd->cvode_mem2, sd->ls, sd->J);
  flag = CVDlsSetJacFn(sd->cvode_mem2, Jac);
#ifdef CAMP_CUSTOM_CVODE
  flag = CVodeSetDlsGuessHelper(sd->cvode_mem2, guess_helper);
#endif
#endif
  */
}

void free_gpu_cu(SolverData *sd) {
  ModelDataGPU *mGPU = sd->mGPU;
  free(sd->flagCells);
  mGPU = sd->mGPU;
  hipFree(mGPU->map_state_deriv);
  hipFree(mGPU->J_solver);
  hipFree(mGPU->J_state);
  hipFree(mGPU->J_deriv);
  hipFree(mGPU->J_tmp);
  hipFree(mGPU->J_tmp2);
  hipFree(mGPU->indexvals);
  hipFree(mGPU->indexptrs);
  hipFree(mGPU->rxn_int);
  hipFree(mGPU->rxn_double);
  hipFree(mGPU->state);
  hipFree(mGPU->env);
  hipFree(mGPU->rxn_env_data);
  hipFree(mGPU->rxn_env_data_idx);
  hipFree(mGPU->production_rates);
  hipFree(mGPU->loss_rates);
  hipFree(mGPU->rxn_int_indices);
  hipFree(mGPU->rxn_float_indices);
  hipFree(mGPU->n_mapped_values);
  hipFree(mGPU->jac_map);
  hipFree(mGPU->yout);
  hipFree(mGPU->cv_Vabstol);
  hipFree(mGPU->cv_l);
  hipFree(mGPU->cv_tau);
  hipFree(mGPU->cv_tq);
  hipFree(mGPU->cv_last_yn);
  hipFree(mGPU->cv_acor_init);
  hipFree(mGPU->dA);
  hipFree(mGPU->djA);
  hipFree(mGPU->diA);
  hipFree(mGPU->dx);
  hipFree(mGPU->dtempv);
  hipFree(mGPU->ddiag);
  hipFree(mGPU->dr0);
  hipFree(mGPU->dr0h);
  hipFree(mGPU->dn0);
  hipFree(mGPU->dp0);
  hipFree(mGPU->dt);
  hipFree(mGPU->ds);
  hipFree(mGPU->dy);
  hipFree(mGPU->dz);
  hipFree(mGPU->dftemp);
  hipFree(mGPU->dcv_y);
  hipFree(mGPU->dtempv1);
  hipFree(mGPU->dtempv2);
  hipFree(mGPU->flag);
  hipFree(mGPU->flagCells);
  hipFree(mGPU->cv_acor);
  hipFree(mGPU->dzn);
  hipFree(mGPU->dewt);
  hipFree(mGPU->dsavedJ);
  hipFree(mGPU->mdv);
  hipFree(mGPU->mdvo);
}